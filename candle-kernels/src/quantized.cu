// Kernels adapted from llama.cpp ggml-cuda.cu
// https://github.com/ggerganov/llama.cpp/blob/master/ggml-cuda.cu

#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"
#include "hip/hip_bf16.h"
#include<stdint.h>

#define GGML_UNUSED(x) (void)(x)
#define GGML_CUDA_ASSUME(x)

#ifdef GGML_QKK_64
#define QK_K 64
#define K_SCALE_SIZE 4
#else
#define QK_K 256
#define K_SCALE_SIZE 12
#endif

#undef GGML_CUDA_F16
#define GGML_CUDA_DMMV_X 32
#define CUDA_QUANTIZE_BLOCK_SIZE 256
#define CUDA_DEQUANTIZE_BLOCK_SIZE 256
#define K_QUANTS_PER_ITERATION 2

typedef uint16_t ggml_fp16_t;
typedef float dfloat; // dequantize float
typedef float2 dfloat2;
typedef void (*dequantize_kernel_t)(const void * vx, const int ib, const int iqs, dfloat2 & v);

static __device__ __forceinline__ float warp_reduce_sum(float x) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        x += __shfl_xor_sync(0xffffffff, x, mask, 32);
    }
    return x;
}

static __device__ __forceinline__ float warp_reduce_max(float x) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        x = fmaxf(x, __shfl_xor_sync(0xffffffff, x, mask, 32));
    }
    return x;
}

static __device__ __forceinline__ int get_int_from_int8(const int8_t * x8, const int & i32) {
    const uint16_t * x16 = (const uint16_t *) (x8 + sizeof(int) * i32); // assume at least 2 byte alignment

    int x32 = 0;
    x32 |= x16[0] <<  0;
    x32 |= x16[1] << 16;

    return x32;
}

static __device__ __forceinline__ int get_int_from_uint8(const uint8_t * x8, const int & i32) {
    const uint16_t * x16 = (const uint16_t *) (x8 + sizeof(int) * i32); // assume at least 2 byte alignment

    int x32 = 0;
    x32 |= x16[0] <<  0;
    x32 |= x16[1] << 16;

    return x32;
}

static __device__ __forceinline__ int get_int_from_int8_aligned(const int8_t * x8, const int & i32) {
    return *((const int *) (x8 + sizeof(int) * i32)); // assume at least 4 byte alignment
}

static __device__ __forceinline__ int get_int_from_uint8_aligned(const uint8_t * x8, const int & i32) {
    return *((const int *) (x8 + sizeof(int) * i32)); // assume at least 4 byte alignment
}


#define WARP_SIZE 32
#define CUDART_HMAX     11070 // CUDA 11.7, min. ver. for which __hmax and __hmax2 are known to work (may be higher than needed)

#define CC_PASCAL     600
#define MIN_CC_DP4A   610 // minimum compute capability for __dp4a, an intrinsic for byte-wise dot products
#define CC_VOLTA      700
#define CC_OFFSET_AMD 1000000
#define CC_RDNA1      (CC_OFFSET_AMD + 1010)
#define CC_RDNA2      (CC_OFFSET_AMD + 1030)
#define CC_RDNA3      (CC_OFFSET_AMD + 1100)

static __device__ __forceinline__ int ggml_cuda_dp4a(const int a, const int b, int c) {
#if __CUDA_ARCH__ >= MIN_CC_DP4A
    return __dp4a(a, b, c);
#else // __CUDA_ARCH__ >= MIN_CC_DP4A
    const int8_t * a8 = (const int8_t *) &a;
    const int8_t * b8 = (const int8_t *) &b;
    return c + a8[0]*b8[0] + a8[1]*b8[1] + a8[2]*b8[2] + a8[3]*b8[3];
#endif // __CUDA_ARCH__ >= MIN_CC_DP4A
}


#define  MMQ_X_Q4_0_RDNA2  64
#define  MMQ_Y_Q4_0_RDNA2  128
#define NWARPS_Q4_0_RDNA2  8
#define  MMQ_X_Q4_0_RDNA1  64
#define  MMQ_Y_Q4_0_RDNA1  64
#define NWARPS_Q4_0_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q4_0_AMPERE 4
#define  MMQ_Y_Q4_0_AMPERE 32
#define NWARPS_Q4_0_AMPERE 4
#else
#define  MMQ_X_Q4_0_AMPERE 64
#define  MMQ_Y_Q4_0_AMPERE 128
#define NWARPS_Q4_0_AMPERE 4
#endif
#define  MMQ_X_Q4_0_PASCAL 64
#define  MMQ_Y_Q4_0_PASCAL 64
#define NWARPS_Q4_0_PASCAL 8

#define  MMQ_X_Q4_1_RDNA2  64
#define  MMQ_Y_Q4_1_RDNA2  128
#define NWARPS_Q4_1_RDNA2  8
#define  MMQ_X_Q4_1_RDNA1  64
#define  MMQ_Y_Q4_1_RDNA1  64
#define NWARPS_Q4_1_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q4_1_AMPERE 4
#define  MMQ_Y_Q4_1_AMPERE 32
#define NWARPS_Q4_1_AMPERE 4
#else
#define  MMQ_X_Q4_1_AMPERE 64
#define  MMQ_Y_Q4_1_AMPERE 128
#define NWARPS_Q4_1_AMPERE 4
#endif
#define  MMQ_X_Q4_1_PASCAL 64
#define  MMQ_Y_Q4_1_PASCAL 64
#define NWARPS_Q4_1_PASCAL 8

#define  MMQ_X_Q5_0_RDNA2  64
#define  MMQ_Y_Q5_0_RDNA2  128
#define NWARPS_Q5_0_RDNA2  8
#define  MMQ_X_Q5_0_RDNA1  64
#define  MMQ_Y_Q5_0_RDNA1  64
#define NWARPS_Q5_0_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q5_0_AMPERE 4
#define  MMQ_Y_Q5_0_AMPERE 32
#define NWARPS_Q5_0_AMPERE 4
#else
#define  MMQ_X_Q5_0_AMPERE 128
#define  MMQ_Y_Q5_0_AMPERE 64
#define NWARPS_Q5_0_AMPERE 4
#endif
#define  MMQ_X_Q5_0_PASCAL 64
#define  MMQ_Y_Q5_0_PASCAL 64
#define NWARPS_Q5_0_PASCAL 8

#define  MMQ_X_Q5_1_RDNA2  64
#define  MMQ_Y_Q5_1_RDNA2  128
#define NWARPS_Q5_1_RDNA2  8
#define  MMQ_X_Q5_1_RDNA1  64
#define  MMQ_Y_Q5_1_RDNA1  64
#define NWARPS_Q5_1_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q5_1_AMPERE 4
#define  MMQ_Y_Q5_1_AMPERE 32
#define NWARPS_Q5_1_AMPERE 4
#else
#define  MMQ_X_Q5_1_AMPERE 128
#define  MMQ_Y_Q5_1_AMPERE 64
#define NWARPS_Q5_1_AMPERE 4
#endif
#define  MMQ_X_Q5_1_PASCAL 64
#define  MMQ_Y_Q5_1_PASCAL 64
#define NWARPS_Q5_1_PASCAL 8

#define  MMQ_X_Q8_0_RDNA2  64
#define  MMQ_Y_Q8_0_RDNA2  128
#define NWARPS_Q8_0_RDNA2  8
#define  MMQ_X_Q8_0_RDNA1  64
#define  MMQ_Y_Q8_0_RDNA1  64
#define NWARPS_Q8_0_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q8_0_AMPERE 4
#define  MMQ_Y_Q8_0_AMPERE 32
#define NWARPS_Q8_0_AMPERE 4
#else
#define  MMQ_X_Q8_0_AMPERE 128
#define  MMQ_Y_Q8_0_AMPERE 64
#define NWARPS_Q8_0_AMPERE 4
#endif
#define  MMQ_X_Q8_0_PASCAL 64
#define  MMQ_Y_Q8_0_PASCAL 64
#define NWARPS_Q8_0_PASCAL 8

#define  MMQ_X_Q2_K_RDNA2  64
#define  MMQ_Y_Q2_K_RDNA2  128
#define NWARPS_Q2_K_RDNA2  8
#define  MMQ_X_Q2_K_RDNA1  128
#define  MMQ_Y_Q2_K_RDNA1  32
#define NWARPS_Q2_K_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q2_K_AMPERE 4
#define  MMQ_Y_Q2_K_AMPERE 32
#define NWARPS_Q2_K_AMPERE 4
#else
#define  MMQ_X_Q2_K_AMPERE 64
#define  MMQ_Y_Q2_K_AMPERE 128
#define NWARPS_Q2_K_AMPERE 4
#endif
#define  MMQ_X_Q2_K_PASCAL 64
#define  MMQ_Y_Q2_K_PASCAL 64
#define NWARPS_Q2_K_PASCAL 8

#define  MMQ_X_Q3_K_RDNA2  128
#define  MMQ_Y_Q3_K_RDNA2  64
#define NWARPS_Q3_K_RDNA2  8
#define  MMQ_X_Q3_K_RDNA1  32
#define  MMQ_Y_Q3_K_RDNA1  128
#define NWARPS_Q3_K_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q3_K_AMPERE 4
#define  MMQ_Y_Q3_K_AMPERE 32
#define NWARPS_Q3_K_AMPERE 4
#else
#define  MMQ_X_Q3_K_AMPERE 128
#define  MMQ_Y_Q3_K_AMPERE 128
#define NWARPS_Q3_K_AMPERE 4
#endif
#define  MMQ_X_Q3_K_PASCAL 64
#define  MMQ_Y_Q3_K_PASCAL 64
#define NWARPS_Q3_K_PASCAL 8

#define  MMQ_X_Q4_K_RDNA2  64
#define  MMQ_Y_Q4_K_RDNA2  128
#define NWARPS_Q4_K_RDNA2  8
#define  MMQ_X_Q4_K_RDNA1  32
#define  MMQ_Y_Q4_K_RDNA1  64
#define NWARPS_Q4_K_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q4_K_AMPERE 4
#define  MMQ_Y_Q4_K_AMPERE 32
#define NWARPS_Q4_K_AMPERE 4
#else
#define  MMQ_X_Q4_K_AMPERE 64
#define  MMQ_Y_Q4_K_AMPERE 128
#define NWARPS_Q4_K_AMPERE 4
#endif
#define  MMQ_X_Q4_K_PASCAL 64
#define  MMQ_Y_Q4_K_PASCAL 64
#define NWARPS_Q4_K_PASCAL 8

#define  MMQ_X_Q5_K_RDNA2  64
#define  MMQ_Y_Q5_K_RDNA2  128
#define NWARPS_Q5_K_RDNA2  8
#define  MMQ_X_Q5_K_RDNA1  32
#define  MMQ_Y_Q5_K_RDNA1  64
#define NWARPS_Q5_K_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q5_K_AMPERE 4
#define  MMQ_Y_Q5_K_AMPERE 32
#define NWARPS_Q5_K_AMPERE 4
#else
#define  MMQ_X_Q5_K_AMPERE 64
#define  MMQ_Y_Q5_K_AMPERE 128
#define NWARPS_Q5_K_AMPERE 4
#endif
#define  MMQ_X_Q5_K_PASCAL 64
#define  MMQ_Y_Q5_K_PASCAL 64
#define NWARPS_Q5_K_PASCAL 8

#define  MMQ_X_Q6_K_RDNA2  64
#define  MMQ_Y_Q6_K_RDNA2  128
#define NWARPS_Q6_K_RDNA2  8
#define  MMQ_X_Q6_K_RDNA1  32
#define  MMQ_Y_Q6_K_RDNA1  64
#define NWARPS_Q6_K_RDNA1  8
#if defined(CUDA_USE_TENSOR_CORES)
#define  MMQ_X_Q6_K_AMPERE 4
#define  MMQ_Y_Q6_K_AMPERE 32
#define NWARPS_Q6_K_AMPERE 4
#else
#define  MMQ_X_Q6_K_AMPERE 64
#define  MMQ_Y_Q6_K_AMPERE 64
#define NWARPS_Q6_K_AMPERE 4
#endif
#define  MMQ_X_Q6_K_PASCAL 64
#define  MMQ_Y_Q6_K_PASCAL 64
#define NWARPS_Q6_K_PASCAL 8


// QK = number of values after dequantization
// QR = QK / number of values before dequantization
// QI = number of 32 bit integers before dequantization

#define QK4_0 32
#define QR4_0 2
#define QI4_0 (QK4_0 / (4 * QR4_0))
typedef struct {
    half    d;              // delta
    uint8_t qs[QK4_0 / 2];  // nibbles / quants
} block_q4_0;
static_assert(sizeof(block_q4_0) == sizeof(ggml_fp16_t) + QK4_0 / 2, "wrong q4_0 block size/padding");

#define QK4_1 32
#define QR4_1 2
#define QI4_1 (QK4_1 / (4 * QR4_1))
typedef struct {
    half2   dm;             // dm.x = delta, dm.y = min
    uint8_t qs[QK4_1 / 2];  // nibbles / quants
} block_q4_1;
static_assert(sizeof(block_q4_1) == sizeof(ggml_fp16_t) * 2 + QK4_1 / 2, "wrong q4_1 block size/padding");

#define QK5_0 32
#define QR5_0 2
#define QI5_0 (QK5_0 / (4 * QR5_0))
typedef struct {
    half d;                 // delta
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_0 / 2];  // nibbles / quants
} block_q5_0;
static_assert(sizeof(block_q5_0) == sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_0 / 2, "wrong q5_0 block size/padding");

#define QK5_1 32
#define QR5_1 2
#define QI5_1 (QK5_1 / (4 * QR5_1))
typedef struct {
    half2 dm;               // dm.x = delta, dm.y = min
    uint8_t qh[4];          // 5-th bit of quants
    uint8_t qs[QK5_1 / 2];  // nibbles / quants
} block_q5_1;
static_assert(sizeof(block_q5_1) == 2 * sizeof(ggml_fp16_t) + sizeof(uint32_t) + QK5_1 / 2, "wrong q5_1 block size/padding");

#define QK8_0 32
#define QR8_0 1
#define QI8_0 (QK8_0 / (4 * QR8_0))
typedef struct {
    half    d;              // delta
    int8_t  qs[QK8_0];      // quants
} block_q8_0;
static_assert(sizeof(block_q8_0) == sizeof(ggml_fp16_t) + QK8_0, "wrong q8_0 block size/padding");

#define QK8_1 32
#define QR8_1 1
#define QI8_1 (QK8_1 / (4 * QR8_1))
typedef struct {
    half2   ds;             // ds.x = delta, ds.y = sum
    int8_t  qs[QK8_0];      // quants
} block_q8_1;
static_assert(sizeof(block_q8_1) == 2*sizeof(ggml_fp16_t) + QK8_0, "wrong q8_1 block size/padding");

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs);
typedef void (*allocate_tiles_cuda_t)(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc);
typedef void (*load_tiles_cuda_t)(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row);
typedef float (*vec_dot_q_mul_mat_cuda_t)(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ms, const int & i, const int & j, const int & k);

#define QR2_K 4
#define QI2_K (QK_K / (4*QR2_K))
typedef struct {
    uint8_t scales[QK_K/16]; // scales and mins, quantized with 4 bits
    uint8_t qs[QK_K/4];      // quants
    half2 dm;                // super-block scale for quantized scales/mins
} block_q2_K;
static_assert(sizeof(block_q2_K) == 2*sizeof(ggml_fp16_t) + QK_K/16 + QK_K/4, "wrong q2_K block size/padding");

#define QR3_K 4
#define QI3_K (QK_K / (4*QR3_K))
typedef struct {
    uint8_t hmask[QK_K/8];     // quants - high bit
    uint8_t qs[QK_K/4];        // quants - low 2 bits
#ifdef GGML_QKK_64
    uint8_t scales[2]; // scales, quantized with 8 bits
#else
    uint8_t scales[K_SCALE_SIZE]; // scales, quantized with 6 bits
#endif
    half d;             // super-block scale
} block_q3_K;
//static_assert(sizeof(block_q3_K) == sizeof(ggml_fp16_t) + QK_K / 4 + QK_K / 8 + K_SCALE_SIZE, "wrong q3_K block size/padding");

#define QR4_K 2
#define QI4_K (QK_K / (4*QR4_K))
#ifdef GGML_QKK_64
typedef struct {
    half    dm[2];             // super-block scales/mins
    uint8_t scales[2];         // 4-bit block scales/mins
    uint8_t qs[QK_K/2];        // 4--bit quants
} block_q4_K;
static_assert(sizeof(block_q4_K) == sizeof(half2) + QK_K/2 + 2, "wrong q4_K block size/padding");
#else
typedef struct {
    half2 dm;                  // super-block scale for quantized scales/mins
    uint8_t scales[3*QK_K/64]; // scales, quantized with 6 bits
    uint8_t qs[QK_K/2];        // 4--bit quants
} block_q4_K;
static_assert(sizeof(block_q4_K) == 2*sizeof(ggml_fp16_t) + 3*QK_K/64 + QK_K/2, "wrong q4_K block size/padding");
#endif

#define QR5_K 2
#define QI5_K (QK_K / (4*QR5_K))
#ifdef GGML_QKK_64
typedef struct {
    half d;                  // super-block scale
    int8_t scales[QK_K/16];  // block scales
    uint8_t qh[QK_K/8];      // quants, high bit
    uint8_t qs[QK_K/2];      // quants, low 4 bits
} block_q5_K;
static_assert(sizeof(block_q5_K) == sizeof(ggml_fp16_t) + QK_K/2 + QK_K/8 + QK_K/16, "wrong q5_K block size/padding");
#else
typedef struct {
    half2 dm;                     // super-block scale for quantized scales/mins
    uint8_t scales[K_SCALE_SIZE]; // scales and mins, quantized with 6 bits
    uint8_t qh[QK_K/8];           // quants, high bit
    uint8_t qs[QK_K/2];           // quants, low 4 bits
} block_q5_K;
static_assert(sizeof(block_q5_K) == 2*sizeof(ggml_fp16_t) + K_SCALE_SIZE + QK_K/2 + QK_K/8, "wrong q5_K block size/padding");
#endif

#define QR6_K 2
#define QI6_K (QK_K / (4*QR6_K))
typedef struct {
    uint8_t ql[QK_K/2];   // quants, lower 4 bits
    uint8_t qh[QK_K/4];   // quants, upper 2 bits
    int8_t  scales[QK_K/16]; // scales
    half    d;         // delta
} block_q6_K;
static_assert(sizeof(block_q6_K) == sizeof(ggml_fp16_t) + 13*QK_K/16, "wrong q6_K block size/padding");

// In llama.cpp this is only used for intermediate quantization and dot products
typedef struct {
    float   d;              // delta
    int8_t  qs[QK_K];       // quants
    int16_t bsums[QK_K/16]; // sum of quants in groups of 16
} block_q8_K;
static_assert(sizeof(block_q8_K) == sizeof(float) + QK_K + QK_K/16*sizeof(int16_t), "wrong q8_K block size/padding");


template <int qk, int qr, int qi, bool need_sum, typename block_q_t, int mmq_x, int mmq_y, int nwarps,
              allocate_tiles_cuda_t allocate_tiles, load_tiles_cuda_t load_tiles, int vdr, vec_dot_q_mul_mat_cuda_t vec_dot>
static __device__ __forceinline__ void mul_mat_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {

    const block_q_t  * x = (const block_q_t  *) vx;
    const block_q8_1 * y = (const block_q8_1 *) vy;

    const int blocks_per_row_x = ncols_x / qk;
    const int blocks_per_col_y = nrows_y / QK8_1;
    const int blocks_per_warp = WARP_SIZE / qi;

    const int & ncols_dst = ncols_y;

    const int row_dst_0 = blockIdx.x*mmq_y;
    const int & row_x_0 = row_dst_0;

    const int col_dst_0 = blockIdx.y*mmq_x;
    const int & col_y_0 = col_dst_0;

    int   * tile_x_ql = nullptr;
    half2 * tile_x_dm = nullptr;
    int   * tile_x_qh = nullptr;
    int   * tile_x_sc = nullptr;

    allocate_tiles(&tile_x_ql, &tile_x_dm, &tile_x_qh, &tile_x_sc);

    __shared__ int    tile_y_qs[mmq_x * WARP_SIZE];
    __shared__ half2  tile_y_ds[mmq_x * WARP_SIZE/QI8_1];

    float sum[mmq_y/WARP_SIZE][mmq_x/nwarps] = {{0.0f}};

    for (int ib0 = 0; ib0 < blocks_per_row_x; ib0 += blocks_per_warp) {

        load_tiles(x + row_x_0*blocks_per_row_x + ib0, tile_x_ql, tile_x_dm, tile_x_qh, tile_x_sc,
                   threadIdx.y, nrows_x-row_x_0-1, threadIdx.x, blocks_per_row_x);

#pragma unroll
        for (int ir = 0; ir < qr; ++ir) {
            const int kqs = ir*WARP_SIZE + threadIdx.x;
            const int kbxd = kqs / QI8_1;

#pragma unroll
            for (int i = 0; i < mmq_x; i += nwarps) {
                const int col_y_eff = min(col_y_0 + threadIdx.y + i, ncols_y-1); // to prevent out-of-bounds memory accesses

                const block_q8_1 * by0 = &y[col_y_eff*blocks_per_col_y + ib0 * (qk/QK8_1) + kbxd];

                const int index_y = (threadIdx.y + i) * WARP_SIZE + kqs % WARP_SIZE;
                tile_y_qs[index_y] = get_int_from_int8_aligned(by0->qs, threadIdx.x % QI8_1);
            }

#pragma unroll
            for (int ids0 = 0; ids0 < mmq_x; ids0 += nwarps * QI8_1) {
                const int ids = (ids0 + threadIdx.y * QI8_1 + threadIdx.x / (WARP_SIZE/QI8_1)) % mmq_x;
                const int kby = threadIdx.x % (WARP_SIZE/QI8_1);
                const int col_y_eff = min(col_y_0 + ids, ncols_y-1);

                // if the sum is not needed it's faster to transform the scale to f32 ahead of time
                const half2 * dsi_src = &y[col_y_eff*blocks_per_col_y + ib0 * (qk/QK8_1) + ir*(WARP_SIZE/QI8_1) + kby].ds;
                half2       * dsi_dst = &tile_y_ds[ids * (WARP_SIZE/QI8_1) + kby];
                if (need_sum) {
                    *dsi_dst = *dsi_src;
                } else {
                    float * dfi_dst = (float *) dsi_dst;
                    *dfi_dst = __low2half(*dsi_src);
                }
            }

            __syncthreads();

// #pragma unroll // unrolling this loop causes too much register pressure
            for (int k = ir*WARP_SIZE/qr; k < (ir+1)*WARP_SIZE/qr; k += vdr) {
#pragma unroll
                for (int j = 0; j < mmq_x; j += nwarps) {
#pragma unroll
                    for (int i = 0; i < mmq_y; i += WARP_SIZE) {
                        sum[i/WARP_SIZE][j/nwarps] += vec_dot(
                            tile_x_ql, tile_x_dm, tile_x_qh, tile_x_sc, tile_y_qs, tile_y_ds,
                            threadIdx.x + i, threadIdx.y + j, k);
                    }
                }
            }

            __syncthreads();
        }
    }

#pragma unroll
    for (int j = 0; j < mmq_x; j += nwarps) {
        const int col_dst = col_dst_0 + j + threadIdx.y;

        if (col_dst >= ncols_dst) {
            return;
        }

#pragma unroll
        for (int i = 0; i < mmq_y; i += WARP_SIZE) {
            const int row_dst = row_dst_0 + threadIdx.x + i;

            if (row_dst >= nrows_dst) {
                continue;
            }

            dst[col_dst*nrows_dst + row_dst] = sum[i/WARP_SIZE][j/nwarps];
        }
    }
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q4_0(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    (void)x_qh; (void)x_sc;

    const int kbx  = k / QI4_0;
    const int kqsx = k % QI4_0;

    const block_q4_0 * bx0 = (const block_q4_0 *) vx;

    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_0 * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8(bxi->qs, kqsx);
        // x_dmf[i * (WARP_SIZE/QI4_0) + i / QI4_0 + kbx] = bxi->d;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI4_0;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI4_0) {
        int i = i0 + i_offset * QI4_0 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_0 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI4_0) + i / QI4_0 + kbxd] = bxi->d;
    }
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q4_1(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI4_1;
    const int kqsx = k % QI4_1;

    const block_q4_1 * bx0 = (const block_q4_1 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_1 * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8_aligned(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI4_1;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI4_1) {
        int i = i0 + i_offset * QI4_1 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_1 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI4_1) + i / QI4_1 + kbxd] = bxi->dm;
    }
}


template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q4_0(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    (void)x_qh; (void)x_sc;

    __shared__ int  tile_x_qs[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ float tile_x_d[mmq_y * (WARP_SIZE/QI4_0) + mmq_y/QI4_0];

    *x_ql = tile_x_qs;
    *x_dm = (half2 *) tile_x_d;
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q4_1(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int   tile_x_qs[mmq_y * (WARP_SIZE) +     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI4_1) + mmq_y/QI4_1];

    *x_ql = tile_x_qs;
    *x_dm = tile_x_dm;
}

static __device__ __forceinline__ void dequantize_q4_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q4_0 * x = (const block_q4_0 *) vx;

    const dfloat d = x[ib].d;

    const int vui = x[ib].qs[iqs];

    v.x = vui & 0xF;
    v.y = vui >> 4;

#ifdef GGML_CUDA_F16
    v = __hsub2(v, {8.0f, 8.0f});
    v = __hmul2(v, {d, d});
#else
    v.x = (v.x - 8.0f) * d;
    v.y = (v.y - 8.0f) * d;
#endif // GGML_CUDA_F16
}

static __device__ __forceinline__ void dequantize_q4_1(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q4_1 * x = (const block_q4_1 *) vx;

    const dfloat d = __low2half(x[ib].dm);
    const dfloat m = __high2half(x[ib].dm);

    const int vui = x[ib].qs[iqs];

    v.x = vui & 0xF;
    v.y = vui >> 4;

#ifdef GGML_CUDA_F16
    v = __hmul2(v, {d, d});
    v = __hadd2(v, {m, m});
#else
    v.x = (v.x * d) + m;
    v.y = (v.y * d) + m;
#endif // GGML_CUDA_F16
}

static __device__ __forceinline__ void dequantize_q5_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q5_0 * x = (const block_q5_0 *) vx;

    const dfloat d = x[ib].d;

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const int xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const int xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    v.x = ((x[ib].qs[iqs] & 0xf) | xh_0);
    v.y = ((x[ib].qs[iqs] >>  4) | xh_1);

#ifdef GGML_CUDA_F16
    v = __hsub2(v, {16.0f, 16.0f});
    v = __hmul2(v, {d, d});
#else
    v.x = (v.x - 16.0f) * d;
    v.y = (v.y - 16.0f) * d;
#endif // GGML_CUDA_F16
}

static __device__ __forceinline__ void dequantize_q5_1(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q5_1 * x = (const block_q5_1 *) vx;

    const dfloat d = __low2half(x[ib].dm);
    const dfloat m = __high2half(x[ib].dm);

    uint32_t qh;
    memcpy(&qh, x[ib].qh, sizeof(qh));

    const int xh_0 = ((qh >> (iqs +  0)) << 4) & 0x10;
    const int xh_1 = ((qh >> (iqs + 12))     ) & 0x10;

    v.x = ((x[ib].qs[iqs] & 0xf) | xh_0);
    v.y = ((x[ib].qs[iqs] >>  4) | xh_1);

#ifdef GGML_CUDA_F16
    v = __hmul2(v, {d, d});
    v = __hadd2(v, {m, m});
#else
    v.x = (v.x * d) + m;
    v.y = (v.y * d) + m;
#endif // GGML_CUDA_F16
}

static __device__ __forceinline__ void dequantize_q8_0(const void * vx, const int ib, const int iqs, dfloat2 & v){
    const block_q8_0 * x = (const block_q8_0 *) vx;

    const dfloat d = x[ib].d;

    v.x = x[ib].qs[iqs + 0];
    v.y = x[ib].qs[iqs + 1];

#ifdef GGML_CUDA_F16
    v = __hmul2(v, {d, d});
#else
    v.x *= d;
    v.y *= d;
#endif // GGML_CUDA_F16
}


template <int qk, int qr, dequantize_kernel_t dequantize_kernel, typename dst_t>
static __device__ void dequantize_block(const void * __restrict__ vx, dst_t * __restrict__ y, const int k) {
    const int i = 2*(blockDim.x*blockIdx.x + threadIdx.x);

    if (i >= k) {
        return;
    }

    const int ib = i/qk; // block index
    const int iqs = (i%qk)/qr; // quant index
    const int iybs = i - i%qk; // y block start index
    const int y_offset = qr == 1 ? 1 : qk/2;

    // dequantize
    dfloat2 v;
    dequantize_kernel(vx, ib, iqs, v);

    y[iybs + iqs + 0]        = v.x;
    y[iybs + iqs + y_offset] = v.y;
}

template<typename dst_t>
static __device__ void dequantize_block_q4_0(const void * __restrict__ vx, dst_t * __restrict__ yy, int nb32) {

    const int64_t i = blockIdx.x;

    // assume 32 threads
    const int tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int64_t ib = 8*i + ir;
    if (ib >= nb32) {
        return;
    }

    dst_t * y = yy + 256*i + 32*ir + 4*il;

    const block_q4_0 * x = (const block_q4_0 *)vx + ib;
    const float d = __half2float(x->d);
    const float dm = -8*d;

    const uint8_t * q = x->qs + 4*il;

    for (int l = 0; l < 4; ++l) {
        y[l+ 0] = d * (q[l] & 0xF) + dm;
        y[l+16] = d * (q[l] >>  4) + dm;
    }
}

template<typename dst_t>
static __device__ void dequantize_block_q4_1(const void * __restrict__ vx, dst_t * __restrict__ yy, int nb32) {

    const int64_t i = blockIdx.x;

    // assume 32 threads
    const int tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int64_t ib = 8*i + ir;
    if (ib >= nb32) {
        return;
    }

    dst_t * y = yy + 256*i + 32*ir + 4*il;

    const block_q4_1 * x = (const block_q4_1 *)vx + ib;
    const float2 d = __half22float2(x->dm);

    const uint8_t * q = x->qs + 4*il;

    for (int l = 0; l < 4; ++l) {
        y[l+ 0] = d.x * (q[l] & 0xF) + d.y;
        y[l+16] = d.x * (q[l] >>  4) + d.y;
    }
}

//================================== k-quants

template<typename dst_t>
static __device__ void dequantize_block_q2_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int i   = blockIdx.x;
    const block_q2_K * x = (const block_q2_K *) vx;

    const int tid = threadIdx.x;
#if QK_K == 256
    const int n   = tid/32;
    const int l   = tid - 32*n;
    const int is  = 8*n + l/16;

    const uint8_t q = x[i].qs[32*n + l];
    dst_t * y = yy + i*QK_K + 128*n;

    float dall = __low2half(x[i].dm);
    float dmin = __high2half(x[i].dm);
    y[l+ 0] = dall * (x[i].scales[is+0] & 0xF) * ((q >> 0) & 3) - dmin * (x[i].scales[is+0] >> 4);
    y[l+32] = dall * (x[i].scales[is+2] & 0xF) * ((q >> 2) & 3) - dmin * (x[i].scales[is+2] >> 4);
    y[l+64] = dall * (x[i].scales[is+4] & 0xF) * ((q >> 4) & 3) - dmin * (x[i].scales[is+4] >> 4);
    y[l+96] = dall * (x[i].scales[is+6] & 0xF) * ((q >> 6) & 3) - dmin * (x[i].scales[is+6] >> 4);
#else
    const int is = tid/16;  // 0 or 1
    const int il = tid%16;  // 0...15
    const uint8_t q = x[i].qs[il] >> (2*is);
    dst_t * y = yy + i*QK_K + 16*is + il;
    float dall = __low2half(x[i].dm);
    float dmin = __high2half(x[i].dm);
    y[ 0] = dall * (x[i].scales[is+0] & 0xF) * ((q >> 0) & 3) - dmin * (x[i].scales[is+0] >> 4);
    y[32] = dall * (x[i].scales[is+2] & 0xF) * ((q >> 4) & 3) - dmin * (x[i].scales[is+2] >> 4);
#endif

}

template<typename dst_t>
static __device__ void dequantize_block_q3_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {

    const int i = blockIdx.x;
    const block_q3_K * x = (const block_q3_K *) vx;

#if QK_K == 256
    const int r = threadIdx.x/4;
    const int tid = r/2;
    const int is0 = r%2;
    const int l0 = 16*is0 + 4*(threadIdx.x%4);
    const int n = tid / 4;
    const int j = tid - 4*n;

    uint8_t m = 1 << (4*n + j);
    int is = 8*n + 2*j + is0;
    int shift = 2*j;

    int8_t us = is <  4 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+8] >> 0) & 3) << 4) :
                is <  8 ? (x[i].scales[is-0] & 0xF) | (((x[i].scales[is+4] >> 2) & 3) << 4) :
                is < 12 ? (x[i].scales[is-8] >>  4) | (((x[i].scales[is+0] >> 4) & 3) << 4) :
                          (x[i].scales[is-8] >>  4) | (((x[i].scales[is-4] >> 6) & 3) << 4);
    float d_all = x[i].d;
    float dl = d_all * (us - 32);

    dst_t * y = yy + i*QK_K + 128*n + 32*j;
    const uint8_t * q = x[i].qs + 32*n;
    const uint8_t * hm = x[i].hmask;

    for (int l = l0; l < l0+4; ++l) y[l] = dl * ((int8_t)((q[l] >> shift) & 3) - ((hm[l] & m) ? 0 : 4));
#else
    const int tid = threadIdx.x;
    const int is  = tid/16;  // 0 or 1
    const int il  = tid%16;  // 0...15
    const int im  = il/8;    // 0...1
    const int in  = il%8;    // 0...7

    dst_t * y = yy + i*QK_K + 16*is + il;

    const uint8_t q = x[i].qs[il] >> (2*is);
    const uint8_t h = x[i].hmask[in] >> (2*is + im);
    const float   d = (float)x[i].d;

    if (is == 0) {
        y[ 0] = d * ((x[i].scales[0] & 0xF) - 8) * ((int8_t)((q >> 0) & 3) - ((h >> 0) & 1 ? 0 : 4));
        y[32] = d * ((x[i].scales[1] & 0xF) - 8) * ((int8_t)((q >> 4) & 3) - ((h >> 4) & 1 ? 0 : 4));
    } else {
        y[ 0] = d * ((x[i].scales[0] >>  4) - 8) * ((int8_t)((q >> 0) & 3) - ((h >> 0) & 1 ? 0 : 4));
        y[32] = d * ((x[i].scales[1] >>  4) - 8) * ((int8_t)((q >> 4) & 3) - ((h >> 4) & 1 ? 0 : 4));
    }
#endif

}

#if QK_K == 256
static inline __device__ void get_scale_min_k4(int j, const uint8_t * q, uint8_t & d, uint8_t & m) {
    if (j < 4) {
        d = q[j] & 63; m = q[j + 4] & 63;
    } else {
        d = (q[j+4] & 0xF) | ((q[j-4] >> 6) << 4);
        m = (q[j+4] >>  4) | ((q[j-0] >> 6) << 4);
    }
}
#endif

template<typename dst_t>
static __device__ void dequantize_block_q4_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q4_K * x = (const block_q4_K *) vx;

    const int i = blockIdx.x;

#if QK_K == 256
    // assume 32 threads
    const int tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int is  = 2*il;
    const int n   = 4;

    dst_t * y = yy + i*QK_K + 64*il + n*ir;

    const float dall = __low2half(x[i].dm);
    const float dmin = __high2half(x[i].dm);

    const uint8_t * q = x[i].qs + 32*il + n*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const float d1 = dall * sc; const float m1 = dmin * m;
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const float d2 = dall * sc; const float m2 = dmin * m;
    for (int l = 0; l < n; ++l) {
        y[l + 0] = d1 * (q[l] & 0xF) - m1;
        y[l +32] = d2 * (q[l] >>  4) - m2;
    }
#else
    const int tid = threadIdx.x;
    const uint8_t * q = x[i].qs;
    dst_t * y = yy + i*QK_K;
    const float d = (float)x[i].dm[0];
    const float m = (float)x[i].dm[1];
    y[tid+ 0] = d * (x[i].scales[0] & 0xF) * (q[tid] & 0xF) - m * (x[i].scales[0] >> 4);
    y[tid+32] = d * (x[i].scales[1] & 0xF) * (q[tid] >>  4) - m * (x[i].scales[1] >> 4);
#endif
}

template<typename dst_t>
static __device__ void dequantize_block_q5_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q5_K * x = (const block_q5_K *) vx;

    const int i = blockIdx.x;

#if QK_K == 256
    // assume 64 threads - this is very slightly better than the one below
    const int tid = threadIdx.x;
    const int il  = tid/16;   // il is in 0...3
    const int ir  = tid%16;   // ir is in 0...15
    const int is  = 2*il;     // is is in 0...6

    dst_t * y = yy + i*QK_K + 64*il + 2*ir;

    const float dall = __low2half(x[i].dm);
    const float dmin = __high2half(x[i].dm);

    const uint8_t * ql = x[i].qs + 32*il + 2*ir;
    const uint8_t * qh = x[i].qh + 2*ir;

    uint8_t sc, m;
    get_scale_min_k4(is + 0, x[i].scales, sc, m);
    const float d1 = dall * sc; const float m1 = dmin * m;
    get_scale_min_k4(is + 1, x[i].scales, sc, m);
    const float d2 = dall * sc; const float m2 = dmin * m;

    uint8_t   hm  = 1 << (2*il);
    y[ 0] = d1 * ((ql[ 0] & 0xF) + (qh[ 0] & hm ? 16 : 0)) - m1;
    y[ 1] = d1 * ((ql[ 1] & 0xF) + (qh[ 1] & hm ? 16 : 0)) - m1;
    hm <<= 1;
    y[32] = d2 * ((ql[ 0] >>  4) + (qh[ 0] & hm ? 16 : 0)) - m2;
    y[33] = d2 * ((ql[ 1] >>  4) + (qh[ 1] & hm ? 16 : 0)) - m2;
#else
    const int tid = threadIdx.x;
    const uint8_t q = x[i].qs[tid];
    const int im = tid/8;  // 0...3
    const int in = tid%8;  // 0...7
    const int is = tid/16; // 0 or 1
    const uint8_t h = x[i].qh[in] >> im;
    const float d = x[i].d;
    dst_t * y = yy + i*QK_K + tid;
    y[ 0] = d * x[i].scales[is+0] * ((q & 0xF) - ((h >> 0) & 1 ? 0 : 16));
    y[32] = d * x[i].scales[is+2] * ((q >>  4) - ((h >> 4) & 1 ? 0 : 16));
#endif
}

template<typename dst_t>
static __device__ void dequantize_block_q6_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q6_K * x = (const block_q6_K *) vx;

    const int64_t i = blockIdx.x;
#if QK_K == 256

    // assume 64 threads - this is very slightly better than the one below
    const int64_t tid = threadIdx.x;
    const int64_t ip  = tid/32;   // ip is 0 or 1
    const int64_t il  = tid - 32*ip; // 0...32
    const int64_t is  = 8*ip + il/16;

    dst_t * y = yy + i*QK_K + 128*ip + il;

    const float d = x[i].d;

    const uint8_t * ql = x[i].ql + 64*ip + il;
    const uint8_t   qh = x[i].qh[32*ip + il];
    const int8_t  * sc = x[i].scales + is;

    y[ 0] = d * sc[0] * ((int8_t)((ql[ 0] & 0xF) | (((qh >> 0) & 3) << 4)) - 32);
    y[32] = d * sc[2] * ((int8_t)((ql[32] & 0xF) | (((qh >> 2) & 3) << 4)) - 32);
    y[64] = d * sc[4] * ((int8_t)((ql[ 0]  >> 4) | (((qh >> 4) & 3) << 4)) - 32);
    y[96] = d * sc[6] * ((int8_t)((ql[32]  >> 4) | (((qh >> 6) & 3) << 4)) - 32);
#else

    // assume 32 threads
    const int64_t tid = threadIdx.x;
    const int64_t ip  = tid/16;         // 0 or 1
    const int64_t il  = tid - 16*ip;    // 0...15

    dst_t * y = yy + i*QK_K + 16*ip + il;

    const float d = x[i].d;

    const uint8_t   ql = x[i].ql[16*ip + il];
    const uint8_t   qh = x[i].qh[il] >> (2*ip);
    const int8_t  * sc = x[i].scales;

    y[ 0] = d * sc[ip+0] * ((int8_t)((ql & 0xF) | (((qh >> 0) & 3) << 4)) - 32);
    y[32] = d * sc[ip+2] * ((int8_t)((ql  >> 4) | (((qh >> 4) & 3) << 4)) - 32);
#endif
}

template<typename dst_t>
static __device__ void dequantize_block_q8_0(const void * __restrict__ vx, dst_t * __restrict__ yy, int nb32) {
    const int i = blockIdx.x;

    // assume 32 threads
    const int tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int ib = 8*i + ir;
    if (ib >= nb32) {
        return;
    }

    dst_t * y = yy + 256*i + 32*ir + 8*il;

    const block_q8_0 * x = (const block_q8_0 *)vx + ib;
    const float d = __half2float(x->d);

    const int8_t * q = x->qs + 8*il;

    for (int l = 0; l < 8; ++l) {
        y[l] = d * q[l];
    }
}

template<typename dst_t>
static __device__ void dequantize_block_q8_K(const void * __restrict__ vx, dst_t * __restrict__ yy) {
    const block_q8_K * x = (const block_q8_K *) vx;

    const int i = blockIdx.x;

#if QK_K == 256
    // assume 32 threads
    const int tid = threadIdx.x;
    const int il  = tid/8;
    const int ir  = tid%8;
    const int n   = 8;

    dst_t * y = yy + i*QK_K + 64*il + n*ir;

    const int8_t * q = x[i].qs + 64*il + n*ir;

    for (int l = 0; l < n; ++l) {
        y[l] = q[l] * x[i].d;
    }
#else
    const int tid = threadIdx.x;
    const uint8_t * q = x[i].qs;
    float * y = yy + i*QK_K;
    y[tid] = x[i].d * x[i].scales[0];
#endif
}

template<typename dst_t>
static __device__ void dequantize_block_q5_0(const void * __restrict__ vx, dst_t * __restrict__ yy, int nb32) {
  return dequantize_block<QK5_0, QR5_0, dequantize_q5_0>(vx, yy, nb32);
}

template<typename dst_t>
static __device__ void dequantize_block_q5_1(const void * __restrict__ vx, dst_t * __restrict__ yy, int nb32) {
  return dequantize_block<QK5_1, QR5_1, dequantize_q5_1>(vx, yy, nb32);
}

#define DEQUANTIZE_K(QNAME) \
extern "C" __global__ void dequantize_block_##QNAME##_f32(const void * __restrict__ vx, float * __restrict__ y) { \
  dequantize_block_##QNAME(vx, y); \
} \
extern "C" __global__ void dequantize_block_##QNAME##_f16(const void * __restrict__ vx, half * __restrict__ y) { \
  dequantize_block_##QNAME(vx, y); \
} \

#define DEQUANTIZE(QNAME) \
extern "C" __global__ void dequantize_block_##QNAME##_f32(const void * __restrict__ vx, float * __restrict__ y, const int k) { \
  dequantize_block_##QNAME(vx, y, k); \
} \
extern "C" __global__ void dequantize_block_##QNAME##_f16(const void * __restrict__ vx, half * __restrict__ y, const int k) { \
  dequantize_block_##QNAME(vx, y, k); \
} \

DEQUANTIZE_K(q2_K)
DEQUANTIZE_K(q3_K)
DEQUANTIZE_K(q4_K)
DEQUANTIZE_K(q5_K)
DEQUANTIZE_K(q6_K)
DEQUANTIZE_K(q8_K)
DEQUANTIZE(q4_0)
DEQUANTIZE(q4_1)
DEQUANTIZE(q5_0)
DEQUANTIZE(q5_1)
DEQUANTIZE(q8_0)

template <int qk, int qr, dequantize_kernel_t dequantize_kernel>
static __device__ void dequantize_mul_mat_vec(const void * __restrict__ vx, const dfloat * __restrict__ y, float * __restrict__ dst, const int ncols, const int nrows) {
    // qk = quantized weights per x block
    // qr = number of quantized weights per data value in x block
    const int row = blockIdx.x*blockDim.y + threadIdx.y;

    if (row >= nrows) {
        return;
    }

    const int tid = threadIdx.x;

    const int iter_stride = 2*GGML_CUDA_DMMV_X;
    const int vals_per_iter = iter_stride / WARP_SIZE; // num quantized vals per thread and i iter
    const int y_offset = qr == 1 ? 1 : qk/2;

// partial sum for each thread
#ifdef GGML_CUDA_F16
    half2 tmp = {0.0f, 0.0f}; // two sums for f16 to take advantage of half2 intrinsics
#else
    float tmp = 0.0f;
#endif // GGML_CUDA_F16

    for (int i = 0; i < ncols; i += iter_stride) {
        const int col = i + vals_per_iter*tid;
        const int ib = (row*ncols + col)/qk; // x block index
        const int iqs = (col%qk)/qr; // x quant index
        const int iybs = col - col%qk; // y block start index

// processing >2 values per i iter is faster for fast GPUs
#pragma unroll
        for (int j = 0; j < vals_per_iter; j += 2) {
            // process 2 vals per j iter

            // dequantize
            // for qr = 2 the iqs needs to increase by 1 per j iter because 2 weights per data val
            dfloat2 v;
            dequantize_kernel(vx, ib, iqs + j/qr, v);

            // matrix multiplication
            // for qr = 2 the y index needs to increase by 1 per j iter because of y_offset = qk/2
#ifdef GGML_CUDA_F16
            tmp += __hmul2(v, {
                y[iybs + iqs + j/qr + 0],
                y[iybs + iqs + j/qr + y_offset]
            });
#else
            tmp += v.x * y[iybs + iqs + j/qr + 0];
            tmp += v.y * y[iybs + iqs + j/qr + y_offset];
#endif // GGML_CUDA_F16
        }
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
#ifdef GGML_CUDA_F16
        dst[row] = tmp.x + tmp.y;
#else
        dst[row] = tmp;
#endif // GGML_CUDA_F16
    }
}

extern "C" __global__ void dequantize_mul_mat_vec_q4_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows) {
    dequantize_mul_mat_vec<QK4_0, QR4_0, dequantize_q4_0>(vx, y, dst, ncols, nrows);
}

extern "C" __global__ void dequantize_mul_mat_vec_q4_1_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows) {
    dequantize_mul_mat_vec<QK4_1, QR4_1, dequantize_q4_1>(vx, y, dst, ncols, nrows);
}

extern "C" __global__ void dequantize_mul_mat_vec_q5_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows) {
    dequantize_mul_mat_vec<QK5_0, QR5_0, dequantize_q5_0>(vx, y, dst, ncols, nrows);
}

extern "C" __global__ void dequantize_mul_mat_vec_q5_1_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows) {
    dequantize_mul_mat_vec<QK5_1, QR5_1, dequantize_q5_1>(vx, y, dst, ncols, nrows);
}
extern "C" __global__ void dequantize_mul_mat_vec_q8_0_cuda(const void * vx, const dfloat * y, float * dst, const int ncols, const int nrows) {
    dequantize_mul_mat_vec<QK8_0, QR8_0, dequantize_q8_0>(vx, y, dst, ncols, nrows);
}

extern "C" __global__ void dequantize_mul_mat_vec_q2_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q2_K * x = (const block_q2_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

#if QK_K == 256
    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...15
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 16/K_QUANTS_PER_ITERATION;

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15 or 0...14 in steps of 2
    const int q_offset = 32*im + l0;
    const int s_offset = 8*im;
    const int y_offset = 128*im + l0;

    uint32_t aux[4];
    const uint8_t * d = (const uint8_t *)aux;
    const uint8_t * m = (const uint8_t *)(aux + 2);

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;

        const float dall = __low2half(x[i].dm);
        const float dmin = __high2half(x[i].dm);

        const uint32_t * a = (const uint32_t *)(x[i].scales + s_offset);
        aux[0] = a[0] & 0x0f0f0f0f;
        aux[1] = a[1] & 0x0f0f0f0f;
        aux[2] = (a[0] >> 4) & 0x0f0f0f0f;
        aux[3] = (a[1] >> 4) & 0x0f0f0f0f;

        float sum1 = 0, sum2 = 0;
        for (int l = 0; l < K_QUANTS_PER_ITERATION; ++l) {
            sum1 += y[l+ 0] * d[0] * ((q[l+ 0] >> 0) & 3)
                  + y[l+32] * d[2] * ((q[l+ 0] >> 2) & 3)
                  + y[l+64] * d[4] * ((q[l+ 0] >> 4) & 3)
                  + y[l+96] * d[6] * ((q[l+ 0] >> 6) & 3)
                  + y[l+16] * d[1] * ((q[l+16] >> 0) & 3)
                  + y[l+48] * d[3] * ((q[l+16] >> 2) & 3)
                  + y[l+80] * d[5] * ((q[l+16] >> 4) & 3)
                  +y[l+112] * d[7] * ((q[l+16] >> 6) & 3);
            sum2 += y[l+ 0] * m[0] + y[l+32] * m[2] + y[l+64] * m[4] + y[ l+96] * m[6]
                  + y[l+16] * m[1] + y[l+48] * m[3] + y[l+80] * m[5] + y[l+112] * m[7];

        }
        tmp += dall * sum1 - dmin * sum2;

    }
#else
    const int tid = threadIdx.x/(2*K_QUANTS_PER_ITERATION);  // 0...15 or 0...7
    const int ix  = threadIdx.x%(2*K_QUANTS_PER_ITERATION);  // 0....1 or 0...3
    const int offset = tid * K_QUANTS_PER_ITERATION;

    uint32_t uaux[2];
    const uint8_t * d = (const uint8_t *)uaux;

    for (int i = ix; i < num_blocks_per_row; i += 2*K_QUANTS_PER_ITERATION) {

        const float   * y = yy + i * QK_K + offset;
        const uint8_t * q = x[i].qs + offset;
        const uint32_t * s = (const uint32_t *)x[i].scales;

        uaux[0] = s[0] & 0x0f0f0f0f;
        uaux[1] = (s[0] >> 4) & 0x0f0f0f0f;

        const float2 dall = __half22float2(x[i].dm);

        float sum1 = 0, sum2 = 0;
        for (int l = 0; l < K_QUANTS_PER_ITERATION; ++l) {
            const uint8_t ql = q[l];
            sum1 += y[l+ 0] * d[0] * ((ql >> 0) & 3)
                  + y[l+16] * d[1] * ((ql >> 2) & 3)
                  + y[l+32] * d[2] * ((ql >> 4) & 3)
                  + y[l+48] * d[3] * ((ql >> 6) & 3);
            sum2 += y[l+0] * d[4] + y[l+16] * d[5] + y[l+32] * d[6] + y[l+48] * d[7];
        }
        tmp += dall.x * sum1 - dall.y * sum2;
    }
#endif

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = tmp;
    }
}

extern "C" __global__ void dequantize_mul_mat_vec_q3_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q3_K * x = (const block_q3_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

#if QK_K == 256

    const uint16_t kmask1 = 0x0303;
    const uint16_t kmask2 = 0x0f0f;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int n  = K_QUANTS_PER_ITERATION;               // iterations in the inner loop
    const int step = 16/K_QUANTS_PER_ITERATION;
    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0....15 or 0...7

    const uint8_t m = 1 << (4*im);

    const int l0 = n*in;                                 // 0...15 or 0...14 in steps of 2
    const int q_offset =  32*im + l0;
    const int y_offset = 128*im + l0;

    uint16_t utmp[4];
    const int8_t * s = (const int8_t *)utmp;

    const uint16_t s_shift = 4*im;

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y  = yy + i * QK_K + y_offset;
        const uint8_t * q = x[i].qs + q_offset;
        const uint8_t * h = x[i].hmask + l0;

        const uint16_t * a = (const uint16_t *)x[i].scales;
        utmp[0] = ((a[0] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 0)) & kmask1) << 4);
        utmp[1] = ((a[1] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 0)) & kmask1) << 4);
        utmp[2] = ((a[2] >> s_shift) & kmask2) | (((a[4] >> (s_shift + 2)) & kmask1) << 4);
        utmp[3] = ((a[3] >> s_shift) & kmask2) | (((a[5] >> (s_shift + 2)) & kmask1) << 4);

        const float d = x[i].d;

        float sum = 0;
        for (int l = 0; l < n; ++l) {
            sum += y[l+ 0] * (s[0] - 32) * (((q[l] >> 0) & 3) - (h[l] & (m << 0) ? 0 : 4))
                 + y[l+32] * (s[2] - 32) * (((q[l] >> 2) & 3) - (h[l] & (m << 1) ? 0 : 4))
                 + y[l+64] * (s[4] - 32) * (((q[l] >> 4) & 3) - (h[l] & (m << 2) ? 0 : 4))
                 + y[l+96] * (s[6] - 32) * (((q[l] >> 6) & 3) - (h[l] & (m << 3) ? 0 : 4));
            sum += y[l+16] * (s[1] - 32) * (((q[l+16] >> 0) & 3) - (h[l+16] & (m << 0) ? 0 : 4))
                 + y[l+48] * (s[3] - 32) * (((q[l+16] >> 2) & 3) - (h[l+16] & (m << 1) ? 0 : 4))
                 + y[l+80] * (s[5] - 32) * (((q[l+16] >> 4) & 3) - (h[l+16] & (m << 2) ? 0 : 4))
                + y[l+112] * (s[7] - 32) * (((q[l+16] >> 6) & 3) - (h[l+16] & (m << 3) ? 0 : 4));
        }
        tmp += d * sum;

    }
#else

    const int tid = threadIdx.x/(2*K_QUANTS_PER_ITERATION);  // 0...15 or 0...7
    const int ix  = threadIdx.x%(2*K_QUANTS_PER_ITERATION);  // 0....1 or 0...3
    const int offset = tid * K_QUANTS_PER_ITERATION;         // 0...15 or 0...14
    const int in = offset/8;                                 // 0 or 1
    const int im = offset%8;                                 // 0...7

    for (int i = ix; i < num_blocks_per_row; i += 2*K_QUANTS_PER_ITERATION) {

        const float   * y = yy + i * QK_K + offset;
        const uint8_t * q = x[i].qs + offset;
        const uint8_t * s = x[i].scales;

        const float dall = (float)x[i].d;

        float sum = 0;
        for (int l = 0; l < K_QUANTS_PER_ITERATION; ++l) {
            const uint8_t hl = x[i].hmask[im+l] >> in;
            const uint8_t ql = q[l];
            sum += y[l+ 0] * dall * ((s[0] & 0xF) - 8) * ((int8_t)((ql >> 0) & 3) - ((hl >> 0) & 1 ? 0 : 4))
                 + y[l+16] * dall * ((s[0] >>  4) - 8) * ((int8_t)((ql >> 2) & 3) - ((hl >> 2) & 1 ? 0 : 4))
                 + y[l+32] * dall * ((s[1] & 0xF) - 8) * ((int8_t)((ql >> 4) & 3) - ((hl >> 4) & 1 ? 0 : 4))
                 + y[l+48] * dall * ((s[1] >>  4) - 8) * ((int8_t)((ql >> 6) & 3) - ((hl >> 6) & 1 ? 0 : 4));
        }
        tmp += sum;
    }
#endif

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = tmp;
    }
}

extern "C" __global__ void dequantize_mul_mat_vec_q4_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q4_K * x = (const block_q4_K *)vx + ib0;

#if QK_K == 256
    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0,1

    const int step = 8/K_QUANTS_PER_ITERATION;           // 8 or 4

    const int il  = tid/step;                            // 0...3
    const int ir  = tid - step*il;                       // 0...7 or 0...3
    const int n   = 2 * K_QUANTS_PER_ITERATION;          // 2 or 4

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

#if K_QUANTS_PER_ITERATION == 2
    uint32_t q32[4];
    const uint8_t * q4 = (const uint8_t *)q32;
#else
    uint16_t q16[4];
    const uint8_t * q4 = (const uint8_t *)q16;
#endif

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y1 = yy + i*QK_K + y_offset;
        const float   * y2 = y1 + 128;

        const float dall = __low2half(x[i].dm);
        const float dmin = __high2half(x[i].dm);

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

#if K_QUANTS_PER_ITERATION == 2
        const uint32_t * q1 = (const uint32_t *)(x[i].qs + q_offset);
        const uint32_t * q2 = q1 + 16;

        q32[0] = q1[0] & 0x0f0f0f0f;
        q32[1] = q1[0] & 0xf0f0f0f0;
        q32[2] = q2[0] & 0x0f0f0f0f;
        q32[3] = q2[0] & 0xf0f0f0f0;

        float4 s = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < 4; ++l) {
            s.x += y1[l] * q4[l+0]; s.y += y1[l+32] * q4[l+ 4];
            s.z += y2[l] * q4[l+8]; s.w += y2[l+32] * q4[l+12];
            smin += y1[l] * sc[2] + y1[l+32] * sc[3] + y2[l] * sc[6] + y2[l+32] * sc[7];
        }
        tmp += dall * (s.x * sc[0] + s.y * sc[1] * 1.f/16.f + s.z * sc[4] + s.w * sc[5] * 1.f/16.f) - dmin * smin;
#else
        const uint16_t * q1 = (const uint16_t *)(x[i].qs + q_offset);
        const uint16_t * q2 = q1 + 32;

        q16[0] = q1[0] & 0x0f0f;
        q16[1] = q1[0] & 0xf0f0;
        q16[2] = q2[0] & 0x0f0f;
        q16[3] = q2[0] & 0xf0f0;

        float4 s = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        for (int l = 0; l < 2; ++l) {
            s.x += y1[l] * q4[l+0]; s.y += y1[l+32] * q4[l+2];
            s.z += y2[l] * q4[l+4]; s.w += y2[l+32] * q4[l+6];
            smin += y1[l] * sc[2] + y1[l+32] * sc[3] + y2[l] * sc[6] + y2[l+32] * sc[7];
        }
        tmp += dall * (s.x * sc[0] + s.y * sc[1] * 1.f/16.f + s.z * sc[4] + s.w * sc[5] * 1.f/16.f) - dmin * smin;
#endif

    }
#else
    const int tid = threadIdx.x/(2*K_QUANTS_PER_ITERATION);  // 0...15
    const int ix  = threadIdx.x%(2*K_QUANTS_PER_ITERATION);

    const int step = tid * K_QUANTS_PER_ITERATION;

    uint16_t aux16[2];
    const uint8_t * s = (const uint8_t *)aux16;

    float tmp = 0;

    for (int i = ix; i < num_blocks_per_row; i += 2*K_QUANTS_PER_ITERATION) {
        const uint8_t * q = x[i].qs + step;
        const float   * y = yy + i*QK_K + step;
        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux16[0] = a[0] & 0x0f0f;
        aux16[1] = (a[0] >> 4) & 0x0f0f;
        const float d = (float)x[i].dm[0];
        const float m = (float)x[i].dm[1];
        float sum = 0.f;
        for (int j = 0; j < K_QUANTS_PER_ITERATION; ++j) {
            sum += y[j+ 0] * (d * s[0] * (q[j+ 0] & 0xF) - m * s[2])
                 + y[j+16] * (d * s[0] * (q[j+16] & 0xF) - m * s[2])
                 + y[j+32] * (d * s[1] * (q[j+ 0] >>  4) - m * s[3])
                 + y[j+48] * (d * s[1] * (q[j+16] >>  4) - m * s[3]);
        }
        tmp += sum;
    }

#endif

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

extern "C" __global__ void dequantize_mul_mat_vec_q5_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols) {

    const int row = blockIdx.x;
    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q5_K * x = (const block_q5_K *)vx + ib0;

    float tmp = 0; // partial sum for thread in warp

#if QK_K == 256
    const uint16_t kmask1 = 0x3f3f;
    const uint16_t kmask2 = 0x0f0f;
    const uint16_t kmask3 = 0xc0c0;

    const int tid = threadIdx.x/2;  // 0...15
    const int ix  = threadIdx.x%2;

    const int il  = tid/4;     // 0...3
    const int ir  = tid - 4*il;// 0...3
    const int n   = 2;

    const int im = il/2;  // 0 or 1. 0 computes 0,32 + 128,160, 1 computes 64,96 + 192,224
    const int in = il%2;

    const int l0 = n*(2*ir + in);
    const int q_offset = 32*im + l0;
    const int y_offset = 64*im + l0;

    const uint8_t hm1  = 1 << (2*im);
    const uint8_t hm2  = hm1 << 4;

    uint16_t aux[4];
    const uint8_t * sc = (const uint8_t *)aux;

    uint16_t q16[8];
    const uint8_t * q4 = (const uint8_t *)q16;

    for (int i = ix; i < num_blocks_per_row; i += 2) {

        const uint8_t * ql1 = x[i].qs + q_offset;
        const uint8_t * qh  = x[i].qh + l0;
        const float   * y1  = yy + i*QK_K + y_offset;
        const float   * y2  = y1 + 128;

        const float dall = __low2half(x[i].dm);
        const float dmin = __high2half(x[i].dm);

        const uint16_t * a = (const uint16_t *)x[i].scales;
        aux[0] = a[im+0] & kmask1;
        aux[1] = a[im+2] & kmask1;
        aux[2] = ((a[im+4] >> 0) & kmask2) | ((a[im+0] & kmask3) >> 2);
        aux[3] = ((a[im+4] >> 4) & kmask2) | ((a[im+2] & kmask3) >> 2);

        float4 sum = {0.f, 0.f, 0.f, 0.f};
        float smin = 0;
        const uint16_t * q1 = (const uint16_t *)ql1;
        const uint16_t * q2 = q1 + 32;
        q16[0] = q1[0] & 0x0f0f;
        q16[1] = q1[8] & 0x0f0f;
        q16[2] = (q1[0] >> 4) & 0x0f0f;
        q16[3] = (q1[8] >> 4) & 0x0f0f;
        q16[4] = q2[0] & 0x0f0f;
        q16[5] = q2[8] & 0x0f0f;
        q16[6] = (q2[0] >> 4) & 0x0f0f;
        q16[7] = (q2[8] >> 4) & 0x0f0f;
        for (int l = 0; l < n; ++l) {
            sum.x += y1[l+ 0] * (q4[l +0] + (qh[l+ 0] & (hm1 << 0) ? 16 : 0))
                   + y1[l+16] * (q4[l +2] + (qh[l+16] & (hm1 << 0) ? 16 : 0));
            sum.y += y1[l+32] * (q4[l +4] + (qh[l+ 0] & (hm1 << 1) ? 16 : 0))
                   + y1[l+48] * (q4[l +6] + (qh[l+16] & (hm1 << 1) ? 16 : 0));
            sum.z += y2[l+ 0] * (q4[l +8] + (qh[l+ 0] & (hm2 << 0) ? 16 : 0))
                   + y2[l+16] * (q4[l+10] + (qh[l+16] & (hm2 << 0) ? 16 : 0));
            sum.w += y2[l+32] * (q4[l+12] + (qh[l+ 0] & (hm2 << 1) ? 16 : 0))
                   + y2[l+48] * (q4[l+14] + (qh[l+16] & (hm2 << 1) ? 16 : 0));
            smin += (y1[l] + y1[l+16]) * sc[2] + (y1[l+32] + y1[l+48]) * sc[3]
                  + (y2[l] + y2[l+16]) * sc[6] + (y2[l+32] + y2[l+48]) * sc[7];
        }
        tmp += dall * (sum.x * sc[0] + sum.y * sc[1] + sum.z * sc[4] + sum.w * sc[5]) - dmin * smin;
    }

#else
    const int tid = threadIdx.x/(2*K_QUANTS_PER_ITERATION);  // 0...15
    const int ix  = threadIdx.x%(2*K_QUANTS_PER_ITERATION);
    const int step = tid * K_QUANTS_PER_ITERATION;
    const int im = step/8;
    const int in = step%8;

    for (int i = ix; i < num_blocks_per_row; i += 2*K_QUANTS_PER_ITERATION) {
        const uint8_t * q = x[i].qs + step;
        const int8_t  * s = x[i].scales;
        const float   * y = yy + i*QK_K + step;
        const float     d = x[i].d;
        float sum = 0.f;
        for (int j = 0; j < K_QUANTS_PER_ITERATION; ++j) {
            const uint8_t h = x[i].qh[in+j] >> im;
            sum += y[j+ 0] * d * s[0] * ((q[j+ 0] & 0xF) - ((h >> 0) & 1 ? 0 : 16))
                 + y[j+16] * d * s[1] * ((q[j+16] & 0xF) - ((h >> 2) & 1 ? 0 : 16))
                 + y[j+32] * d * s[2] * ((q[j+ 0] >>  4) - ((h >> 4) & 1 ? 0 : 16))
                 + y[j+48] * d * s[3] * ((q[j+16] >>  4) - ((h >> 6) & 1 ? 0 : 16));
        }
        tmp += sum;
    }
#endif

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (threadIdx.x == 0) {
        dst[row] = tmp;
    }
}

extern "C" __global__ void dequantize_mul_mat_vec_q6_k(const void * __restrict__ vx, const float * __restrict__ yy, float * __restrict__ dst, const int ncols, int nrows) {

    static_assert(16%K_QUANTS_PER_ITERATION == 0, "16 must be divisible by K_QUANTS_PER_ITERATION");

    const int row = blockIdx.x*blockDim.y + threadIdx.y;
    if (row > nrows) return;

    const int num_blocks_per_row = ncols / QK_K;
    const int ib0 = row*num_blocks_per_row;

    const block_q6_K * x = (const block_q6_K *)vx + ib0;

#if QK_K == 256

    const int tid = threadIdx.x/K_QUANTS_PER_ITERATION;  // 0...31 or 0...16
    const int ix  = threadIdx.x%K_QUANTS_PER_ITERATION;  // 0 or 0, 1

    const int step = 16/K_QUANTS_PER_ITERATION;          // 16 or 8

    const int im = tid/step;                             // 0 or 1. 0 computes 0..., 1 computes 128...
    const int in = tid - step*im;                        // 0...15 or 0...7

#if K_QUANTS_PER_ITERATION == 1
    const int l0 = K_QUANTS_PER_ITERATION*in;            // 0...15
    const int is = 0;
#else
    const int l0 = 4 * in;                               // 0, 4, 8, ..., 28
    const int is = in / 4;
#endif
    const int ql_offset = 64*im + l0;
    const int qh_offset = 32*im + l0;
    const int s_offset  =  8*im + is;
    const int y_offset = 128*im + l0;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += K_QUANTS_PER_ITERATION) {

        const float   * y  = yy + i * QK_K + y_offset;
        const uint8_t * ql = x[i].ql + ql_offset;
        const uint8_t * qh = x[i].qh + qh_offset;
        const int8_t  * s  = x[i].scales + s_offset;

        const float d = x[i].d;

#if K_QUANTS_PER_ITERATION == 1
        float sum = y[ 0] * s[0] * d * ((int8_t)((ql[ 0] & 0xF) | ((qh[ 0] & 0x03) << 4)) - 32)
                  + y[16] * s[1] * d * ((int8_t)((ql[16] & 0xF) | ((qh[16] & 0x03) << 4)) - 32)
                  + y[32] * s[2] * d * ((int8_t)((ql[32] & 0xF) | ((qh[ 0] & 0x0c) << 2)) - 32)
                  + y[48] * s[3] * d * ((int8_t)((ql[48] & 0xF) | ((qh[16] & 0x0c) << 2)) - 32)
                  + y[64] * s[4] * d * ((int8_t)((ql[ 0]  >> 4) | ((qh[ 0] & 0x30) >> 0)) - 32)
                  + y[80] * s[5] * d * ((int8_t)((ql[16]  >> 4) | ((qh[16] & 0x30) >> 0)) - 32)
                  + y[96] * s[6] * d * ((int8_t)((ql[32]  >> 4) | ((qh[ 0] & 0xc0) >> 2)) - 32)
                  +y[112] * s[7] * d * ((int8_t)((ql[48]  >> 4) | ((qh[16] & 0xc0) >> 2)) - 32);
        tmp += sum;
#else
        float sum = 0;
        for (int l = 0; l < 4; ++l) {
            sum += y[l+ 0] * s[0] * d * ((int8_t)((ql[l+ 0] & 0xF) | (((qh[l] >> 0) & 3) << 4)) - 32)
                 + y[l+32] * s[2] * d * ((int8_t)((ql[l+32] & 0xF) | (((qh[l] >> 2) & 3) << 4)) - 32)
                 + y[l+64] * s[4] * d * ((int8_t)((ql[l+ 0]  >> 4) | (((qh[l] >> 4) & 3) << 4)) - 32)
                 + y[l+96] * s[6] * d * ((int8_t)((ql[l+32]  >> 4) | (((qh[l] >> 6) & 3) << 4)) - 32);
        }
        tmp += sum;
#endif

    }

#else

    const int tid = threadIdx.x/(2*K_QUANTS_PER_ITERATION);  // 0...7
    const int ix  = threadIdx.x%(2*K_QUANTS_PER_ITERATION);  // 0...3

    const int step = tid * K_QUANTS_PER_ITERATION;

    float tmp = 0; // partial sum for thread in warp

    for (int i = ix; i < num_blocks_per_row; i += 2*K_QUANTS_PER_ITERATION) {

        const float   * y  = yy + i * QK_K + step;
        const uint8_t * ql = x[i].ql + step;
        const uint8_t * qh = x[i].qh + step;
        const int8_t  * s  = x[i].scales;

        const float d = x[i+0].d;

        float sum = 0;
        for (int j = 0; j < K_QUANTS_PER_ITERATION; ++j) {
            sum += y[j+ 0] * s[0] * d * ((int8_t)((ql[j+ 0] & 0xF) | ((qh[j] & 0x03) << 4)) - 32)
                 + y[j+16] * s[1] * d * ((int8_t)((ql[j+16] & 0xF) | ((qh[j] & 0x0c) << 2)) - 32)
                 + y[j+32] * s[2] * d * ((int8_t)((ql[j+ 0] >>  4) | ((qh[j] & 0x30) >> 0)) - 32)
                 + y[j+48] * s[3] * d * ((int8_t)((ql[j+16] >>  4) | ((qh[j] & 0xc0) >> 2)) - 32);
        }
        tmp += sum;

    }

#endif

    // sum up partial sums and write back result
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        tmp += __shfl_xor_sync(0xffffffff, tmp, mask, 32);
    }

    if (tid == 0) {
        dst[row] = tmp;
    }
}

// VDR = vec dot ratio, how many contiguous integers each thread processes when the vec dot kernel is called
// MMVQ = mul_mat_vec_q, MMQ = mul_mat_q

#define VDR_Q4_0_Q8_1_MMVQ 2
#define VDR_Q4_0_Q8_1_MMQ  4

template <int vdr> static __device__ __forceinline__ float vec_dot_q4_0_q8_1_impl(
    const int * v, const int * u, const float & d4, const half2 & ds8) {

    int sumi = 0;

#pragma unroll
    for (int i = 0; i < vdr; ++i) {
        const int vi0 = (v[i] >> 0) & 0x0F0F0F0F;
        const int vi1 = (v[i] >> 4) & 0x0F0F0F0F;

        // SIMD dot product of quantized values
        sumi = ggml_cuda_dp4a(vi0, u[2*i+0], sumi);
        sumi = ggml_cuda_dp4a(vi1, u[2*i+1], sumi);
    }

    const float2 ds8f = __half22float2(ds8);

    // second part effectively subtracts 8 from each quant value
    return d4 * (sumi * ds8f.x - (8*vdr/QI4_0) * ds8f.y);
}

#define VDR_Q4_1_Q8_1_MMVQ 2
#define VDR_Q4_1_Q8_1_MMQ  4

template <int vdr> static __device__ __forceinline__ float vec_dot_q4_1_q8_1_impl(
    const int * v, const int * u, const half2 & dm4, const half2 & ds8) {
    int sumi = 0;

#pragma unroll
    for (int i = 0; i < vdr; ++i) {
        const int vi0 = (v[i] >> 0) & 0x0F0F0F0F;
        const int vi1 = (v[i] >> 4) & 0x0F0F0F0F;

        // SIMD dot product of quantized values
        sumi = ggml_cuda_dp4a(vi0, u[2*i+0], sumi);
        sumi = ggml_cuda_dp4a(vi1, u[2*i+1], sumi);
    }

#ifdef GGML_CUDA_F16
    const float2 tmp = __half22float2(__hmul2(dm4, ds8));
    const float d4d8 = tmp.x;
    const float m4s8 = tmp.y;
#else
    const float2 dm4f = __half22float2(dm4);
    const float2 ds8f = __half22float2(ds8);
    const float d4d8 = dm4f.x * ds8f.x;
    const float m4s8 = dm4f.y * ds8f.y;
#endif // GGML_CUDA_F16

    // scale second part of sum by QI8_1/(vdr * QR4_1) to compensate for multiple threads adding it
    return sumi * d4d8 + m4s8 / (QI8_1 / (vdr * QR4_1));
}

#define VDR_Q5_0_Q8_1_MMVQ 2
#define VDR_Q5_0_Q8_1_MMQ  4

template <int vdr> static __device__ __forceinline__ float vec_dot_q5_0_q8_1_impl(
    const int * vl, const int * vh, const int * u, const float & d5, const half2 & ds8) {

    int sumi = 0;

#pragma unroll
    for (int i = 0; i < vdr; ++i) {
        int vi0 = (vl[i] >>  0) & 0x0F0F0F0F; // lower 4 qs bits, still need qh as 5th bits
        vi0    |= (vh[i] <<  4) & 0x00000010; // 0 ->  4
        vi0    |= (vh[i] << 11) & 0x00001000; // 1 -> 12
        vi0    |= (vh[i] << 18) & 0x00100000; // 2 -> 20
        vi0    |= (vh[i] << 25) & 0x10000000; // 3 -> 28
        sumi = ggml_cuda_dp4a(vi0, u[2*i+0], sumi); // SIMD dot product of quantized values

        int vi1 = (vl[i] >>  4) & 0x0F0F0F0F; // upper 4 qs bits, still need qh as 5th bits
        vi1    |= (vh[i] >> 12) & 0x00000010; // 16 ->  4
        vi1    |= (vh[i] >>  5) & 0x00001000; // 17 -> 12
        vi1    |= (vh[i] <<  2) & 0x00100000; // 18 -> 20
        vi1    |= (vh[i] <<  9) & 0x10000000; // 19 -> 28
        sumi = ggml_cuda_dp4a(vi1, u[2*i+1], sumi); // SIMD dot product of quantized values
    }

    const float2 ds8f = __half22float2(ds8);

    // second part effectively subtracts 16 from each quant value
    return d5 * (sumi * ds8f.x - (16*vdr/QI5_0) * ds8f.y);
}

#define VDR_Q5_1_Q8_1_MMVQ 2
#define VDR_Q5_1_Q8_1_MMQ  4

template <int vdr> static __device__ __forceinline__ float vec_dot_q5_1_q8_1_impl(
    const int * vl, const int * vh, const int * u, const half2 & dm5, const half2 & ds8) {

    int sumi = 0;

#pragma unroll
    for (int i = 0; i < vdr; ++i) {
        int vi0 = (vl[i] >>  0) & 0x0F0F0F0F; // lower 4 qs bits, still need qh as 5th bits
        vi0    |= (vh[i] <<  4) & 0x00000010; // 0 ->  4
        vi0    |= (vh[i] << 11) & 0x00001000; // 1 -> 12
        vi0    |= (vh[i] << 18) & 0x00100000; // 2 -> 20
        vi0    |= (vh[i] << 25) & 0x10000000; // 3 -> 28
        sumi = ggml_cuda_dp4a(vi0, u[2*i+0], sumi); // SIMD dot product of quantized values

        int vi1 = (vl[i] >>  4) & 0x0F0F0F0F; // upper 4 qs bits, still need qh as 5th bits
        vi1    |= (vh[i] >> 12) & 0x00000010; // 16 ->  4
        vi1    |= (vh[i] >>  5) & 0x00001000; // 17 -> 12
        vi1    |= (vh[i] <<  2) & 0x00100000; // 18 -> 20
        vi1    |= (vh[i] <<  9) & 0x10000000; // 19 -> 28
        sumi = ggml_cuda_dp4a(vi1, u[2*i+1], sumi); // SIMD dot product of quantized values
    }

#ifdef GGML_CUDA_F16
    const float2 tmp = __half22float2(__hmul2(dm5, ds8));
    const float d5d8 = tmp.x;
    const float m5s8 = tmp.y;
#else
    const float2 dm5f = __half22float2(dm5);
    const float2 ds8f = __half22float2(ds8);
    const float d5d8 = dm5f.x * ds8f.x;
    const float m5s8 = dm5f.y * ds8f.y;
#endif // GGML_CUDA_F16

    // scale second part of sum by QI5_1 / vdr to compensate for multiple threads adding it
    return sumi*d5d8 + m5s8 / (QI5_1 / vdr);
}

#define VDR_Q8_0_Q8_1_MMVQ 2
#define VDR_Q8_0_Q8_1_MMQ 8

template <int vdr> static __device__ __forceinline__ float vec_dot_q8_0_q8_1_impl(
    const int * v, const int * u, const float & d8_0, const float & d8_1) {

    int sumi = 0;

#pragma unroll
    for (int i = 0; i < vdr; ++i) {
        // SIMD dot product of quantized values
        sumi = ggml_cuda_dp4a(v[i], u[i], sumi);
    }

    return d8_0*d8_1 * sumi;
}

template <int vdr> static __device__ __forceinline__ float vec_dot_q8_1_q8_1_impl(
    const int * v, const int * u, const half2 & dm8, const half2 & ds8) {

    int sumi = 0;

#pragma unroll
    for (int i = 0; i < vdr; ++i) {
        // SIMD dot product of quantized values
        sumi = ggml_cuda_dp4a(v[i], u[i], sumi);
    }

#ifdef GGML_CUDA_F16
    const float2 tmp = __half22float2(__hmul2(dm8, ds8));
    const float d8d8 = tmp.x;
    const float m8s8 = tmp.y;
#else
    const float2 dm8f = __half22float2(dm8);
    const float2 ds8f = __half22float2(ds8);
    const float d8d8 = dm8f.x * ds8f.x;
    const float m8s8 = dm8f.y * ds8f.y;
#endif // GGML_CUDA_F16

    // scale second part of sum by QI8_1/ vdr to compensate for multiple threads adding it
    return sumi*d8d8 + m8s8 / (QI8_1 / vdr);
}

#define VDR_Q2_K_Q8_1_MMVQ 1
#define VDR_Q2_K_Q8_1_MMQ  2

// contiguous v/x values
static __device__ __forceinline__ float vec_dot_q2_K_q8_1_impl_mmvq(
    const int & v, const int * __restrict__ u, const uint8_t * __restrict__ scales,
    const half2 & dm2, const float * __restrict__ d8) {

    float sumf_d = 0.0f;
    float sumf_m = 0.0f;

#pragma unroll
    for (int i = 0; i < QR2_K; ++i) {
        const int sc = scales[2*i];

        const int vi = (v >> (2*i)) & 0x03030303;

        sumf_d += d8[i] * (ggml_cuda_dp4a(vi, u[i], 0) * (sc & 0xF)); // SIMD dot product

        // fill int with 4x m
        int m = sc >> 4;
        m |= m <<  8;
        m |= m << 16;
        sumf_m += d8[i] * ggml_cuda_dp4a(m, u[i], 0); // multiply constant q2_K part with sum of q8_1 values
    }

    const float2 dm2f = __half22float2(dm2);

    return dm2f.x*sumf_d - dm2f.y*sumf_m;
}

// contiguous u/y values
static __device__ __forceinline__ float vec_dot_q2_K_q8_1_impl_mmq(
    const int * __restrict__ v, const int * __restrict__ u, const uint8_t * __restrict__ scales,
    const half2 & dm2, const float & d8) {

    int sumi_d = 0;
    int sumi_m = 0;

#pragma unroll
    for (int i0 = 0; i0 < QI8_1; i0 += QI8_1/2) {
        int sumi_d_sc = 0;

        const int sc = scales[i0 / (QI8_1/2)];

        // fill int with 4x m
        int m = sc >> 4;
        m |= m <<  8;
        m |= m << 16;

#pragma unroll
        for (int i = i0; i < i0 + QI8_1/2; ++i) {
            sumi_d_sc = ggml_cuda_dp4a(v[i], u[i], sumi_d_sc); // SIMD dot product
            sumi_m    = ggml_cuda_dp4a(m,    u[i], sumi_m); // multiply sum of q8_1 values with m
        }

        sumi_d += sumi_d_sc * (sc & 0xF);
    }

    const float2 dm2f = __half22float2(dm2);

    return d8 * (dm2f.x*sumi_d - dm2f.y*sumi_m);
}

#define VDR_Q3_K_Q8_1_MMVQ 1
#define VDR_Q3_K_Q8_1_MMQ  2

// contiguous v/x values
static __device__ __forceinline__ float vec_dot_q3_K_q8_1_impl_mmvq(
    const int & vl, const int & vh, const int * __restrict__ u, const uint8_t * __restrict__ scales,
    const int & scale_offset, const float & d3, const float * __restrict__ d8) {

    float sumf = 0.0f;

#pragma unroll
    for (int i = 0; i < QR3_K; ++i) {
        const int isc = scale_offset + 2*i;

        const int isc_low = isc % (QK_K/32);
        const int sc_shift_low = 4 * (isc / (QK_K/32));
        const int sc_low  = (scales[isc_low] >> sc_shift_low) & 0xF;

        const int isc_high = isc % (QK_K/64);
        const int sc_shift_high = 2 * (isc / (QK_K/64));
        const int sc_high = ((scales[(QK_K/32) + isc_high] >> sc_shift_high) & 3) << 4;

        const int sc = (sc_low | sc_high) - 32;

        const int vil = (vl >> (2*i)) & 0x03030303;

        const int vih = ((vh >> i) << 2) & 0x04040404;

        const int vi = __vsubss4(vil, vih);

        sumf += d8[i] * (ggml_cuda_dp4a(vi, u[i], 0) * sc); // SIMD dot product
    }

    return d3 * sumf;
}

// contiguous u/y values
static __device__ __forceinline__ float vec_dot_q3_K_q8_1_impl_mmq(
    const int * __restrict__ v, const int * __restrict__ u, const int8_t * __restrict__ scales,
    const float & d3, const float & d8) {

    int sumi = 0;

#pragma unroll
    for (int i0 = 0; i0 < QR3_K*VDR_Q3_K_Q8_1_MMQ; i0 += QI8_1/2) {
        int sumi_sc = 0;

        for (int i = i0; i < i0 + QI8_1/2; ++i) {
            sumi_sc = ggml_cuda_dp4a(v[i], u[i], sumi_sc); // SIMD dot product
        }

        sumi += sumi_sc * scales[i0 / (QI8_1/2)];
    }

    return d3*d8 * sumi;
}

#define VDR_Q4_K_Q8_1_MMVQ 2
#define VDR_Q4_K_Q8_1_MMQ  8

// contiguous v/x values
static __device__ __forceinline__ float vec_dot_q4_K_q8_1_impl_vmmq(
    const int * __restrict__ v, const int * __restrict__ u, const uint8_t * __restrict__ sc,
    const uint8_t * __restrict__ m, const half2 & dm4, const float * __restrict__ d8) {

    float sumf_d = 0.0f;
    float sumf_m = 0.0f;

#pragma unroll
    for (int i = 0; i < QR4_K; ++i) {
        const int v0i = (v[0] >> (4*i)) & 0x0F0F0F0F;
        const int v1i = (v[1] >> (4*i)) & 0x0F0F0F0F;

        const int dot1 = ggml_cuda_dp4a(v1i, u[2*i+1], ggml_cuda_dp4a(v0i, u[2*i+0], 0)); // SIMD dot product
        const int dot2 = ggml_cuda_dp4a(0x01010101, u[2*i+1], ggml_cuda_dp4a(0x01010101, u[2*i+0], 0)); // sum of u

        sumf_d += d8[i] * (dot1 * sc[i]);
        sumf_m += d8[i] * (dot2 * m[i]);  // multiply constant part of q4_K with sum of q8_1 values
    }

    const float2 dm4f = __half22float2(dm4);

    return dm4f.x*sumf_d - dm4f.y*sumf_m;
}

// contiguous u/y values
static __device__ __forceinline__ float vec_dot_q4_K_q8_1_impl_mmq(
    const int * __restrict__ v, const int * __restrict__ u, const uint8_t * __restrict__ sc,
    const uint8_t * __restrict__ m, const half2 & dm4, const half2 * __restrict__ ds8) {

    float sumf_d = 0.0f;
    float sumf_m = 0.0f;

#pragma unroll
    for (int i = 0; i < QR4_K*VDR_Q4_K_Q8_1_MMQ/QI8_1; ++i) {
        int sumi_d = 0;

#pragma unroll
        for (int j = 0; j < QI8_1; ++j) {
            sumi_d = ggml_cuda_dp4a((v[j] >> (4*i)) & 0x0F0F0F0F, u[i*QI8_1 + j], sumi_d); // SIMD dot product
        }

        const float2 ds8f = __half22float2(ds8[i]);

        sumf_d += ds8f.x * (sc[i] * sumi_d);
        sumf_m += ds8f.y *   m[i]; // sum of q8_1 block * q4_K min val
    }

    const float2 dm4f = __half22float2(dm4);

    return dm4f.x*sumf_d - dm4f.y*sumf_m;
}

#define VDR_Q5_K_Q8_1_MMVQ 2
#define VDR_Q5_K_Q8_1_MMQ  8

// contiguous v/x values
static __device__ __forceinline__ float vec_dot_q5_K_q8_1_impl_vmmq(
    const int * __restrict__ vl, const int * __restrict__ vh, const int * __restrict__ u, const uint8_t * __restrict__ sc,
    const uint8_t * __restrict__ m, const half2 & dm5, const float * __restrict__ d8) {

    float sumf_d = 0.0f;
    float sumf_m = 0.0f;

#pragma unroll
    for (int i = 0; i < QR5_K; ++i) {
        const int vl0i = (vl[0] >> (4*i)) & 0x0F0F0F0F;
        const int vl1i = (vl[1] >> (4*i)) & 0x0F0F0F0F;

        const int vh0i = ((vh[0] >> i) << 4) & 0x10101010;
        const int vh1i = ((vh[1] >> i) << 4) & 0x10101010;

        const int v0i = vl0i | vh0i;
        const int v1i = vl1i | vh1i;

        const int dot1 = ggml_cuda_dp4a(v0i, u[2*i+0], ggml_cuda_dp4a(v1i, u[2*i+1], 0)); // SIMD dot product
        const int dot2 = ggml_cuda_dp4a(0x01010101, u[2*i+0], ggml_cuda_dp4a(0x01010101, u[2*i+1], 0)); // sum of u

        sumf_d += d8[i] * (dot1 * sc[i]);
        sumf_m += d8[i] * (dot2 * m[i]);

    }

    const float2 dm5f = __half22float2(dm5);

    return dm5f.x*sumf_d - dm5f.y*sumf_m;
}

// contiguous u/y values
static __device__ __forceinline__ float vec_dot_q5_K_q8_1_impl_mmq(
    const int * __restrict__ v, const int * __restrict__ u, const uint8_t * __restrict__ sc,
    const uint8_t * __restrict__ m, const half2 & dm4, const half2 * __restrict__ ds8) {

    float sumf_d = 0.0f;
    float sumf_m = 0.0f;

#pragma unroll
    for (int i = 0; i < QR5_K*VDR_Q5_K_Q8_1_MMQ/QI8_1; ++i) {
        int sumi_d = 0;

#pragma unroll
        for (int j = 0; j < QI8_1; ++j) {
            sumi_d = ggml_cuda_dp4a(v[i*QI8_1 + j], u[i*QI8_1 + j], sumi_d); // SIMD dot product
        }

        const float2 ds8f = __half22float2(ds8[i]);

        sumf_d += ds8f.x * (sc[i] * sumi_d);
        sumf_m += ds8f.y *   m[i]; // sum of q8_1 block * q4_K min val
    }

    const float2 dm4f = __half22float2(dm4);

    return dm4f.x*sumf_d - dm4f.y*sumf_m;
}

#define VDR_Q6_K_Q8_1_MMVQ 1
#define VDR_Q6_K_Q8_1_MMQ  8

// contiguous v/x values
static __device__ __forceinline__ float vec_dot_q6_K_q8_1_impl_mmvq(
    const int & vl, const int & vh, const int * __restrict__ u, const int8_t * __restrict__ scales,
    const float & d, const float * __restrict__ d8) {

    float sumf = 0.0f;

#pragma unroll
    for (int i = 0; i < QR6_K; ++i) {
        const int sc = scales[4*i];

        const int vil = (vl >> (4*i)) & 0x0F0F0F0F;

        const int vih = ((vh >> (4*i)) << 4) & 0x30303030;

        const int vi = __vsubss4((vil | vih), 0x20202020); // vi = (vil | vih) - 32

        sumf += d8[i] * (ggml_cuda_dp4a(vi, u[i], 0) * sc); // SIMD dot product
    }

    return d*sumf;
}

// contiguous u/y values
static __device__ __forceinline__ float vec_dot_q6_K_q8_1_impl_mmq(
    const int * __restrict__ v, const int * __restrict__ u, const int8_t * __restrict__ sc,
    const float & d6, const float * __restrict__ d8) {

    float sumf_d = 0.0f;

#pragma unroll
    for (int i0 = 0; i0 < VDR_Q6_K_Q8_1_MMQ; i0 += 4) {
        int2 sumi_d = {0, 0}; // 2 q6_K scales per q8_1 scale

#pragma unroll
        for (int i = i0; i < i0 + 2; ++i) {
            sumi_d.x = ggml_cuda_dp4a(v[2*i+0], u[2*i+0], sumi_d.x); // SIMD dot product
            sumi_d.x = ggml_cuda_dp4a(v[2*i+1], u[2*i+1], sumi_d.x); // SIMD dot product

            sumi_d.y = ggml_cuda_dp4a(v[2*i+4], u[2*i+4], sumi_d.y); // SIMD dot product
            sumi_d.y = ggml_cuda_dp4a(v[2*i+5], u[2*i+5], sumi_d.y); // SIMD dot product
        }

        sumf_d += d8[i0/4] * (sc[i0/2+0]*sumi_d.x + sc[i0/2+1]*sumi_d.y);
    }

    return d6 * sumf_d;
}

static __device__ __forceinline__ float vec_dot_q4_0_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q4_0 * bq4_0 = (const block_q4_0 *) vbq;

    int v[VDR_Q4_0_Q8_1_MMVQ];
    int u[2*VDR_Q4_0_Q8_1_MMVQ];

#pragma unroll
    for (int i = 0; i < VDR_Q4_0_Q8_1_MMVQ; ++i) {
        v[i]     = get_int_from_uint8(bq4_0->qs, iqs + i);
        u[2*i+0] = get_int_from_int8_aligned(bq8_1->qs, iqs + i);
        u[2*i+1] = get_int_from_int8_aligned(bq8_1->qs, iqs + i + QI4_0);
    }

    return vec_dot_q4_0_q8_1_impl<VDR_Q4_0_Q8_1_MMVQ>(v, u, bq4_0->d, bq8_1->ds);
}


static __device__ __forceinline__ float vec_dot_q4_1_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q4_1 * bq4_1 = (const block_q4_1 *) vbq;

    int v[VDR_Q4_1_Q8_1_MMVQ];
    int u[2*VDR_Q4_1_Q8_1_MMVQ];

#pragma unroll
    for (int i = 0; i < VDR_Q4_1_Q8_1_MMVQ; ++i) {
        v[i]    = get_int_from_uint8_aligned(bq4_1->qs, iqs + i);
        u[2*i+0] = get_int_from_int8_aligned(bq8_1->qs, iqs + i);
        u[2*i+1] = get_int_from_int8_aligned(bq8_1->qs, iqs + i + QI4_1);
    }

    return vec_dot_q4_1_q8_1_impl<VDR_Q4_1_Q8_1_MMVQ>(v, u, bq4_1->dm, bq8_1->ds);
}

static __device__ __forceinline__ float vec_dot_q5_0_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q5_0 * bq5_0 = (const block_q5_0 *) vbq;

    int vl[VDR_Q5_0_Q8_1_MMVQ];
    int vh[VDR_Q5_0_Q8_1_MMVQ];
    int  u[2*VDR_Q5_0_Q8_1_MMVQ];

#pragma unroll
    for (int i = 0; i < VDR_Q5_0_Q8_1_MMVQ; ++i) {
        vl[i]    = get_int_from_uint8(bq5_0->qs, iqs + i);
        vh[i]    = get_int_from_uint8(bq5_0->qh, 0) >> (4 * (iqs + i));
        u[2*i+0] = get_int_from_int8_aligned(bq8_1->qs, iqs + i);
        u[2*i+1] = get_int_from_int8_aligned(bq8_1->qs, iqs + i + QI5_0);
    }

    return vec_dot_q5_0_q8_1_impl<VDR_Q5_0_Q8_1_MMVQ>(vl, vh, u, bq5_0->d, bq8_1->ds);
}

static __device__ __forceinline__ float vec_dot_q5_1_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q5_1 * bq5_1 = (const block_q5_1 *) vbq;

    int vl[VDR_Q5_1_Q8_1_MMVQ];
    int vh[VDR_Q5_1_Q8_1_MMVQ];
    int  u[2*VDR_Q5_1_Q8_1_MMVQ];

#pragma unroll
    for (int i = 0; i < VDR_Q5_1_Q8_1_MMVQ; ++i) {
        vl[i]   = get_int_from_uint8_aligned(bq5_1->qs, iqs + i);
        vh[i]   = get_int_from_uint8_aligned(bq5_1->qh, 0) >> (4 * (iqs + i));
        u[2*i+0] = get_int_from_int8_aligned(bq8_1->qs, iqs + i);
        u[2*i+1] = get_int_from_int8_aligned(bq8_1->qs, iqs + i + QI5_1);
    }

    return vec_dot_q5_1_q8_1_impl<VDR_Q5_1_Q8_1_MMVQ>(vl, vh, u, bq5_1->dm, bq8_1->ds);
}

static __device__ __forceinline__ float vec_dot_q8_0_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q8_0 * bq8_0 = (const block_q8_0 *) vbq;

    int v[VDR_Q8_0_Q8_1_MMVQ];
    int u[VDR_Q8_0_Q8_1_MMVQ];

#pragma unroll
    for (int i = 0; i < VDR_Q8_0_Q8_1_MMVQ; ++i) {
        v[i] = get_int_from_int8(bq8_0->qs, iqs + i);
        u[i] = get_int_from_int8_aligned(bq8_1->qs, iqs + i);
    }

    return vec_dot_q8_0_q8_1_impl<VDR_Q8_0_Q8_1_MMVQ>(v, u, bq8_0->d, __low2half(bq8_1->ds));
}

static __device__ __forceinline__ float vec_dot_q2_K_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q2_K * bq2_K = (const block_q2_K *) vbq;

    const int bq8_offset = QR2_K * (iqs / QI8_1);
    const int scale_offset = iqs - iqs % QI8_1 + (iqs % QI8_1) / (QI8_1/2);

    const uint8_t * scales = bq2_K->scales + scale_offset;

    const int v = get_int_from_uint8_aligned(bq2_K->qs, iqs);
    int    u[QR2_K];
    float d8[QR2_K];

#pragma unroll
    for (int i = 0; i < QR2_K; ++ i) {
        u[i]  = get_int_from_int8_aligned(bq8_1[bq8_offset + i].qs, iqs % QI8_1);
        d8[i] = __low2float(bq8_1[bq8_offset + i].ds);
    }

    return vec_dot_q2_K_q8_1_impl_mmvq(v, u, scales, bq2_K->dm, d8);
}

static __device__ __forceinline__ float vec_dot_q3_K_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q3_K * bq3_K = (const block_q3_K *) vbq;

    const int bq8_offset = QR3_K * (iqs / (QI3_K/2));
    const int scale_offset = iqs - iqs % QI8_1 + (iqs % QI8_1) / (QI8_1/2);

    const float d = bq3_K->d;

    const int vl = get_int_from_uint8(bq3_K->qs, iqs);

    // invert the mask with ~ so that a 0/1 results in 4/0 being subtracted
    const int vh = ~get_int_from_uint8(bq3_K->hmask, iqs % (QI3_K/2)) >> bq8_offset;

    int    u[QR3_K];
    float d8[QR3_K];

#pragma unroll
    for (int i = 0; i < QR3_K; ++i) {
        u[i]  = get_int_from_int8_aligned(bq8_1[bq8_offset + i].qs, iqs % QI8_1);
        d8[i] = __low2float(bq8_1[bq8_offset + i].ds);
    }

    return vec_dot_q3_K_q8_1_impl_mmvq(vl, vh, u, bq3_K->scales, scale_offset, d, d8);
}

static __device__ __forceinline__ float vec_dot_q4_K_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

#ifndef GGML_QKK_64
    const block_q4_K * bq4_K = (const block_q4_K *) vbq;

    int    v[2];
    int    u[2*QR4_K];
    float d8[QR4_K];

    // iqs is in 0,2..30. bq8_offset = iqs/4 -> bq8_offset = 0, 2, 4, 6
    const int bq8_offset = QR4_K * ((iqs/2) / (QI8_1/2));

    // iqs = 0....3 -> bq8_offset = 0, want q4_offset = 0, 4, 8, 12
    // iqs = 4....7 -> bq8_offset = 2, want q4_offset = 32, 36, 40, 44
    // iqs = 8...11 -> bq8_offset = 4, want q4_offset = 64, 68, 72, 76
    // iqs = 12..15 -> bq8_offset = 6, want q4_offset = 96, 100, 104, 108

    const int * q4 = (const int *)(bq4_K->qs + 16 * bq8_offset + 4 * ((iqs/2)%4));
    v[0] = q4[0];
    v[1] = q4[4];

    const uint16_t * scales = (const uint16_t *)bq4_K->scales;
    uint16_t aux[2];
    const int j = bq8_offset/2;
    if (j < 2) {
        aux[0] = scales[j+0] & 0x3f3f;
        aux[1] = scales[j+2] & 0x3f3f;
    } else {
        aux[0] = ((scales[j+2] >> 0) & 0x0f0f) | ((scales[j-2] & 0xc0c0) >> 2);
        aux[1] = ((scales[j+2] >> 4) & 0x0f0f) | ((scales[j-0] & 0xc0c0) >> 2);
    }
    const uint8_t * sc = (const uint8_t *)aux;
    const uint8_t * m  = sc + 2;

    for (int i = 0; i < QR4_K; ++i) {
        const block_q8_1 * bq8i = bq8_1 + bq8_offset + i;
        d8[i] = __low2float(bq8i->ds);

        const int * q8 = (const int *)bq8i->qs + ((iqs/2)%4);
        u[2*i+0] = q8[0];
        u[2*i+1] = q8[4];
    }

    return vec_dot_q4_K_q8_1_impl_vmmq(v, u, sc, m, bq4_K->dm, d8);

#else

    const block_q4_K * bq4_K = (const block_q4_K *) vbq;

    float sumf_d = 0.0f;
    float sumf_m = 0.0f;

    uint16_t aux16[2];
    const uint8_t * s = (const uint8_t *)aux16;

    const uint16_t * a = (const uint16_t *)bq4_K->scales;
    aux16[0] = a[0] & 0x0f0f;
    aux16[1] = (a[0] >> 4) & 0x0f0f;

    const float dall = bq4_K->dm[0];
    const float dmin = bq4_K->dm[1];

    const float d8_1 = __low2float(bq8_1[0].ds);
    const float d8_2 = __low2float(bq8_1[1].ds);

    const int ui1 = *((const int *)bq8_1[0].qs + (iqs/2));
    const int ui2 = *((const int *)bq8_1[0].qs + (iqs/2) + 4);
    const int ui3 = *((const int *)bq8_1[1].qs + (iqs/2));
    const int ui4 = *((const int *)bq8_1[1].qs + (iqs/2) + 4);

    const int * q4 = (const int *)bq4_K->qs + (iqs/2);
    const int v1 = q4[0];
    const int v2 = q4[4];

    const int dot1 = ggml_cuda_dp4a(ui2, v2 & 0x0f0f0f0f, ggml_cuda_dp4a(ui1, v1 & 0x0f0f0f0f, 0));
    const int dot2 = ggml_cuda_dp4a(ui4, (v2 >> 4) & 0x0f0f0f0f, ggml_cuda_dp4a(ui3, (v1 >> 4) & 0x0f0f0f0f, 0));
    const int dot3 = ggml_cuda_dp4a(0x01010101, ui2, ggml_cuda_dp4a(0x01010101, ui1, 0));
    const int dot4 = ggml_cuda_dp4a(0x01010101, ui4, ggml_cuda_dp4a(0x01010101, ui3, 0));

    sumf_d += d8_1 * (dot1 * s[0]) + d8_2 * (dot2 * s[1]);
    sumf_m += d8_1 * (dot3 * s[2]) + d8_2 * (dot4 * s[3]);

    return dall * sumf_d - dmin * sumf_m;
#endif
}

static __device__ __forceinline__ float vec_dot_q5_K_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

#ifndef GGML_QKK_64
    const block_q5_K * bq5_K = (const block_q5_K *) vbq;

    int   vl[2];
    int   vh[2];
    int    u[2*QR5_K];
    float d8[QR5_K];

    const int bq8_offset = QR5_K * ((iqs/2) / (QI8_1/2));
    const int * ql = (const int *)(bq5_K->qs + 16 * bq8_offset + 4 * ((iqs/2)%4));
    const int * qh = (const int *)(bq5_K->qh + 4 * ((iqs/2)%4));

    vl[0] = ql[0];
    vl[1] = ql[4];

    vh[0] = qh[0] >> bq8_offset;
    vh[1] = qh[4] >> bq8_offset;

    const uint16_t * scales = (const uint16_t *)bq5_K->scales;
    uint16_t aux[2];
    const int j = bq8_offset/2;
    if (j < 2) {
        aux[0] = scales[j+0] & 0x3f3f;
        aux[1] = scales[j+2] & 0x3f3f;
    } else {
        aux[0] = ((scales[j+2] >> 0) & 0x0f0f) | ((scales[j-2] & 0xc0c0) >> 2);
        aux[1] = ((scales[j+2] >> 4) & 0x0f0f) | ((scales[j-0] & 0xc0c0) >> 2);
    }
    const uint8_t * sc = (const uint8_t *)aux;
    const uint8_t * m  = sc + 2;

#pragma unroll
    for (int i = 0; i < QR5_K; ++i) {
        const block_q8_1 * bq8i = bq8_1 + bq8_offset + i;
        d8[i] = __low2float(bq8i->ds);

        const int * q8 = (const int *)bq8i->qs + ((iqs/2)%4);
        u[2*i+0] = q8[0];
        u[2*i+1] = q8[4];
    }

    return vec_dot_q5_K_q8_1_impl_vmmq(vl, vh, u, sc, m, bq5_K->dm, d8);

#else

    const block_q5_K * bq5_K = (const block_q5_K *) vbq;

    const int8_t * s = bq5_K->scales;

    const float d = bq5_K->d;

    const float d8_1 = __low2half(bq8_1[0].ds);
    const float d8_2 = __low2half(bq8_1[1].ds);

    const int ui1 = *((const int *)bq8_1[0].qs + (iqs/2));
    const int ui2 = *((const int *)bq8_1[0].qs + (iqs/2) + 4);
    const int ui3 = *((const int *)bq8_1[1].qs + (iqs/2));
    const int ui4 = *((const int *)bq8_1[1].qs + (iqs/2) + 4);

    const int * ql = (const int *)bq5_K->qs + (iqs/2);
    const int vl1 = ql[0];
    const int vl2 = ql[4];

    const int step = 4 * (iqs/2); // 0, 4, 8, 12
    const int im = step/8; // = 0 for iqs = 0, 2, = 1 for iqs = 4, 6
    const int in = step%8; // 0, 4, 0, 4
    const int vh = (*((const int *)(bq5_K->qh + in))) >> im;

    const int v1 = (((vh << 4) & 0x10101010) ^ 0x10101010) | ((vl1 >> 0) & 0x0f0f0f0f);
    const int v2 = (((vh << 2) & 0x10101010) ^ 0x10101010) | ((vl2 >> 0) & 0x0f0f0f0f);
    const int v3 = (((vh >> 0) & 0x10101010) ^ 0x10101010) | ((vl1 >> 4) & 0x0f0f0f0f);
    const int v4 = (((vh >> 2) & 0x10101010) ^ 0x10101010) | ((vl2 >> 4) & 0x0f0f0f0f);

    const float sumf_d = d8_1 * (ggml_cuda_dp4a(ui1, v1, 0) * s[0] + ggml_cuda_dp4a(ui2, v2, 0) * s[1])
                       + d8_2 * (ggml_cuda_dp4a(ui3, v3, 0) * s[2] + ggml_cuda_dp4a(ui4, v4, 0) * s[3]);

    return d * sumf_d;
#endif
}

static __device__ __forceinline__ float vec_dot_q6_K_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs) {

    const block_q6_K * bq6_K = (const block_q6_K *) vbq;

    const int bq8_offset = 2 * QR6_K * (iqs / (QI6_K/2)) + (iqs % (QI6_K/2)) / (QI6_K/4);
    const int scale_offset = (QI6_K/4) * (iqs / (QI6_K/2)) + (iqs % (QI6_K/2)) / (QI6_K/8);
    const int vh_shift = 2 * ((iqs % (QI6_K/2)) / (QI6_K/4));

    const int vl = get_int_from_uint8(bq6_K->ql, iqs);
    const int vh = get_int_from_uint8(bq6_K->qh, (QI6_K/4) * (iqs / (QI6_K/2)) + iqs % (QI6_K/4)) >> vh_shift;

    const int8_t * scales = bq6_K->scales + scale_offset;

    int    u[QR6_K];
    float d8[QR6_K];

#pragma unroll
    for (int i = 0; i < QR6_K; ++i) {
        u[i]  = get_int_from_int8_aligned(bq8_1[bq8_offset + 2*i].qs, iqs % QI8_1);
        d8[i] = __low2float(bq8_1[bq8_offset + 2*i].ds);
    }

    return vec_dot_q6_K_q8_1_impl_mmvq(vl, vh, u, scales, bq6_K->d, d8);
}

// https://github.com/ggerganov/llama.cpp/blob/c50a82ce0f71558cbb8e555146ba124251504b38/ggml-cuda/mmvq.cu#L4
typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & iqs);

template <int ncols_y, int qk, int qi, typename block_q_t, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda>
static __device__ void mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int nwarps              = 1;
    constexpr int rows_per_cuda_block = 1;
#else
    constexpr int nwarps              = ncols_y <= 4 ? 4 : 2;
    constexpr int rows_per_cuda_block = ncols_y == 1 ? 1 : 2;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q_t  * x = (const block_q_t  *) vx;
    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp[j][i] += vec_dot_q_cuda(
                    &x[kbx + (row0 + i)*blocks_per_row_x], &y[j*blocks_per_col_y + kby], kqs);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block) {
            dst[j*nrows_dst + row0 + threadIdx.x] = tmp[j][threadIdx.x];
        }
    }
}

// batch size = 1
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda1(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<1, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

// batch size = 2
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda2(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<2, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

// batch size = 3
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda3(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<3, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

// batch size = 4
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda4(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<4, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

// batch size = 5
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda5(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<5, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

// batch size = 6
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda6(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<6, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

// batch size = 7
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda7(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<7, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

// batch size = 8
extern "C" __global__ void mul_mat_vec_q4_0_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK4_0, QI4_0, block_q4_0, VDR_Q4_0_Q8_1_MMVQ, vec_dot_q4_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_1_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK4_1, QI4_1, block_q4_1, VDR_Q4_1_Q8_1_MMVQ, vec_dot_q4_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_0_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK5_0, QI5_0, block_q5_0, VDR_Q5_0_Q8_1_MMVQ, vec_dot_q5_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_1_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK5_1, QI5_1, block_q5_1, VDR_Q5_1_Q8_1_MMVQ, vec_dot_q5_1_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q8_0_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK8_0, QI8_0, block_q8_0, VDR_Q8_0_Q8_1_MMVQ, vec_dot_q8_0_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q2_K_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK_K, QI2_K, block_q2_K, VDR_Q2_K_Q8_1_MMVQ, vec_dot_q2_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q3_K_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK_K, QI3_K, block_q3_K, VDR_Q3_K_Q8_1_MMVQ, vec_dot_q3_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q4_K_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK_K, QI4_K, block_q4_K, VDR_Q4_K_Q8_1_MMVQ, vec_dot_q4_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q5_K_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK_K, QI5_K, block_q5_K, VDR_Q5_K_Q8_1_MMVQ, vec_dot_q5_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void mul_mat_vec_q6_K_q8_1_cuda8(
    const void * vx, const void * vy, float * dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    mul_mat_vec_q<8, QK_K, QI6_K, block_q6_K, VDR_Q6_K_Q8_1_MMVQ, vec_dot_q6_K_q8_1>
        (vx, vy, dst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

extern "C" __global__ void quantize_q8_1(const float * __restrict__ x, void * __restrict__ vy, const int kx, const int kx_padded) {
    const int ix = blockDim.x*blockIdx.x + threadIdx.x;

    if (ix >= kx_padded) {
        return;
    }

    const int iy = blockDim.y*blockIdx.y + threadIdx.y;

    const int i_padded = iy*kx_padded + ix;

    block_q8_1 * y = (block_q8_1 *) vy;

    const int ib = i_padded / QK8_1; // block index
    const int iqs = i_padded % QK8_1; // quant index

    const float xi = ix < kx ? x[iy*kx + ix] : 0.0f;
    float amax = fabsf(xi);
    float sum = xi;

    amax = warp_reduce_max(amax);
    sum = warp_reduce_sum(sum);

    const float d = amax / 127;
    const int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    y[ib].qs[iqs] = q;

    if (iqs > 0) {
        return;
    }

    reinterpret_cast<half&>(y[ib].ds.x) = d;
    reinterpret_cast<half&>(y[ib].ds.y) = sum;
}

// Kernels from https://github.com/ggerganov/llama.cpp/blob/master/ggml-cuda/mmq.cu

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q5_0(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int  tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ float tile_x_d[mmq_y * (WARP_SIZE/QI5_0) + mmq_y/QI5_0];

    *x_ql = tile_x_ql;
    *x_dm = (half2 *) tile_x_d;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q5_0(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI5_0;
    const int kqsx = k % QI5_0;

    const block_q5_0 * bx0 = (const block_q5_0 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_0 * bxi = bx0 + i*blocks_per_row + kbx;

        const int ql = get_int_from_uint8(bxi->qs, kqsx);
        const int qh = get_int_from_uint8(bxi->qh, 0) >> (4 * (k % QI5_0));

        int qs0 = (ql >>  0)   & 0x0F0F0F0F;
        qs0    |= (qh <<  4)   & 0x00000010;  // 0 ->  4
        qs0    |= (qh << 11)   & 0x00001000;  // 1 -> 12
        qs0    |= (qh << 18)   & 0x00100000;  // 2 -> 20
        qs0    |= (qh << 25)   & 0x10000000;  // 3 -> 28
        qs0     = __vsubss4(qs0, 0x10101010); // subtract 16

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+0] = qs0;

        int qs1 = (ql >>  4)   & 0x0F0F0F0F;
        qs1    |= (qh >> 12)   & 0x00000010;  // 16 ->  4
        qs1    |= (qh >>  5)   & 0x00001000;  // 17 -> 12
        qs1    |= (qh <<  2)   & 0x00100000;  // 18 -> 20
        qs1    |= (qh <<  9)   & 0x10000000;  // 19 -> 28
        qs1     = __vsubss4(qs1, 0x10101010); // subtract 16

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+1] = qs1;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI5_0;
    const int kbxd = k % blocks_per_tile_x_row;
    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI5_0) {
        int i = i0 + i_offset * QI5_0 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_0 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI5_0) + i / QI5_0 + kbxd] = bxi->d;
    }
}

static __device__ __forceinline__ float vec_dot_q5_0_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));
    const int index_bx = i * (WARP_SIZE/QI5_0) + i/QI5_0 + k/QI5_0;
    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    int u[2*VDR_Q5_0_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q5_0_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI5_0) % WARP_SIZE];
    }

    return vec_dot_q8_0_q8_1_impl<QR5_0*VDR_Q5_0_Q8_1_MMQ>
        (&x_ql[i * (2*WARP_SIZE + 1) + 2 * k], u, x_dmf[index_bx], y_df[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}


template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q5_1(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int   tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI5_1) + mmq_y/QI5_1];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q5_1(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset < nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI5_1;
    const int kqsx = k % QI5_1;

    const block_q5_1 * bx0 = (const block_q5_1 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_1 * bxi = bx0 + i*blocks_per_row + kbx;

        const int ql = get_int_from_uint8_aligned(bxi->qs, kqsx);
        const int qh = get_int_from_uint8_aligned(bxi->qh, 0) >> (4 * (k % QI5_1));

        int qs0 = (ql >>  0) & 0x0F0F0F0F;
        qs0    |= (qh <<  4) & 0x00000010; // 0 ->  4
        qs0    |= (qh << 11) & 0x00001000; // 1 -> 12
        qs0    |= (qh << 18) & 0x00100000; // 2 -> 20
        qs0    |= (qh << 25) & 0x10000000; // 3 -> 28

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+0] = qs0;

        int qs1 = (ql >>  4) & 0x0F0F0F0F;
        qs1    |= (qh >> 12) & 0x00000010; // 16 ->  4
        qs1    |= (qh >>  5) & 0x00001000; // 17 -> 12
        qs1    |= (qh <<  2) & 0x00100000; // 18 -> 20
        qs1    |= (qh <<  9) & 0x10000000; // 19 -> 28

        x_ql[i * (2*WARP_SIZE + 1) + 2*k+1] = qs1;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI5_1;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI5_1) {
        int i = i0 + i_offset * QI5_1 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_1 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI5_1) + i / QI5_1 + kbxd] = bxi->dm;
    }
}

static __device__ __forceinline__ float vec_dot_q5_1_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));
    const int index_bx = i * (WARP_SIZE/QI5_1) + + i/QI5_1 + k/QI5_1;

    int u[2*VDR_Q5_1_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q5_1_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI5_1) % WARP_SIZE];
    }

    return vec_dot_q8_1_q8_1_impl<QR5_1*VDR_Q5_1_Q8_1_MMQ>
        (&x_ql[i * (2*WARP_SIZE + 1) + 2 * k], u, x_dm[index_bx], y_ds[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q8_0(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    __shared__ int  tile_x_qs[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ float tile_x_d[mmq_y * (WARP_SIZE/QI8_0) + mmq_y/QI8_0];

    *x_ql = tile_x_qs;
    *x_dm = (half2 *) tile_x_d;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q8_0(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI8_0;
    const int kqsx = k % QI8_0;
    float * x_dmf = (float *) x_dm;

    const block_q8_0 * bx0 = (const block_q8_0 *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q8_0 * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_int8(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI8_0;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI8_0) {
        int i = i0 + i_offset * QI8_0 + k / blocks_per_tile_x_row;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q8_0 * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI8_0) + i / QI8_0 + kbxd] = bxi->d;
    }
}

static __device__ __forceinline__ float vec_dot_q8_0_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    return vec_dot_q8_0_q8_1_impl<VDR_Q8_0_Q8_1_MMQ>
        (&x_ql[i * (WARP_SIZE + 1) + k], &y_qs[j * WARP_SIZE + k], x_dmf[i * (WARP_SIZE/QI8_0) + i/QI8_0 + k/QI8_0],
         y_df[j * (WARP_SIZE/QI8_1) + k/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q2_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI2_K) + mmq_y/QI2_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/4)     + mmq_y/4];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q2_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI2_K;
    const int kqsx = k % QI2_K;

    const block_q2_K * bx0 = (const block_q2_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q2_K * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8_aligned(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI2_K;
    const int kbxd = k % blocks_per_tile_x_row;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI2_K) {
        int i = (i0 + i_offset * QI2_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q2_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dm[i * (WARP_SIZE/QI2_K) + i / QI2_K + kbxd] = bxi->dm;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 4) {
        int i = i0 + i_offset * 4 + k / (WARP_SIZE/4);

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q2_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/4)) / (QI2_K/4);

        x_sc[i * (WARP_SIZE/4) + i / 4 + k % (WARP_SIZE/4)] = get_int_from_uint8_aligned(bxi->scales, k % (QI2_K/4));
    }
}

static __device__ __forceinline__ float vec_dot_q2_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const int kbx = k / QI2_K;
    const int ky  = (k % QI2_K) * QR2_K;
    const float * y_df = (const float *) y_ds;

    int v[QR2_K*VDR_Q2_K_Q8_1_MMQ];

    const int kqsx = i * (WARP_SIZE + 1) + kbx*QI2_K + (QI2_K/2) * (ky/(2*QI2_K)) + ky % (QI2_K/2);
    const int shift = 2 * ((ky % (2*QI2_K)) / (QI2_K/2));

#pragma unroll
    for (int l = 0; l < QR2_K*VDR_Q2_K_Q8_1_MMQ; ++l) {
        v[l] = (x_ql[kqsx + l] >> shift) & 0x03030303;
    }

    const uint8_t * scales = ((const uint8_t *) &x_sc[i * (WARP_SIZE/4) + i/4 + kbx*4]) + ky/4;

    const int index_y = j * WARP_SIZE + (QR2_K*k) % WARP_SIZE;
    return vec_dot_q2_K_q8_1_impl_mmq(v, &y_qs[index_y], scales, x_dm[i * (WARP_SIZE/QI2_K) + i/QI2_K + kbx], y_df[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q3_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {

    __shared__ int   tile_x_ql[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI3_K) + mmq_y/QI3_K];
    __shared__ int   tile_x_qh[mmq_y * (WARP_SIZE/2)     + mmq_y/2];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/4)     + mmq_y/4];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_qh = tile_x_qh;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q3_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI3_K;
    const int kqsx = k % QI3_K;

    const block_q3_K * bx0 = (const block_q3_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI3_K;
    const int kbxd = k % blocks_per_tile_x_row;
    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI3_K) {
        int i = (i0 + i_offset * QI3_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI3_K) + i / QI3_K + kbxd] = bxi->d;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 2) {
        int i = i0 + i_offset * 2 + k / (WARP_SIZE/2);

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/2)) / (QI3_K/2);

        // invert the mask with ~ so that a 0/1 results in 4/0 being subtracted
        x_qh[i * (WARP_SIZE/2) + i / 2 + k % (WARP_SIZE/2)] = ~get_int_from_uint8(bxi->hmask, k % (QI3_K/2));
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 4) {
        int i = i0 + i_offset * 4 + k / (WARP_SIZE/4);

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q3_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/4)) / (QI3_K/4);

        const int ksc = k % (QI3_K/4);

        const int ksc_low = ksc % (QI3_K/8);
        const int shift_low = 4 * (ksc / (QI3_K/8));
        const int sc_low = (get_int_from_uint8(bxi->scales, ksc_low) >> shift_low) & 0x0F0F0F0F;

        const int ksc_high = QI3_K/8;
        const int shift_high = 2 * ksc;
        const int sc_high = ((get_int_from_uint8(bxi->scales, ksc_high) >> shift_high) << 4) & 0x30303030;

        const int sc = __vsubss4(sc_low | sc_high, 0x20202020);

        x_sc[i * (WARP_SIZE/4) + i / 4 + k % (WARP_SIZE/4)] = sc;
    }
}

static __device__ __forceinline__ float vec_dot_q3_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {

    const int kbx  = k / QI3_K;
    const int ky  = (k % QI3_K) * QR3_K;
    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    const int8_t * scales = ((const int8_t *) (x_sc + i * (WARP_SIZE/4) + i/4 + kbx*4)) + ky/4;

    int v[QR3_K*VDR_Q3_K_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < QR3_K*VDR_Q3_K_Q8_1_MMQ; ++l) {
        const int kqsx = i * (WARP_SIZE + 1) + kbx*QI3_K + (QI3_K/2) * (ky/(2*QI3_K)) + ky % (QI3_K/2);
        const int shift = 2 * ((ky % 32) / 8);
        const int vll = (x_ql[kqsx + l] >> shift) & 0x03030303;

        const int vh = x_qh[i * (WARP_SIZE/2) + i/2 + kbx * (QI3_K/2) + (ky+l)%8] >> ((ky+l) / 8);
        const int vlh = (vh << 2) & 0x04040404;

        v[l] = __vsubss4(vll, vlh);
    }

    const int index_y = j * WARP_SIZE + (k*QR3_K) % WARP_SIZE;
    return vec_dot_q3_K_q8_1_impl_mmq(v, &y_qs[index_y], scales, x_dmf[i * (WARP_SIZE/QI3_K) + i/QI3_K + kbx], y_df[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q4_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (WARP_SIZE)       + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI4_K) + mmq_y/QI4_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/8)     + mmq_y/8];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q4_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI4_K; // == 0 if QK_K == 256
    const int kqsx = k % QI4_K; // == k if QK_K == 256

    const block_q4_K * bx0 = (const block_q4_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_K * bxi = bx0 + i*blocks_per_row + kbx;

        x_ql[i * (WARP_SIZE + 1) + k] = get_int_from_uint8_aligned(bxi->qs, kqsx);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI4_K; // == 1 if QK_K == 256
    const int kbxd = k % blocks_per_tile_x_row;          // == 0 if QK_K == 256

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI4_K) {
        int i = (i0 + i_offset * QI4_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_K * bxi = bx0 + i*blocks_per_row + kbxd;

#if QK_K == 256
        x_dm[i * (WARP_SIZE/QI4_K) + i / QI4_K + kbxd] = bxi->dm;
#else
        x_dm[i * (WARP_SIZE/QI4_K) + i / QI4_K + kbxd] = {bxi->dm[0], bxi->dm[1]};
#endif
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 8) {
        int i = (i0 + i_offset * 8 + k / (WARP_SIZE/8)) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q4_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/8)) / (QI4_K/8);

        const int * scales = (const int *) bxi->scales;

        const int ksc = k % (WARP_SIZE/8);

        // scale arrangement after the following two lines: sc0,...,sc3, sc4,...,sc7, m0,...,m3, m4,...,m8
        int scales8 = (scales[(ksc%2) + (ksc!=0)] >> (4 * (ksc & (ksc/2)))) & 0x0F0F0F0F; // lower 4 bits
        scales8    |= (scales[ksc/2]              >> (2 * (ksc % 2)))       & 0x30303030; // upper 2 bits

        x_sc[i * (WARP_SIZE/8) + i / 8 + ksc] = scales8;
    }
}

static __device__ __forceinline__ float vec_dot_q4_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const uint8_t * sc = ((const uint8_t *) &x_sc[i * (WARP_SIZE/8) + i/8 + k/16]) + 2*((k % 16) / 8);

    const int index_y = j * WARP_SIZE + (QR4_K*k) % WARP_SIZE;
    return vec_dot_q4_K_q8_1_impl_mmq(&x_ql[i * (WARP_SIZE + 1) + k], &y_qs[index_y], sc, sc+8,
                                      x_dm[i * (WARP_SIZE/QI4_K) + i/QI4_K], &y_ds[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q5_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI5_K) + mmq_y/QI5_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/8)     + mmq_y/8];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q5_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI5_K; // == 0 if QK_K == 256
    const int kqsx = k % QI5_K; // == k if QK_K == 256

    const block_q5_K * bx0 = (const block_q5_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_K * bxi = bx0 + i*blocks_per_row + kbx;
        const int ky = QR5_K*kqsx;

        const int ql = get_int_from_uint8_aligned(bxi->qs, kqsx);
        const int ql0 = (ql >> 0) & 0x0F0F0F0F;
        const int ql1 = (ql >> 4) & 0x0F0F0F0F;

        const int qh = get_int_from_uint8_aligned(bxi->qh, kqsx % (QI5_K/4));
        const int qh0 = ((qh >> (2 * (kqsx / (QI5_K/4)) + 0)) << 4) & 0x10101010;
        const int qh1 = ((qh >> (2 * (kqsx / (QI5_K/4)) + 1)) << 4) & 0x10101010;

        const int kq0 = ky - ky % (QI5_K/2) + k % (QI5_K/4) + 0;
        const int kq1 = ky - ky % (QI5_K/2) + k % (QI5_K/4) + (QI5_K/4);

        x_ql[i * (2*WARP_SIZE + 1) + kq0] = ql0 | qh0;
        x_ql[i * (2*WARP_SIZE + 1) + kq1] = ql1 | qh1;
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI5_K; // == 1 if QK_K == 256
    const int kbxd = k % blocks_per_tile_x_row;          // == 0 if QK_K == 256

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI5_K) {
        int i = (i0 + i_offset * QI5_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_K * bxi = bx0 + i*blocks_per_row + kbxd;

#if QK_K == 256
        x_dm[i * (WARP_SIZE/QI5_K) + i / QI5_K + kbxd] = bxi->dm;
#endif
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 8) {
        int i = (i0 + i_offset * 8 + k / (WARP_SIZE/8)) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q5_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/8)) / (QI5_K/8);

        const int * scales = (const int *) bxi->scales;

        const int ksc = k % (WARP_SIZE/8);

        // scale arrangement after the following two lines: sc0,...,sc3, sc4,...,sc7, m0,...,m3, m4,...,m8
        int scales8 = (scales[(ksc%2) + (ksc!=0)] >> (4 * (ksc & (ksc/2)))) & 0x0F0F0F0F; // lower 4 bits
        scales8    |= (scales[ksc/2]              >> (2 * (ksc % 2)))       & 0x30303030; // upper 2 bits

        x_sc[i * (WARP_SIZE/8) + i / 8 + ksc] = scales8;
    }
}

static __device__ __forceinline__ float vec_dot_q5_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const uint8_t * sc = ((const uint8_t *) &x_sc[i * (WARP_SIZE/8) + i/8 + k/16]) + 2 * ((k % 16) / 8);

    const int index_x = i * (QR5_K*WARP_SIZE + 1) +  QR5_K*k;
    const int index_y = j * WARP_SIZE             + (QR5_K*k) % WARP_SIZE;
    return vec_dot_q5_K_q8_1_impl_mmq(&x_ql[index_x], &y_qs[index_y], sc, sc+8,
                                      x_dm[i * (WARP_SIZE/QI5_K) + i/QI5_K], &y_ds[index_y/QI8_1]);
}

template <int mmq_y> static __device__ __forceinline__ void allocate_tiles_q6_K(int ** x_ql, half2 ** x_dm, int ** x_qh, int ** x_sc) {
    GGML_UNUSED(x_qh);

    __shared__ int   tile_x_ql[mmq_y * (2*WARP_SIZE)     + mmq_y];
    __shared__ half2 tile_x_dm[mmq_y * (WARP_SIZE/QI6_K) + mmq_y/QI6_K];
    __shared__ int   tile_x_sc[mmq_y * (WARP_SIZE/8)     + mmq_y/8];

    *x_ql = tile_x_ql;
    *x_dm = tile_x_dm;
    *x_sc = tile_x_sc;
}

template <int mmq_y, int nwarps, bool need_check> static __device__ __forceinline__ void load_tiles_q6_K(
    const void * __restrict__ vx, int * __restrict__ x_ql, half2 * __restrict__ x_dm, int * __restrict__ x_qh,
    int * __restrict__ x_sc, const int & i_offset, const int & i_max, const int & k, const int & blocks_per_row) {
    GGML_UNUSED(x_qh);

    GGML_CUDA_ASSUME(i_offset >= 0);
    GGML_CUDA_ASSUME(i_offset <  nwarps);
    GGML_CUDA_ASSUME(k >= 0);
    GGML_CUDA_ASSUME(k <  WARP_SIZE);

    const int kbx  = k / QI6_K; // == 0 if QK_K == 256
    const int kqsx = k % QI6_K; // == k if QK_K == 256

    const block_q6_K * bx0 = (const block_q6_K *) vx;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps) {
        int i = i0 + i_offset;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q6_K * bxi = bx0 + i*blocks_per_row + kbx;
        const int ky = QR6_K*kqsx;

        const int ql = get_int_from_uint8(bxi->ql, kqsx);
        const int ql0 = (ql >> 0) & 0x0F0F0F0F;
        const int ql1 = (ql >> 4) & 0x0F0F0F0F;

        const int qh = get_int_from_uint8(bxi->qh, (QI6_K/4) * (kqsx / (QI6_K/2)) + kqsx % (QI6_K/4));
        const int qh0 = ((qh >> (2 * ((kqsx % (QI6_K/2)) / (QI6_K/4)))) << 4) & 0x30303030;
        const int qh1 =  (qh >> (2 * ((kqsx % (QI6_K/2)) / (QI6_K/4))))       & 0x30303030;

        const int kq0 = ky - ky % QI6_K + k % (QI6_K/2) + 0;
        const int kq1 = ky - ky % QI6_K + k % (QI6_K/2) + (QI6_K/2);

        x_ql[i * (2*WARP_SIZE + 1) + kq0] = __vsubss4(ql0 | qh0, 0x20202020);
        x_ql[i * (2*WARP_SIZE + 1) + kq1] = __vsubss4(ql1 | qh1, 0x20202020);
    }

    const int blocks_per_tile_x_row = WARP_SIZE / QI6_K; // == 1 if QK_K == 256
    const int kbxd = k % blocks_per_tile_x_row;          // == 0 if QK_K == 256
    float * x_dmf = (float *) x_dm;

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * QI6_K) {
        int i = (i0 + i_offset * QI6_K + k / blocks_per_tile_x_row) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q6_K * bxi = bx0 + i*blocks_per_row + kbxd;

        x_dmf[i * (WARP_SIZE/QI6_K) + i / QI6_K + kbxd] = bxi->d;
    }

#pragma unroll
    for (int i0 = 0; i0 < mmq_y; i0 += nwarps * 8) {
        int i = (i0 + i_offset * 8 + k / (WARP_SIZE/8)) % mmq_y;

        if (need_check) {
            i = min(i, i_max);
        }

        const block_q6_K * bxi = bx0 + i*blocks_per_row + (k % (WARP_SIZE/8)) / 4;

        x_sc[i * (WARP_SIZE/8) + i / 8 + k % (WARP_SIZE/8)] = get_int_from_int8(bxi->scales, k % (QI6_K/8));
    }
}

static __device__ __forceinline__ float vec_dot_q6_K_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh);

    const float * x_dmf = (const float *) x_dm;
    const float * y_df  = (const float *) y_ds;

    const int8_t * sc = ((const int8_t *) &x_sc[i * (WARP_SIZE/8) + i/8 + k/8]);

    const int index_x = i * (QR6_K*WARP_SIZE + 1) +  QR6_K*k;
    const int index_y = j * WARP_SIZE             + (QR6_K*k) % WARP_SIZE;
    return vec_dot_q6_K_q8_1_impl_mmq(&x_ql[index_x], &y_qs[index_y], sc, x_dmf[i * (WARP_SIZE/QI6_K) + i/QI6_K], &y_df[index_y/QI8_1]);
}


static __device__ __forceinline__ float vec_dot_q4_0_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));
    const float * x_dmf = (const float *) x_dm;

    int u[2*VDR_Q4_0_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q4_0_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI4_0) % WARP_SIZE];
    }

    return vec_dot_q4_0_q8_1_impl<VDR_Q4_0_Q8_1_MMQ>
        (&x_ql[i * (WARP_SIZE + 1) + k], u, x_dmf[i * (WARP_SIZE/QI4_0) + i/QI4_0 + k/QI4_0],
         y_ds[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}

static __device__ __forceinline__ float vec_dot_q4_1_q8_1_mul_mat(
    const int * __restrict__ x_ql, const half2 * __restrict__ x_dm, const int * __restrict__ x_qh, const int * __restrict__ x_sc,
    const int * __restrict__ y_qs, const half2 * __restrict__ y_ds, const int & i, const int & j, const int & k) {
    GGML_UNUSED(x_qh); GGML_UNUSED(x_sc);

    const int kyqs = k % (QI8_1/2) + QI8_1 * (k / (QI8_1/2));

    int u[2*VDR_Q4_1_Q8_1_MMQ];

#pragma unroll
    for (int l = 0; l < VDR_Q4_1_Q8_1_MMQ; ++l) {
        u[2*l+0] = y_qs[j * WARP_SIZE + (kyqs + l)         % WARP_SIZE];
        u[2*l+1] = y_qs[j * WARP_SIZE + (kyqs + l + QI4_1) % WARP_SIZE];
    }

    return vec_dot_q4_1_q8_1_impl<VDR_Q4_1_Q8_1_MMQ>
        (&x_ql[i * (WARP_SIZE + 1) + k], u, x_dm[i * (WARP_SIZE/QI4_1) + i/QI4_1 + k/QI4_1],
         y_ds[j * (WARP_SIZE/QI8_1) + (2*k/QI8_1) % (WARP_SIZE/QI8_1)]);
}


extern "C" __global__ void
    mul_mat_q4_0(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q4_0_AMPERE;
    const int mmq_y  =  MMQ_Y_Q4_0_AMPERE;
    const int nwarps = NWARPS_Q4_0_AMPERE;

    mul_mat_q<QK4_0, QR4_0, QI4_0, true, block_q4_0, mmq_x, mmq_y, nwarps, allocate_tiles_q4_0<mmq_y>,
        load_tiles_q4_0<mmq_y, nwarps, true>, VDR_Q4_0_Q8_1_MMQ, vec_dot_q4_0_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
    mul_mat_q4_1(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q4_1_AMPERE;
    const int mmq_y  =  MMQ_Y_Q4_1_AMPERE;
    const int nwarps = NWARPS_Q4_1_AMPERE;

    mul_mat_q<QK4_1, QR4_1, QI4_1, true, block_q4_1, mmq_x, mmq_y, nwarps, allocate_tiles_q4_1<mmq_y>,
        load_tiles_q4_1<mmq_y, nwarps, true>, VDR_Q4_1_Q8_1_MMQ, vec_dot_q4_1_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}


extern "C" __global__ void
    mul_mat_q5_0(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q5_0_AMPERE;
    const int mmq_y  =  MMQ_Y_Q5_0_AMPERE;
    const int nwarps = NWARPS_Q5_0_AMPERE;

    mul_mat_q<QK5_0, QR5_0, QI5_0, false, block_q5_0, mmq_x, mmq_y, nwarps, allocate_tiles_q5_0<mmq_y>,
        load_tiles_q5_0<mmq_y, nwarps, true>, VDR_Q5_0_Q8_1_MMQ, vec_dot_q5_0_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
mul_mat_q5_1(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q5_1_AMPERE;
    const int mmq_y  =  MMQ_Y_Q5_1_AMPERE;
    const int nwarps = NWARPS_Q5_1_AMPERE;

    mul_mat_q<QK5_1, QR5_1, QI5_1, true, block_q5_1, mmq_x, mmq_y, nwarps, allocate_tiles_q5_1<mmq_y>,
        load_tiles_q5_1<mmq_y, nwarps, true>, VDR_Q5_1_Q8_1_MMQ, vec_dot_q5_1_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
    mul_mat_q8_0(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q8_0_AMPERE;
    const int mmq_y  =  MMQ_Y_Q8_0_AMPERE;
    const int nwarps = NWARPS_Q8_0_AMPERE;

    mul_mat_q<QK8_0, QR8_0, QI8_0, false, block_q8_0, mmq_x, mmq_y, nwarps, allocate_tiles_q8_0<mmq_y>,
        load_tiles_q8_0<mmq_y, nwarps, true>, VDR_Q8_0_Q8_1_MMQ, vec_dot_q8_0_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
mul_mat_q2_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q2_K_AMPERE;
    const int mmq_y  =  MMQ_Y_Q2_K_AMPERE;
    const int nwarps = NWARPS_Q2_K_AMPERE;
    mul_mat_q<QK_K, QR2_K, QI2_K, false, block_q2_K, mmq_x, mmq_y, nwarps, allocate_tiles_q2_K<mmq_y>,
        load_tiles_q2_K<mmq_y, nwarps, true>, VDR_Q2_K_Q8_1_MMQ, vec_dot_q2_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
    mul_mat_q3_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q3_K_AMPERE;
    const int mmq_y  =  MMQ_Y_Q3_K_AMPERE;
    const int nwarps = NWARPS_Q3_K_AMPERE;
    mul_mat_q<QK_K, QR3_K, QI3_K, false, block_q3_K, mmq_x, mmq_y, nwarps, allocate_tiles_q3_K<mmq_y>,
        load_tiles_q3_K<mmq_y, nwarps, true>, VDR_Q3_K_Q8_1_MMQ, vec_dot_q3_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
    mul_mat_q4_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q4_K_AMPERE;
    const int mmq_y  =  MMQ_Y_Q4_K_AMPERE;
    const int nwarps = NWARPS_Q4_K_AMPERE;
    mul_mat_q<QK_K, QR4_K, QI4_K, true, block_q4_K, mmq_x, mmq_y, nwarps, allocate_tiles_q4_K<mmq_y>,
        load_tiles_q4_K<mmq_y, nwarps, true>, VDR_Q4_K_Q8_1_MMQ, vec_dot_q4_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
mul_mat_q5_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q5_K_AMPERE;
    const int mmq_y  =  MMQ_Y_Q5_K_AMPERE;
    const int nwarps = NWARPS_Q5_K_AMPERE;
    mul_mat_q<QK_K, QR5_K, QI5_K, true, block_q5_K, mmq_x, mmq_y, nwarps, allocate_tiles_q5_K<mmq_y>,
        load_tiles_q5_K<mmq_y, nwarps, true>, VDR_Q5_K_Q8_1_MMQ, vec_dot_q5_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}

extern "C" __global__ void
    mul_mat_q6_K(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int ncols_y, const int nrows_y, const int nrows_dst) {
    const int mmq_x  =  MMQ_X_Q6_K_AMPERE;
    const int mmq_y  =  MMQ_Y_Q6_K_AMPERE;
    const int nwarps = NWARPS_Q6_K_AMPERE;
    mul_mat_q<QK_K, QR6_K, QI6_K, false, block_q6_K, mmq_x, mmq_y, nwarps, allocate_tiles_q6_K<mmq_y>,
        load_tiles_q6_K<mmq_y, nwarps, true>, VDR_Q6_K_Q8_1_MMQ, vec_dot_q6_K_q8_1_mul_mat>
        (vx, vy, dst, ncols_x, nrows_x, ncols_y, nrows_y, nrows_dst);
}
