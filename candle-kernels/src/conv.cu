#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include<stdint.h>

// Naive implementation of conv1d.
template <typename T, typename A>
__device__ void conv1d(
    const size_t src_numel,
    const size_t l_out,
    const size_t stride,
    const size_t padding,
    const size_t dilation,
    const size_t *info,
    const T *src,
    const T *kernel,
    T *dst
) {
  // src: (b_size, c_in, l_in)
  // k: (c_out, c_in, k_size)
  const size_t *src_dims = info;
  const size_t *src_s = info + 3;
  const size_t *k_dims = info + 6;
  const size_t *k_s = info + 9;
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t k_size = k_dims[2];
  const size_t c_out = k_dims[0];
  const size_t c_in = src_dims[1];
  const size_t l_in = src_dims[2];
  if (dst_i >= src_dims[0] * c_out * l_out) {
    return;
  }

  // TODO
  const size_t b_idx = dst_i / (l_out * c_out);
  const size_t dst_c_idx = (dst_i / l_out) % c_out;
  const size_t dst_l = dst_i % l_out;

  const size_t src_idx0 = b_idx * src_s[0];
  A d = 0;
  for (size_t offset = 0; offset < k_size; ++offset) {
    size_t src_l = (stride * dst_l + offset) * dilation;
    if (src_l < padding || src_l >= padding + l_in) {
      continue;
    }
    src_l -= padding;
    for (size_t src_c_idx = 0; src_c_idx < c_in; ++src_c_idx) {
      const size_t src_idx = src_idx0 + src_c_idx * src_s[1] + src_l * src_s[2];
      const size_t k_idx = dst_c_idx * k_s[0] + src_c_idx * k_s[1] + offset * k_s[2];
      d += static_cast<A>(src[src_idx]) * static_cast<A>(kernel[k_idx]);
    }
  }
  dst[dst_i] = static_cast<T>(d);
}

template <typename T>
__device__ void im2col1d(
    const size_t dst_numel,
    const size_t l_out,
    const size_t l_k,
    const size_t stride,
    const size_t padding,
    const size_t dilation,
    const size_t *info,
    const T *src,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // dst: (b_size, l_out, c_in, l_k)
  // src: (b_size, c_in, l_in)
  if (dst_i >= dst_numel) {
    return;
  }
  const size_t *src_dims = info;
  const size_t *src_s = info + 3;
  const size_t c_in = src_dims[1];
  const size_t l_in = src_dims[2];

  const size_t dst_s2 = l_k;
  const size_t dst_s1 = c_in * dst_s2;
  const size_t dst_s0 = l_out * dst_s1;

  size_t tmp_dst_i = dst_i;
  const size_t b_idx = tmp_dst_i / dst_s0;
  tmp_dst_i -= b_idx * dst_s0;
  const size_t l_idx = tmp_dst_i / dst_s1;
  tmp_dst_i -= l_idx * dst_s1;
  const size_t c_idx = tmp_dst_i / dst_s2;
  tmp_dst_i -= c_idx * dst_s2;
  const size_t l_k_idx = tmp_dst_i;
  size_t src_l_idx = l_idx * stride + l_k_idx * dilation;
  if (src_l_idx < padding || src_l_idx >= l_in + padding) {
    dst[dst_i] = static_cast<T>(0);
  }
  else {
    src_l_idx -= padding;
    const size_t src_i = b_idx * src_s[0] + c_idx * src_s[1] + src_l_idx * src_s[2];
    dst[dst_i] = src[src_i];
  }
}

template <typename T>
__device__ void col2im1d(
    const size_t dst_el,
    const size_t l_out,
    const size_t l_in,
    const size_t c_out,
    const size_t k_size,
    const size_t stride,
    const T *src,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // src: (b_size, l_in, c_out, l_k)
  // dst: (b_size, c_out, l_out)
  if (dst_i >= dst_el) {
    return;
  }

  const size_t dst_s0 = c_out * l_out;
  const size_t dst_s1 = l_out;
  const size_t src_s0 = c_out * k_size * l_in;
  const size_t src_s1 = c_out * k_size;
  const size_t src_s2 = k_size;

  size_t tmp_dst_i = dst_i;
  const size_t b_idx = tmp_dst_i / dst_s0;
  tmp_dst_i -= b_idx * dst_s0;
  const size_t c_idx = tmp_dst_i / dst_s1;
  tmp_dst_i -= c_idx * dst_s1;
  const int l_out_idx = tmp_dst_i;

  dst[dst_i] = static_cast<T>(0);

  int l_in_idx = l_out_idx / stride;
  int k0 = l_out_idx - l_in_idx * stride;
  // l_out_idx = l_in_idx * stride + k0
  for (; k0 < k_size && l_in_idx >= 0; k0 += stride, --l_in_idx) {
    if (l_in_idx < l_in) {
      const size_t src_i = b_idx * src_s0 + l_in_idx * src_s1 + c_idx * src_s2 + k0;
      dst[dst_i] += src[src_i];
    }
  }
}

template <typename T>
__device__ void im2col(
    const size_t dst_numel,
    const size_t h_out,
    const size_t w_out,
    const size_t h_k,
    const size_t w_k,
    const size_t stride,
    const size_t padding,
    const size_t dilation,
    const size_t *info,
    const T *src,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // dst: (b_size, h_out, w_out, c_in, h_k, w_k)
  // src: (b_size, c_in, h_in, w_in)
  if (dst_i >= dst_numel) {
    return;
  }
  const size_t *src_dims = info;
  const size_t *src_s = info + 4;
  const size_t c_in = src_dims[1];
  const size_t h_in = src_dims[2];
  const size_t w_in = src_dims[3];

  const size_t dst_s4 = w_k;
  const size_t dst_s3 = h_k * dst_s4;
  const size_t dst_s2 = c_in * dst_s3;
  const size_t dst_s1 = w_out * dst_s2;
  const size_t dst_s0 = h_out * dst_s1;

  size_t tmp_dst_i = dst_i;
  const size_t b_idx = tmp_dst_i / dst_s0;
  tmp_dst_i -= b_idx * dst_s0;
  const size_t h_idx = tmp_dst_i / dst_s1;
  tmp_dst_i -= h_idx * dst_s1;
  const size_t w_idx = tmp_dst_i / dst_s2;
  tmp_dst_i -= w_idx * dst_s2;
  const size_t c_idx = tmp_dst_i / dst_s3;
  tmp_dst_i -= c_idx * dst_s3;
  const size_t h_k_idx = tmp_dst_i / dst_s4;
  tmp_dst_i -= h_k_idx * dst_s4;
  const size_t w_k_idx = tmp_dst_i;
  size_t src_h_idx = h_idx * stride + h_k_idx * dilation;
  size_t src_w_idx = w_idx * stride + w_k_idx * dilation;
  if (src_h_idx < padding || src_h_idx >= h_in + padding) {
    dst[dst_i] = static_cast<T>(0);
  }
  else if (src_w_idx < padding || src_w_idx >= w_in + padding) {
    dst[dst_i] = static_cast<T>(0);
  }
  else {
    src_h_idx -= padding;
    src_w_idx -= padding;
    const size_t src_i =
      b_idx * src_s[0]
      + c_idx * src_s[1]
      + src_h_idx * src_s[2]
      + src_w_idx * src_s[3];
    dst[dst_i] = src[src_i];
  }
}

// Naive implementation of conv2d.
template <typename T, typename A>
__device__ void conv2d(
    const size_t src_numel,
    const size_t w_out,
    const size_t h_out,
    const size_t stride,
    const size_t padding,
    const size_t dilation,
    const size_t *info,
    const T *src,
    const T *kernel,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // src: (b_size, c_in, h_in, w_in)
  // k: (c_out, c_in, h_k, w_k)
  const size_t *src_dims = info;
  const size_t *src_s = info + 4;
  const size_t *k_dims = info + 8;
  const size_t *k_s = info + 12;
  const size_t h_k = k_dims[2];
  const size_t w_k = k_dims[3];
  const size_t c_out = k_dims[0];
  const size_t c_in = src_dims[1];
  const size_t h_in = src_dims[2];
  const size_t w_in = src_dims[3];
  if (dst_i >= src_dims[0] * c_out * w_out * h_out) {
    return;
  }

  // TODO
  const size_t b_idx = dst_i / (w_out * h_out * c_out);
  const size_t dst_c_idx = (dst_i / (w_out * h_out)) % c_out;
  // NCHW layout.
  const size_t dst_h = (dst_i / w_out) % h_out;
  const size_t dst_w = dst_i % w_out;

  const size_t src_idx0 = b_idx * src_s[0];
  A d = 0;
  for (size_t w_offset = 0; w_offset < w_k; ++w_offset) {
    size_t src_w = stride * dst_w + w_offset * dilation;
    if (src_w < padding || src_w >= w_in + padding) {
      continue;
    }
    src_w -= padding;
    for (size_t h_offset = 0; h_offset < h_k; ++h_offset) {
      size_t src_h = stride * dst_h + h_offset * dilation;
      if (src_h < padding || src_h >= h_in + padding) {
        continue;
      }
      src_h -= padding;
      for (size_t src_c_idx = 0; src_c_idx < c_in; ++src_c_idx) {
        const size_t src_idx = src_idx0 + src_c_idx * src_s[1] + src_h * src_s[2] + src_w * src_s[3];
        const size_t k_idx = dst_c_idx * k_s[0] + src_c_idx * k_s[1] + h_offset * k_s[2] + w_offset * k_s[3];
        d += static_cast<A>(src[src_idx]) * static_cast<A>(kernel[k_idx]);
      }
    }
  }
  dst[dst_i] = static_cast<T>(d);
}

// Naive implementation of conv_transpose1d.
template <typename T, typename A>
__device__ void conv_transpose1d(
    const size_t src_numel,
    const size_t l_out,
    const size_t stride,
    const size_t padding,
    const size_t out_padding,
    const size_t dilation,
    const size_t *info,
    const T *src,
    const T *kernel,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // src: (b_size, c_in, l_in)
  // k: (c_in, c_out, l_k)
  const size_t *src_dims = info;
  const size_t *src_s = info + 3;
  const size_t *k_dims = info + 6;
  const size_t *k_s = info + 9;
  const size_t l_k = k_dims[2];
  const size_t c_out = k_dims[1];
  const size_t c_in = src_dims[1];
  const size_t l_in = src_dims[2];
  if (dst_i >= src_dims[0] * c_out * l_out) {
    return;
  }

  // TODO
  const size_t b_idx = dst_i / (l_out * c_out);
  const size_t dst_c_idx = (dst_i / l_out) % c_out;
  // NCL layout.
  const size_t out_x = dst_i % l_out;

  const size_t src_idx0 = b_idx * src_s[0];
  A d = 0;
  for (int k_x = 0; k_x < (int)l_k; ++k_x) {
      // let out_x = inp_x * p.stride + k_x * p.dilation - p.padding;
      int inp_x_stride = (int)(out_x + padding) - k_x * dilation;
      if (inp_x_stride < 0 || inp_x_stride % stride) {
          continue;
      }
      int inp_x = inp_x_stride / stride;
      if (inp_x >= l_in) continue;
      for (size_t src_c_idx = 0; src_c_idx < c_in; ++src_c_idx) {
          const size_t src_idx = src_idx0 + src_c_idx * src_s[1] + inp_x * src_s[2];
          const size_t k_idx = src_c_idx * k_s[0] + dst_c_idx * k_s[1] + k_x * k_s[2];
          d += static_cast<A>(src[src_idx]) * static_cast<A>(kernel[k_idx]);
      }
  }
  dst[dst_i] = static_cast<T>(d);
}

// Naive implementation of conv_transpose2d.
template <typename T, typename A>
__device__ void conv_transpose2d(
    const size_t src_numel,
    const size_t w_out,
    const size_t h_out,
    const size_t stride,
    const size_t padding,
    const size_t out_padding,
    const size_t dilation,
    const size_t *info,
    const T *src,
    const T *kernel,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // src: (b_size, c_in, h_in, w_in)
  // k: (c_in, c_out, h_k, w_k)
  const size_t *src_dims = info;
  const size_t *src_s = info + 4;
  const size_t *k_dims = info + 8;
  const size_t *k_s = info + 12;
  const size_t h_k = k_dims[2];
  const size_t w_k = k_dims[3];
  const size_t c_out = k_dims[1];
  const size_t c_in = src_dims[1];
  const size_t h_in = src_dims[2];
  const size_t w_in = src_dims[3];
  if (dst_i >= src_dims[0] * c_out * w_out * h_out) {
    return;
  }

  // TODO
  const size_t b_idx = dst_i / (w_out * h_out * c_out);
  const size_t dst_c_idx = (dst_i / (w_out * h_out)) % c_out;
  // NCHW layout.
  const size_t out_y = (dst_i / w_out) % h_out;
  const size_t out_x = dst_i % w_out;

  const size_t src_idx0 = b_idx * src_s[0];
  A d = 0;
  for (int k_x = 0; k_x < (int)w_k; ++k_x) {
      // let out_x = inp_x * p.stride + k_x * p.dilation - p.padding;
      int inp_x_stride = (int)(out_x + padding) - k_x * dilation;
      if (inp_x_stride < 0 || inp_x_stride % stride) {
          continue;
      }
      int inp_x = inp_x_stride / stride;
      if (inp_x >= w_in) continue;
      for (int k_y = 0; k_y < (int)h_k; ++k_y) {
          int inp_y_stride = (int)(out_y + padding) - k_y * dilation;
          if (inp_y_stride < 0 || inp_y_stride % stride) {
              continue;
          }
          int inp_y = inp_y_stride / stride;
          if (inp_y >= h_in) continue;
          for (size_t src_c_idx = 0; src_c_idx < c_in; ++src_c_idx) {
              const size_t src_idx = src_idx0 + src_c_idx * src_s[1] + inp_y * src_s[2] + inp_x * src_s[3];
              const size_t k_idx = src_c_idx * k_s[0] + dst_c_idx * k_s[1] + k_y * k_s[2] + k_x * k_s[3];
              d += static_cast<A>(src[src_idx]) * static_cast<A>(kernel[k_idx]);
          }
      }
  }
  dst[dst_i] = static_cast<T>(d);
}

template <typename T, typename A>
__device__ void avg_pool2d(
    const size_t src_numel,
    const size_t w_k,
    const size_t h_k,
    const size_t w_stride,
    const size_t h_stride,
    const size_t *info,
    const T *src,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // src: (b_size, c_in, w_in, h_in)
  const size_t *src_dims = info;
  const size_t *src_s = info + 4;

  const size_t c = src_dims[1];
  const size_t w_in = src_dims[2];
  const size_t h_in = src_dims[3];

  const size_t w_out = (w_in - w_k) / w_stride + 1;
  const size_t h_out = (h_in - h_k) / h_stride + 1;
  if (dst_i >= src_dims[0] * c * w_out * h_out) {
    return;
  }

  // TODO: Improve this.
  const size_t b_idx = dst_i / (w_out * h_out * c);
  const size_t c_idx = (dst_i / (w_out * h_out)) % c;
  const size_t dst_w = (dst_i / h_out) % w_out;
  const size_t dst_h = dst_i % h_out;

  const size_t src_idx0 = b_idx * src_s[0];
  const float scale = 1.0 / (w_k * h_k);
  A d = 0;
  for (size_t w_offset = 0; w_offset < w_k; ++w_offset) {
    size_t src_w = w_stride * dst_w + w_offset;
    if (src_w >= w_in) {
      continue;
    }
    for (size_t h_offset = 0; h_offset < h_k; ++h_offset) {
      size_t src_h = h_stride * dst_h + h_offset;
      if (src_h >= h_in) {
        continue;
      }
      const size_t src_idx = src_idx0 + c_idx * src_s[1] + src_w * src_s[2] + src_h * src_s[3];
      d += static_cast<A>(src[src_idx]);
    }
  }
  dst[dst_i] = static_cast<T>(d * scale);
}

template <typename T>
__device__ void max_pool2d(
    const size_t src_numel,
    const size_t w_k,
    const size_t h_k,
    const size_t w_stride,
    const size_t h_stride,
    const size_t *info,
    const T *src,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // src: (b_size, c_in, w_in, h_in)
  const size_t *src_dims = info;
  const size_t *src_s = info + 4;

  const size_t c = src_dims[1];
  const size_t w_in = src_dims[2];
  const size_t h_in = src_dims[3];

  const size_t w_out = (w_in - w_k) / w_stride + 1;
  const size_t h_out = (h_in - h_k) / h_stride + 1;
  if (dst_i >= src_dims[0] * c * w_out * h_out) {
    return;
  }

  // TODO: Improve this.
  const size_t b_idx = dst_i / (w_out * h_out * c);
  const size_t c_idx = (dst_i / (w_out * h_out)) % c;
  const size_t dst_w = (dst_i / h_out) % w_out;
  const size_t dst_h = dst_i % h_out;

  const size_t src_idx0 = b_idx * src_s[0];
  T d = 0;
  bool set = false;
  for (size_t w_offset = 0; w_offset < w_k; ++w_offset) {
    size_t src_w = w_stride * dst_w + w_offset;
    if (src_w >= w_in) {
      continue;
    }
    for (size_t h_offset = 0; h_offset < h_k; ++h_offset) {
      size_t src_h = h_stride * dst_h + h_offset;
      if (src_h >= h_in) {
        continue;
      }
      const size_t src_idx = src_idx0 + c_idx * src_s[1] + src_w * src_s[2] + src_h * src_s[3];
      if (set) {
        d = maxg(d, src[src_idx]);
      }
      else {
        d = src[src_idx];
        set = true;
      }
    }
  }
  dst[dst_i] = d;
}

template <typename T>
__device__ void upsample_nearest2d(
    const size_t w_out,
    const size_t h_out,
    const double w_scale,
    const double h_scale,
    const size_t *info,
    const T *src,
    T *dst
) {
  const size_t dst_i = blockIdx.x * blockDim.x + threadIdx.x;
  // src: (b_size, c_in, w_in, h_in)
  const size_t *src_dims = info;
  const size_t *src_s = info + 4;

  const size_t c = src_dims[1];
  const size_t w_in = src_dims[2];
  const size_t h_in = src_dims[3];

  if (dst_i >= src_dims[0] * c * w_out * h_out) {
    return;
  }

  // TODO: Improve this.
  const size_t b_idx = dst_i / (w_out * h_out * c);
  const size_t c_idx = (dst_i / (w_out * h_out)) % c;
  const size_t dst_w = (dst_i / h_out) % w_out;
  const size_t dst_h = dst_i % h_out;

  size_t src_w = static_cast<size_t>(dst_w * w_scale);
  size_t src_h = static_cast<size_t>(dst_h * h_scale);
  if (src_w >= w_in) {
    src_w = w_in - 1;
  }
  if (src_h >= h_in) {
    src_h = h_in - 1;
  }

  const size_t src_i = b_idx * src_s[0] + c_idx * src_s[1] + src_w * src_s[2] + src_h * src_s[3];
  dst[dst_i] = src[src_i];
}


#define CONV1D_OP(TYPENAME, TYPEACC, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t num_dims, \
    const size_t stride, \
    const size_t padding, \
    const size_t dilation, \
    const size_t *info, \
    const TYPENAME *src, \
    const TYPENAME *kernel, \
    TYPENAME *dst \
) {  \
  conv1d<TYPENAME, TYPEACC>(src_numel, num_dims, stride, padding, dilation, info, src, kernel, dst); \
} \

#define CONV2D_OP(TYPENAME, TYPEACC, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t w_out, \
    const size_t h_out, \
    const size_t stride, \
    const size_t padding, \
    const size_t dilation, \
    const size_t *info, \
    const TYPENAME *src, \
    const TYPENAME *kernel, \
    TYPENAME *dst \
) {  \
  conv2d<TYPENAME, TYPEACC>(src_numel, w_out, h_out, stride, padding, dilation, info, src, kernel, dst); \
} \

#define IM2COL1D_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t dst_numel, \
    const size_t l_out, \
    const size_t l_k, \
    const size_t stride, \
    const size_t padding, \
    const size_t dilation, \
    const size_t *info, \
    const TYPENAME *src, \
    TYPENAME *dst \
) {  \
  im2col1d<TYPENAME>(dst_numel, l_out, l_k, stride, padding, dilation, info, src, dst); \
} \

#define COL2IM1D_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t dst_el, \
    const size_t l_out, \
    const size_t l_in, \
    const size_t c_out, \
    const size_t k_size, \
    const size_t stride, \
    const TYPENAME *src, \
    TYPENAME *dst \
) {  \
  col2im1d<TYPENAME>(dst_el, l_out, l_in, c_out, k_size, stride, src, dst); \
} \

#define IM2COL_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t dst_numel, \
    const size_t h_out, \
    const size_t w_out, \
    const size_t h_k, \
    const size_t w_k, \
    const size_t stride, \
    const size_t padding, \
    const size_t dilation, \
    const size_t *info, \
    const TYPENAME *src, \
    TYPENAME *dst \
) {  \
  im2col<TYPENAME>(dst_numel, h_out, w_out, h_k, w_k, stride, padding, dilation, info, src, dst); \
} \

#define CONVT1D_OP(TYPENAME, TYPEACC, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t l_out, \
    const size_t stride, \
    const size_t padding, \
    const size_t out_padding, \
    const size_t dilation, \
    const size_t *info, \
    const TYPENAME *src, \
    const TYPENAME *kernel, \
    TYPENAME *dst \
) {  \
  conv_transpose1d<TYPENAME, TYPEACC>(src_numel, l_out, stride, padding, out_padding, dilation, info, src, kernel, dst); \
} \

#define CONVT2D_OP(TYPENAME, TYPEACC, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t w_out, \
    const size_t h_out, \
    const size_t stride, \
    const size_t padding, \
    const size_t out_padding, \
    const size_t dilation, \
    const size_t *info, \
    const TYPENAME *src, \
    const TYPENAME *kernel, \
    TYPENAME *dst \
) {  \
  conv_transpose2d<TYPENAME, TYPEACC>(src_numel, w_out, h_out, stride, padding, out_padding, dilation, info, src, kernel, dst); \
} \

#define AVG_POOL2D_OP(TYPENAME, TYPEACC, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t w_k, \
    const size_t h_k, \
    const size_t w_stride, \
    const size_t h_stride, \
    const size_t *info, \
    const TYPENAME *src, \
    TYPENAME *dst \
) {  \
  avg_pool2d<TYPENAME, TYPEACC>(src_numel, w_k, h_k, w_stride, h_stride, info, src, dst); \
} \

#define MAX_POOL2D_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t src_numel, \
    const size_t w_k, \
    const size_t h_k, \
    const size_t w_stride, \
    const size_t h_stride, \
    const size_t *info, \
    const TYPENAME *src, \
    TYPENAME *dst \
) {  \
  max_pool2d<TYPENAME>(src_numel, w_k, h_k, w_stride, h_stride, info, src, dst); \
} \

#define UPSAMPLE_NEAREST2D_OP(TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t w_out, \
    const size_t h_out, \
    const double w_scale, \
    const double h_scale, \
    const size_t *info, \
    const TYPENAME *src, \
    TYPENAME *dst \
) {  \
  upsample_nearest2d<TYPENAME>(w_out, h_out, w_scale, h_scale, info, src, dst); \
} \

#if __CUDA_ARCH__ >= 800
CONV1D_OP(__hip_bfloat16, float, conv1d_bf16)
CONV2D_OP(__hip_bfloat16, float, conv2d_bf16)
CONVT1D_OP(__hip_bfloat16, float, conv_transpose1d_bf16)
CONVT2D_OP(__hip_bfloat16, float, conv_transpose2d_bf16)
AVG_POOL2D_OP(__hip_bfloat16, float, avg_pool2d_bf16)
MAX_POOL2D_OP(__hip_bfloat16, max_pool2d_bf16)
UPSAMPLE_NEAREST2D_OP(__hip_bfloat16, upsample_nearest2d_bf16)
IM2COL_OP(__hip_bfloat16, im2col_bf16)
IM2COL1D_OP(__hip_bfloat16, im2col1d_bf16)
COL2IM1D_OP(__hip_bfloat16, col2im1d_bf16)
#endif

#if __CUDA_ARCH__ >= 530
CONV1D_OP(__half, float, conv1d_f16)
CONV2D_OP(__half, float, conv2d_f16)
CONVT1D_OP(__half, float, conv_transpose1d_f16)
CONVT2D_OP(__half, float, conv_transpose2d_f16)
AVG_POOL2D_OP(__half, float, avg_pool2d_f16)
MAX_POOL2D_OP(__half, max_pool2d_f16)
UPSAMPLE_NEAREST2D_OP(__half, upsample_nearest2d_f16)
IM2COL_OP(__half, im2col_f16)
IM2COL1D_OP(__half, im2col1d_f16)
COL2IM1D_OP(__half, col2im1d_f16)
#endif

CONV1D_OP(float, float, conv1d_f32)
CONV1D_OP(double, double, conv1d_f64)
CONV1D_OP(uint8_t, uint8_t, conv1d_u8)
CONV1D_OP(uint32_t, uint32_t, conv1d_u32)

CONV2D_OP(float, float, conv2d_f32)
CONV2D_OP(double, double, conv2d_f64)
CONV2D_OP(uint8_t, uint8_t, conv2d_u8)
CONV2D_OP(uint32_t, uint32_t, conv2d_u32)

CONVT1D_OP(float, float, conv_transpose1d_f32)
CONVT1D_OP(double, double, conv_transpose1d_f64)
CONVT1D_OP(uint8_t, uint8_t, conv_transpose1d_u8)
CONVT1D_OP(uint32_t, uint32_t, conv_transpose1d_u32)

CONVT2D_OP(float, float, conv_transpose2d_f32)
CONVT2D_OP(double, double, conv_transpose2d_f64)
CONVT2D_OP(uint8_t, uint8_t, conv_transpose2d_u8)
CONVT2D_OP(uint32_t, uint32_t, conv_transpose2d_u32)

AVG_POOL2D_OP(float, float, avg_pool2d_f32)
AVG_POOL2D_OP(double, double, avg_pool2d_f64)
AVG_POOL2D_OP(uint8_t, uint8_t, avg_pool2d_u8)
AVG_POOL2D_OP(uint32_t, uint32_t, avg_pool2d_u32)

MAX_POOL2D_OP(float, max_pool2d_f32)
MAX_POOL2D_OP(double, max_pool2d_f64)
MAX_POOL2D_OP(uint8_t, max_pool2d_u8)
MAX_POOL2D_OP(uint32_t, max_pool2d_u32)

UPSAMPLE_NEAREST2D_OP(float, upsample_nearest2d_f32)
UPSAMPLE_NEAREST2D_OP(double, upsample_nearest2d_f64)
UPSAMPLE_NEAREST2D_OP(uint8_t, upsample_nearest2d_u8)
UPSAMPLE_NEAREST2D_OP(uint32_t, upsample_nearest2d_u32)

IM2COL_OP(float, im2col_f32)
IM2COL_OP(double, im2col_f64)
IM2COL_OP(uint8_t, im2col_u8)
IM2COL_OP(uint32_t, im2col_u32)

IM2COL1D_OP(float, im2col1d_f32)
IM2COL1D_OP(double, im2col1d_f64)
IM2COL1D_OP(uint8_t, im2col1d_u8)
IM2COL1D_OP(uint32_t, im2col1d_u32)

COL2IM1D_OP(float, col2im1d_f32)
COL2IM1D_OP(double, col2im1d_f64)
COL2IM1D_OP(uint8_t, col2im1d_u8)
COL2IM1D_OP(uint32_t, col2im1d_u32)
