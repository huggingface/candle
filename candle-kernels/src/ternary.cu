#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include<stdint.h>

#define WHERE_OP(TYPENAME, ID_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const ID_TYPENAME *ids, \
    const TYPENAME *t, \
    const TYPENAME *f, \
    TYPENAME *out \
) {  \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    const size_t *strides_t = info + 2*num_dims; \
    const size_t *strides_f = info + 3*num_dims; \
    if (is_contiguous(num_dims, dims, strides) \
        && is_contiguous(num_dims, dims, strides_f) \
        && is_contiguous(num_dims, dims, strides_t)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            out[i] = ids[i] ? t[i] : f[i]; \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            unsigned strided_i_t = get_strided_index(i, num_dims, dims, strides_t); \
            unsigned strided_i_f = get_strided_index(i, num_dims, dims, strides_f); \
            out[i] = ids[strided_i] ? t[strided_i_t] : f[strided_i_f]; \
        } \
    } \
} \

#if __CUDA_ARCH__ >= 800
WHERE_OP(__hip_bfloat16, int64_t, where_i64_bf16)
WHERE_OP(__hip_bfloat16, uint32_t, where_u32_bf16)
WHERE_OP(__hip_bfloat16, uint8_t, where_u8_bf16)

WHERE_OP(__hip_fp8_e4m3_fnuz, int16_t, where_i16_fp8_e4m3)
WHERE_OP(__hip_fp8_e4m3_fnuz, int32_t, where_i32_fp8_e4m3)
WHERE_OP(__hip_fp8_e4m3_fnuz, int64_t, where_i64_fp8_e4m3)
WHERE_OP(__hip_fp8_e4m3_fnuz, uint32_t, where_u32_fp8_e4m3)
WHERE_OP(__hip_fp8_e4m3_fnuz, uint8_t, where_u8_fp8_e4m3)
#endif

#if __CUDA_ARCH__ >= 530
WHERE_OP(__half, int64_t, where_i64_f16)
WHERE_OP(__half, uint32_t, where_u32_f16)
WHERE_OP(__half, uint8_t, where_u8_f16)
#endif

WHERE_OP(float, int64_t, where_i64_f32)
WHERE_OP(double, int64_t, where_i64_f64)
WHERE_OP(uint8_t, int64_t, where_i64_u8)
WHERE_OP(uint32_t, int64_t, where_i64_u32)
WHERE_OP(int64_t, int64_t, where_i64_i64)

WHERE_OP(float, uint32_t, where_u32_f32)
WHERE_OP(double, uint32_t, where_u32_f64)
WHERE_OP(uint8_t, uint32_t, where_u32_u8)
WHERE_OP(uint32_t, uint32_t, where_u32_u32)
WHERE_OP(int64_t, uint32_t, where_u32_i64)

WHERE_OP(float, uint8_t, where_u8_f32)
WHERE_OP(double, uint8_t, where_u8_f64)
WHERE_OP(uint8_t, uint8_t, where_u8_u8)
WHERE_OP(uint32_t, uint8_t, where_u8_u32)
WHERE_OP(int64_t, uint8_t, where_u8_i64)
