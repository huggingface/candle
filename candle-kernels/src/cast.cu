#include "hip/hip_runtime.h"
#include "cuda_utils.cuh"
#include<stdint.h>

template <typename S, typename T>
__device__ void cast_(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const S *inp,
    T *out
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    if (info == nullptr || is_contiguous(num_dims, dims, strides)) {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            out[i] = inp[i];
        }
    }
    else {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides);
            out[i] = inp[strided_i];
        }
    }
}

#define F8E4M3_TO_FLOAT(x) __half2float(__hip_cvt_fp8_to_halfraw(x.__x, __HIP_E4M3_FNUZ))

template <typename T>
__device__ void cast_fp8_(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const __hip_fp8_e4m3_fnuz *inp,
    T *out
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    if (info == nullptr || is_contiguous(num_dims, dims, strides)) {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            out[i] = F8E4M3_TO_FLOAT(inp[i]);
        }
    }
    else {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides);
            out[i] = F8E4M3_TO_FLOAT(inp[strided_i]);
        }
    }
}
template <typename S>
__device__ void cast_fp8_into_(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const S *inp,
    __hip_fp8_e4m3_fnuz *out
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    if (info == nullptr || is_contiguous(num_dims, dims, strides)) {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            out[i] = __hip_fp8_e4m3_fnuz((float)inp[i]);
        }
    }
    else {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides);
            out[i] = __hip_fp8_e4m3_fnuz((float)inp[strided_i]);
        }
    }
}

template <typename S, typename T, typename I>
__device__ void cast_through(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const S *inp,
    T *out
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    if (info == nullptr || is_contiguous(num_dims, dims, strides)) {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            out[i] = static_cast<T>(static_cast<I>(inp[i]));
        }
    }
    else {
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides);
            out[i] = static_cast<T>(static_cast<I>(inp[strided_i]));
        }
    }
}


#define CAST_OP(SRC_TYPENAME, DST_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const SRC_TYPENAME *inp, \
    DST_TYPENAME *out \
) { \
    cast_<SRC_TYPENAME, DST_TYPENAME>(numel, num_dims, info, inp, out); \
} \


#define CAST_OP_FP8(SRC_TYPENAME, DST_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const SRC_TYPENAME *inp, \
    DST_TYPENAME *out \
) { \
    cast_fp8_<DST_TYPENAME>(numel, num_dims, info, inp, out); \
} \


#define CAST_OP_FP8_INTO(SRC_TYPENAME, DST_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const SRC_TYPENAME *inp, \
    DST_TYPENAME *out \
) { \
    cast_fp8_into_<SRC_TYPENAME>(numel, num_dims, info, inp, out); \
} \

#define CAST_THROUGH_OP(SRC_TYPENAME, DST_TYPENAME, INT_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const SRC_TYPENAME *inp, \
    DST_TYPENAME *out \
) { \
    cast_through<SRC_TYPENAME, DST_TYPENAME, INT_TYPENAME>(numel, num_dims, info, inp, out); \
} \

#if __CUDA_ARCH__ >= 800
CAST_OP(__hip_bfloat16, __hip_bfloat16, cast_bf16_bf16)
CAST_OP(__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, cast_f8_e4m3_f8_e4m3)

CAST_OP(__hip_bfloat16, uint32_t, cast_bf16_u32)
CAST_OP(__hip_bfloat16, float,    cast_bf16_f32)
CAST_OP(__hip_bfloat16, double,   cast_bf16_f64)
CAST_OP(uint8_t, __hip_bfloat16, cast_u8_bf16)
CAST_OP(uint32_t, __hip_bfloat16, cast_u32_bf16)
CAST_OP(float,    __hip_bfloat16, cast_f32_bf16)
CAST_OP(double,   __hip_bfloat16, cast_f64_bf16)
CAST_THROUGH_OP(__hip_bfloat16, uint8_t, float, cast_bf16_u8)
CAST_THROUGH_OP(__hip_bfloat16, __half,   float, cast_bf16_f16)
CAST_THROUGH_OP(__half,   __hip_bfloat16, float, cast_f16_bf16)

CAST_OP_FP8(__hip_fp8_e4m3_fnuz, float,    cast_f8_e4m3_f32)
CAST_OP_FP8_INTO(float,    __hip_fp8_e4m3_fnuz, cast_f32_f8_e4m3)
CAST_OP_FP8(__hip_fp8_e4m3_fnuz, uint8_t, cast_f8_e4m3_u8)
CAST_OP_FP8(__hip_fp8_e4m3_fnuz, __half, cast_f8_e4m3_f16)
CAST_OP_FP8(__hip_fp8_e4m3_fnuz, double,  cast_f8_e4m3_f64)
CAST_OP_FP8_INTO(__half,   __hip_fp8_e4m3_fnuz, cast_f16_f8_e4m3)
CAST_OP_FP8_INTO(double,   __hip_fp8_e4m3_fnuz, cast_f64_f8_e4m3)
CAST_OP_FP8_INTO(uint8_t,   __hip_fp8_e4m3_fnuz, cast_u8_f8_e4m3)
CAST_OP_FP8_INTO(int32_t,   __hip_fp8_e4m3_fnuz, cast_i32_f8_e4m3)
CAST_OP_FP8(__hip_fp8_e4m3_fnuz, int32_t, cast_f8_e4m3_i32)
CAST_OP_FP8(__hip_fp8_e4m3_fnuz, __hip_bfloat16, cast_f8_e4m3_bf16)
CAST_OP_FP8_INTO(__hip_bfloat16, __hip_fp8_e4m3_fnuz, cast_bf16_f8_e4m3)
#else
#include <hip/hip_runtime.h>
#if CUDA_VERSION >= 11000
CAST_OP(__hip_bfloat16, float,    cast_bf16_f32)
CAST_OP(float,    __hip_bfloat16, cast_f32_bf16)
CAST_THROUGH_OP(__hip_bfloat16, uint8_t, float, cast_bf16_u8)
CAST_THROUGH_OP(__hip_bfloat16, __half,  float, cast_bf16_f16)
CAST_THROUGH_OP(__hip_bfloat16, double,  float, cast_bf16_f64)
CAST_THROUGH_OP(__half,   __hip_bfloat16, float, cast_f16_bf16)
CAST_THROUGH_OP(double,   __hip_bfloat16, float, cast_f64_bf16)
CAST_THROUGH_OP(uint8_t,   __hip_bfloat16, float, cast_u8_bf16)
CAST_THROUGH_OP(__hip_bfloat16, __hip_fp8_e4m3_fnuz, float, cast_bf16_f8_e4m3)
#endif
#endif

#if __CUDA_ARCH__ >= 530
CAST_OP(__half, __half, cast_f16_f16)

CAST_THROUGH_OP(__half, uint8_t,  float, cast_f16_u8)
CAST_OP(__half, uint32_t, cast_f16_u32)
CAST_OP(__half, float,    cast_f16_f32)
CAST_OP(__half, double,   cast_f16_f64)
CAST_OP(uint8_t,  __half, cast_u8_f16 )
CAST_OP(uint32_t, __half, cast_u32_f16)
CAST_OP(float,    __half, cast_f32_f16)
CAST_OP(double,   __half, cast_f64_f16)
#endif

CAST_OP(uint32_t, uint32_t, cast_u32_u32)
CAST_OP(uint32_t, uint8_t,  cast_u32_u8 )
CAST_OP(uint32_t, int64_t,  cast_u32_i64 )
CAST_OP(uint32_t, float,    cast_u32_f32)
CAST_OP(uint32_t, double,   cast_u32_f64)

CAST_OP(uint8_t, uint32_t, cast_u8_u32)
CAST_OP(uint8_t, uint8_t,  cast_u8_u8 )
CAST_OP(uint8_t, int64_t,  cast_u8_i64 )
CAST_OP(uint8_t, float,    cast_u8_f32)
CAST_OP(uint8_t, double,   cast_u8_f64)

CAST_OP(int64_t, uint32_t, cast_i64_u32)
CAST_OP(int64_t, uint8_t,  cast_i64_u8 )
CAST_OP(int64_t, int64_t,  cast_i64_i64 )
CAST_OP(int64_t, float,    cast_i64_f32)
CAST_OP(int64_t, double,   cast_i64_f64)

CAST_OP(float, uint8_t,  cast_f32_u8 )
CAST_OP(float, uint32_t, cast_f32_u32)
CAST_OP(float, int64_t,  cast_f32_i64 )
CAST_OP(float, float,    cast_f32_f32)
CAST_OP(float, double,   cast_f32_f64)

CAST_OP(double, uint8_t,  cast_f64_u8 )
CAST_OP(double, uint32_t, cast_f64_u32)
CAST_OP(double, int64_t,  cast_f64_i64 )
CAST_OP(double, float,    cast_f64_f32)
CAST_OP(double, double,   cast_f64_f64)
