#include "hip/hip_runtime.h"
// WARNING: THIS IS ONLY VALID ASSUMING THAT inp IS CONTIGUOUS!
// TODO: proper error reporting when ids are larger than v_size.
#include "cuda_utils.cuh"
#include<stdint.h>

template<typename T, typename I>
__device__ void index_select(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t ids_dim_size,
    const size_t right_size
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    bool b = is_contiguous(num_dims, dims, strides);
    for (unsigned int dst_i = blockIdx.x * blockDim.x + threadIdx.x; dst_i < numel; dst_i += blockDim.x * gridDim.x) {
          unsigned int left_i = dst_i / (ids_dim_size * right_size);
          unsigned int id_i = dst_i / right_size % ids_dim_size;
          unsigned int right_i = dst_i % right_size;
          unsigned int src_i = left_i * (src_dim_size * right_size) + ids[id_i] * right_size + right_i;
          unsigned strided_i = b ? src_i : get_strided_index(src_i, num_dims, dims, strides);
          out[dst_i] = inp[strided_i];
    }
}

#define IS_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t ids_dim_size, \
    const size_t right_size \
) { index_select(numel, num_dims, info, ids, inp, out, left_size, src_dim_size, ids_dim_size, right_size); } \

template<typename T, typename I>
__device__ void gather(
    const size_t numel,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t ids_dim_size,
    const size_t right_size
) {
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
        size_t post = i % right_size;
        size_t idx = ids[i];
        size_t pre = i / (right_size * ids_dim_size);
        size_t src_i = (pre * src_dim_size + idx) * right_size + post;
        out[i] = inp[src_i];
    }
}

#define GATHER_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t ids_dim_size, \
    const size_t right_size \
) { gather(numel, ids, inp, out, left_size, src_dim_size, ids_dim_size, right_size); } \

template<typename T, typename I>
__device__ void index_add(
    const I *ids,
    const size_t ids_dim_size,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < ids_dim_size; ++j) {
              const size_t idx = ids[j];
              const size_t src_i = (pre * ids_dim_size + j) * right_size + post;
              const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
              out[dst_i] += inp[src_i];
          }
      }
}

#if __CUDA_ARCH__ >= 800
#define F8E4M3_TO_FLOAT(x) __half2float(__hip_cvt_fp8_to_halfraw(x.__x, __HIP_E4M3_FNUZ))

template<typename I>
__device__ void scatter_add_f8(
    const I *ids,
    const __hip_fp8_e4m3_fnuz *inp,
    __hip_fp8_e4m3_fnuz *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < src_dim_size; ++j) {
              const size_t src_i = (pre * src_dim_size + j) * right_size + post;
              const size_t idx = ids[src_i];
              const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
              out[dst_i] = __hip_fp8_e4m3_fnuz(F8E4M3_TO_FLOAT(out[dst_i]) + F8E4M3_TO_FLOAT(inp[src_i]));
          }
      }
}

template<typename I>
__device__ void index_add_f8(
    const I *ids,
    const size_t ids_dim_size,
    const __hip_fp8_e4m3_fnuz *inp,
    __hip_fp8_e4m3_fnuz *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < ids_dim_size; ++j) {
              const size_t idx = ids[j];
              const size_t src_i = (pre * ids_dim_size + j) * right_size + post;
              const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
              out[dst_i] = __hip_fp8_e4m3_fnuz(F8E4M3_TO_FLOAT(out[dst_i]) + F8E4M3_TO_FLOAT(inp[src_i]));
          }
      }
}
#endif

#define IA_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const size_t ids_dim_size, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { index_add(ids, ids_dim_size, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \

#define IA_OP_F8(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const size_t ids_dim_size, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { index_add_f8(ids, ids_dim_size, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \

template<typename T, typename I>
__device__ void scatter_add(
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < src_dim_size; ++j) {
              const size_t src_i = (pre * src_dim_size + j) * right_size + post;
              const size_t idx = ids[src_i];
              const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
              out[dst_i] += inp[src_i];
          }
      }
}

#define SA_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { scatter_add(ids, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \

#define SA_OP_F8(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { scatter_add_f8(ids, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \


#if __CUDA_ARCH__ >= 800
IS_OP(__hip_bfloat16, int64_t, is_i64_bf16)
IS_OP(__hip_bfloat16, uint32_t, is_u32_bf16)
IS_OP(__hip_bfloat16, uint8_t, is_u8_bf16)
GATHER_OP(__hip_bfloat16, int64_t, gather_i64_bf16)
GATHER_OP(__hip_bfloat16, uint32_t, gather_u32_bf16)
GATHER_OP(__hip_bfloat16, uint8_t, gather_u8_bf16)
IA_OP(__hip_bfloat16, int64_t, ia_i64_bf16)
IA_OP(__hip_bfloat16, uint32_t, ia_u32_bf16)
IA_OP(__hip_bfloat16, uint8_t, ia_u8_bf16)
SA_OP(__hip_bfloat16, int64_t, sa_i64_bf16)
SA_OP(__hip_bfloat16, uint32_t, sa_u32_bf16)
SA_OP(__hip_bfloat16, uint8_t, sa_u8_bf16)

IS_OP(__hip_fp8_e4m3_fnuz, int16_t, is_i16_f8_e4m3)
IS_OP(__hip_fp8_e4m3_fnuz, int32_t, is_i32_f8_e4m3)
IS_OP(__hip_fp8_e4m3_fnuz, int64_t, is_i64_f8_e4m3)
IS_OP(__hip_fp8_e4m3_fnuz, uint32_t, is_u32_f8_e4m3)
IS_OP(__hip_fp8_e4m3_fnuz, uint8_t, is_u8_f8_e4m3)
GATHER_OP(__hip_fp8_e4m3_fnuz, int16_t, gather_i16_f8_e4m3)
GATHER_OP(__hip_fp8_e4m3_fnuz, int32_t, gather_i32_f8_e4m3)
GATHER_OP(__hip_fp8_e4m3_fnuz, int64_t, gather_i64_f8_e4m3)
GATHER_OP(__hip_fp8_e4m3_fnuz, uint32_t, gather_u32_f8_e4m3)
GATHER_OP(__hip_fp8_e4m3_fnuz, uint8_t, gather_u8_f8_e4m3)
IA_OP_F8(__hip_fp8_e4m3_fnuz, int16_t, ia_i16_f8_e4m3)
IA_OP_F8(__hip_fp8_e4m3_fnuz, int32_t, ia_i32_f8_e4m3)
IA_OP_F8(__hip_fp8_e4m3_fnuz, int64_t, ia_i64_f8_e4m3)
IA_OP_F8(__hip_fp8_e4m3_fnuz, uint32_t, ia_u32_f8_e4m3)
IA_OP_F8(__hip_fp8_e4m3_fnuz, uint8_t, ia_u8_f8_e4m3)
SA_OP_F8(__hip_fp8_e4m3_fnuz, int16_t, sa_i16_f8_e4m3)
SA_OP_F8(__hip_fp8_e4m3_fnuz, int32_t, sa_i32_f8_e4m3)
SA_OP_F8(__hip_fp8_e4m3_fnuz, int64_t, sa_i64_f8_e4m3)
SA_OP_F8(__hip_fp8_e4m3_fnuz, uint32_t, sa_u32_f8_e4m3)
SA_OP_F8(__hip_fp8_e4m3_fnuz, uint8_t, sa_u8_f8_e4m3)
#endif

#if __CUDA_ARCH__ >= 530
IS_OP(__half, int64_t, is_i64_f16)
IS_OP(__half, uint32_t, is_u32_f16)
IS_OP(__half, uint8_t, is_u8_f16)
GATHER_OP(__half, int64_t, gather_i64_f16)
GATHER_OP(__half, uint32_t, gather_u32_f16)
GATHER_OP(__half, uint8_t, gather_u8_f16)
IA_OP(__half, int64_t, ia_i64_f16)
IA_OP(__half, uint32_t, ia_u32_f16)
IA_OP(__half, uint8_t, ia_u8_f16)
SA_OP(__half, int64_t, sa_i64_f16)
SA_OP(__half, uint32_t, sa_u32_f16)
SA_OP(__half, uint8_t, sa_u8_f16)
#endif

IS_OP(float, int64_t, is_i64_f32)
IS_OP(double, int64_t, is_i64_f64)
IS_OP(uint8_t, int64_t, is_i64_u8)
IS_OP(uint32_t, int64_t, is_i64_u32)
IS_OP(int64_t, int64_t, is_i64_i64)

IS_OP(float, uint32_t, is_u32_f32)
IS_OP(double, uint32_t, is_u32_f64)
IS_OP(uint8_t, uint32_t, is_u32_u8)
IS_OP(int64_t, uint32_t, is_u32_i64)
IS_OP(uint32_t, uint32_t, is_u32_u32)

IS_OP(float, uint8_t, is_u8_f32)
IS_OP(double, uint8_t, is_u8_f64)
IS_OP(uint8_t, uint8_t, is_u8_u8)
IS_OP(uint32_t, uint8_t, is_u8_u32)
IS_OP(int64_t, uint8_t, is_u8_i64)

GATHER_OP(float, int64_t, gather_i64_f32)
GATHER_OP(double, int64_t, gather_i64_f64)
GATHER_OP(uint8_t, int64_t, gather_i64_u8)
GATHER_OP(uint32_t, int64_t, gather_i64_u32)
GATHER_OP(int64_t, int64_t, gather_i64_i64)

GATHER_OP(float, uint32_t, gather_u32_f32)
GATHER_OP(double, uint32_t, gather_u32_f64)
GATHER_OP(uint8_t, uint32_t, gather_u32_u8)
GATHER_OP(int64_t, uint32_t, gather_u32_i64)
GATHER_OP(uint32_t, uint32_t, gather_u32_u32)

GATHER_OP(float, uint8_t, gather_u8_f32)
GATHER_OP(double, uint8_t, gather_u8_f64)
GATHER_OP(uint8_t, uint8_t, gather_u8_u8)
GATHER_OP(uint32_t, uint8_t, gather_u8_u32)
GATHER_OP(int64_t, uint8_t, gather_u8_i64)

IA_OP(float, int64_t, ia_i64_f32)
IA_OP(double, int64_t, ia_i64_f64)
IA_OP(uint8_t, int64_t, ia_i64_u8)
IA_OP(int64_t, int64_t, ia_i64_i64)
IA_OP(uint32_t, int64_t, ia_i64_u32)

IA_OP(float, uint32_t, ia_u32_f32)
IA_OP(double, uint32_t, ia_u32_f64)
IA_OP(uint8_t, uint32_t, ia_u32_u8)
IA_OP(int64_t, uint32_t, ia_u32_i64)
IA_OP(uint32_t, uint32_t, ia_u32_u32)

IA_OP(float, uint8_t, ia_u8_f32)
IA_OP(double, uint8_t, ia_u8_f64)
IA_OP(uint8_t, uint8_t, ia_u8_u8)
IA_OP(uint32_t, uint8_t, ia_u8_u32)
IA_OP(int64_t, uint8_t, ia_u8_i64)

SA_OP(float, int64_t, sa_i64_f32)
SA_OP(double, int64_t, sa_i64_f64)
SA_OP(uint8_t, int64_t, sa_i64_u8)
SA_OP(int64_t, int64_t, sa_i64_i64)
SA_OP(uint32_t, int64_t, sa_i64_u32)

SA_OP(float, uint32_t, sa_u32_f32)
SA_OP(double, uint32_t, sa_u32_f64)
SA_OP(uint8_t, uint32_t, sa_u32_u8)
SA_OP(int64_t, uint32_t, sa_u32_i64)
SA_OP(uint32_t, uint32_t, sa_u32_u32)

SA_OP(float, uint8_t, sa_u8_f32)
SA_OP(double, uint8_t, sa_u8_f64)
SA_OP(uint8_t, uint8_t, sa_u8_u8)
SA_OP(uint32_t, uint8_t, sa_u8_u32)
SA_OP(int64_t, uint8_t, sa_u8_i64)
