#include "hip/hip_runtime.h"
// WARNING: THIS IS ONLY VALID ASSUMING THAT inp IS CONTIGUOUS!
// TODO: proper error reporting when ids are larger than v_size.
#include "cuda_utils.cuh"
#include<stdint.h>

template <typename T>
__host__ __device__
constexpr T max_value();

template <>
__host__ __device__
constexpr int64_t max_value<int64_t>() {
    return 0x7FFFFFFFFFFFFFFFLL;
}

template <>
__host__ __device__
constexpr uint32_t max_value<uint32_t>() {
    return 0xFFFFFFFFu;
}

template <>
__host__ __device__
constexpr uint8_t max_value<uint8_t>() {
    return 0xFFu;
}

template<typename T, typename I>
__device__ void index_select(
    const size_t numel,
    const size_t num_dims,
    const size_t *info,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t ids_dim_size,
    const size_t right_size
) {
    const size_t *dims = info;
    const size_t *strides = info + num_dims;
    bool b = is_contiguous(num_dims, dims, strides);
    for (unsigned int dst_i = blockIdx.x * blockDim.x + threadIdx.x; dst_i < numel; dst_i += blockDim.x * gridDim.x) {
          unsigned int left_i = dst_i / (ids_dim_size * right_size);
          unsigned int id_i = dst_i / right_size % ids_dim_size;
          unsigned int right_i = dst_i % right_size;
          if (ids[id_i] == max_value<I>()) {
            out[dst_i] = static_cast<T>(0);
          } else {
            assert(ids[id_i] < src_dim_size);
            unsigned int src_i = left_i * (src_dim_size * right_size) + ids[id_i] * right_size + right_i;
            unsigned strided_i = b ? src_i : get_strided_index(src_i, num_dims, dims, strides);
            out[dst_i] = inp[strided_i];
          }
    }
}

#define IS_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const size_t num_dims, \
    const size_t *info, \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t ids_dim_size, \
    const size_t right_size \
) { index_select(numel, num_dims, info, ids, inp, out, left_size, src_dim_size, ids_dim_size, right_size); } \

template<typename T, typename I>
__device__ void gather(
    const size_t numel,
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t ids_dim_size,
    const size_t right_size
) {
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
        size_t post = i % right_size;
        const I idx = ids[i];
        if (ids[i] == max_value<I>()) {
          out[i] = static_cast<T>(0);
        } else {
          assert(idx < src_dim_size);
          size_t pre = i / (right_size * ids_dim_size);
          size_t src_i = (pre * src_dim_size + idx) * right_size + post;
          out[i] = inp[src_i];
        }
    }
}

#define GATHER_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const size_t numel,  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t ids_dim_size, \
    const size_t right_size \
) { gather(numel, ids, inp, out, left_size, src_dim_size, ids_dim_size, right_size); } \

template<typename T, typename I>
__device__ void index_add(
    const I *ids,
    const size_t ids_dim_size,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < ids_dim_size; ++j) {
              const I idx = ids[j];
              const size_t src_i = (pre * ids_dim_size + j) * right_size + post;
              if (idx < max_value<I>()) {
                assert(idx < dst_dim_size);
                const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
                out[dst_i] += inp[src_i];
              }
          }
      }
}

#define IA_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const size_t ids_dim_size, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { index_add(ids, ids_dim_size, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \

template<typename T, typename I>
__device__ void scatter(
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < src_dim_size; ++j) {
              const size_t src_i = (pre * src_dim_size + j) * right_size + post;
              const I idx = ids[src_i];
              if (idx < max_value<I>()) {
                assert(idx < dst_dim_size);
                const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
                out[dst_i] = inp[src_i];
              }
          }
      }
}

template<typename T, typename I>
__device__ void scatter_add(
    const I *ids,
    const T *inp,
    T *out,
    const size_t left_size,
    const size_t src_dim_size,
    const size_t dst_dim_size,
    const size_t right_size
) {
      const size_t numel = left_size * right_size;
      for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
          const size_t pre = i / right_size;
          const size_t post = i % right_size;
          for (unsigned int j = 0; j < src_dim_size; ++j) {
              const size_t src_i = (pre * src_dim_size + j) * right_size + post;
              const I idx = ids[src_i];
              if (idx < max_value<I>()) {
                assert(idx < dst_dim_size);
                const size_t dst_i = (pre * dst_dim_size + idx) * right_size + post;
                out[dst_i] += inp[src_i];
              }
          }
      }
}

#define S_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { scatter(ids, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \

#define SA_OP(TYPENAME, INDEX_TYPENAME, FN_NAME) \
extern "C" __global__ void FN_NAME(  \
    const INDEX_TYPENAME *ids, \
    const TYPENAME *inp, \
    TYPENAME *out, \
    const size_t left_size, \
    const size_t src_dim_size, \
    const size_t dst_dim_size, \
    const size_t right_size \
) { scatter_add(ids, inp, out, left_size, src_dim_size, dst_dim_size, right_size); } \


#if __CUDA_ARCH__ >= 800
IS_OP(__hip_bfloat16, int64_t, is_i64_bf16)
IS_OP(__hip_bfloat16, uint32_t, is_u32_bf16)
IS_OP(__hip_bfloat16, uint8_t, is_u8_bf16)
GATHER_OP(__hip_bfloat16, int64_t, gather_i64_bf16)
GATHER_OP(__hip_bfloat16, uint32_t, gather_u32_bf16)
GATHER_OP(__hip_bfloat16, uint8_t, gather_u8_bf16)
IA_OP(__hip_bfloat16, int64_t, ia_i64_bf16)
IA_OP(__hip_bfloat16, uint32_t, ia_u32_bf16)
IA_OP(__hip_bfloat16, uint8_t, ia_u8_bf16)
SA_OP(__hip_bfloat16, int64_t, sa_i64_bf16)
SA_OP(__hip_bfloat16, uint32_t, sa_u32_bf16)
SA_OP(__hip_bfloat16, uint8_t, sa_u8_bf16)
S_OP(__hip_bfloat16, int64_t, s_i64_bf16)
S_OP(__hip_bfloat16, uint32_t, s_u32_bf16)
S_OP(__hip_bfloat16, uint8_t, s_u8_bf16)
#endif

#if __CUDA_ARCH__ >= 530
IS_OP(__half, int64_t, is_i64_f16)
IS_OP(__half, uint32_t, is_u32_f16)
IS_OP(__half, uint8_t, is_u8_f16)
GATHER_OP(__half, int64_t, gather_i64_f16)
GATHER_OP(__half, uint32_t, gather_u32_f16)
GATHER_OP(__half, uint8_t, gather_u8_f16)
IA_OP(__half, int64_t, ia_i64_f16)
IA_OP(__half, uint32_t, ia_u32_f16)
IA_OP(__half, uint8_t, ia_u8_f16)
SA_OP(__half, int64_t, sa_i64_f16)
SA_OP(__half, uint32_t, sa_u32_f16)
SA_OP(__half, uint8_t, sa_u8_f16)
S_OP(__half, int64_t, s_i64_f16)
S_OP(__half, uint32_t, s_u32_f16)
S_OP(__half, uint8_t, s_u8_f16)
#endif

IS_OP(float, int64_t, is_i64_f32)
IS_OP(double, int64_t, is_i64_f64)
IS_OP(uint8_t, int64_t, is_i64_u8)
IS_OP(uint32_t, int64_t, is_i64_u32)
IS_OP(int64_t, int64_t, is_i64_i64)

IS_OP(float, uint32_t, is_u32_f32)
IS_OP(double, uint32_t, is_u32_f64)
IS_OP(uint8_t, uint32_t, is_u32_u8)
IS_OP(int64_t, uint32_t, is_u32_i64)
IS_OP(uint32_t, uint32_t, is_u32_u32)

IS_OP(float, uint8_t, is_u8_f32)
IS_OP(double, uint8_t, is_u8_f64)
IS_OP(uint8_t, uint8_t, is_u8_u8)
IS_OP(uint32_t, uint8_t, is_u8_u32)
IS_OP(int64_t, uint8_t, is_u8_i64)

GATHER_OP(float, int64_t, gather_i64_f32)
GATHER_OP(double, int64_t, gather_i64_f64)
GATHER_OP(uint8_t, int64_t, gather_i64_u8)
GATHER_OP(uint32_t, int64_t, gather_i64_u32)
GATHER_OP(int64_t, int64_t, gather_i64_i64)

GATHER_OP(float, uint32_t, gather_u32_f32)
GATHER_OP(double, uint32_t, gather_u32_f64)
GATHER_OP(uint8_t, uint32_t, gather_u32_u8)
GATHER_OP(int64_t, uint32_t, gather_u32_i64)
GATHER_OP(uint32_t, uint32_t, gather_u32_u32)

GATHER_OP(float, uint8_t, gather_u8_f32)
GATHER_OP(double, uint8_t, gather_u8_f64)
GATHER_OP(uint8_t, uint8_t, gather_u8_u8)
GATHER_OP(uint32_t, uint8_t, gather_u8_u32)
GATHER_OP(int64_t, uint8_t, gather_u8_i64)

IA_OP(float, int64_t, ia_i64_f32)
IA_OP(double, int64_t, ia_i64_f64)
IA_OP(uint8_t, int64_t, ia_i64_u8)
IA_OP(int64_t, int64_t, ia_i64_i64)
IA_OP(uint32_t, int64_t, ia_i64_u32)

IA_OP(float, uint32_t, ia_u32_f32)
IA_OP(double, uint32_t, ia_u32_f64)
IA_OP(uint8_t, uint32_t, ia_u32_u8)
IA_OP(int64_t, uint32_t, ia_u32_i64)
IA_OP(uint32_t, uint32_t, ia_u32_u32)

IA_OP(float, uint8_t, ia_u8_f32)
IA_OP(double, uint8_t, ia_u8_f64)
IA_OP(uint8_t, uint8_t, ia_u8_u8)
IA_OP(uint32_t, uint8_t, ia_u8_u32)
IA_OP(int64_t, uint8_t, ia_u8_i64)

SA_OP(float, int64_t, sa_i64_f32)
SA_OP(double, int64_t, sa_i64_f64)
SA_OP(uint8_t, int64_t, sa_i64_u8)
SA_OP(int64_t, int64_t, sa_i64_i64)
SA_OP(uint32_t, int64_t, sa_i64_u32)

SA_OP(float, uint32_t, sa_u32_f32)
SA_OP(double, uint32_t, sa_u32_f64)
SA_OP(uint8_t, uint32_t, sa_u32_u8)
SA_OP(int64_t, uint32_t, sa_u32_i64)
SA_OP(uint32_t, uint32_t, sa_u32_u32)

SA_OP(float, uint8_t, sa_u8_f32)
SA_OP(double, uint8_t, sa_u8_f64)
SA_OP(uint8_t, uint8_t, sa_u8_u8)
SA_OP(uint32_t, uint8_t, sa_u8_u32)
SA_OP(int64_t, uint8_t, sa_u8_i64)

S_OP(float, int64_t, s_i64_f32)
S_OP(double, int64_t, s_i64_f64)
S_OP(uint8_t, int64_t, s_i64_u8)
S_OP(int64_t, int64_t, s_i64_i64)
S_OP(uint32_t, int64_t, s_i64_u32)

S_OP(float, uint32_t, s_u32_f32)
S_OP(double, uint32_t, s_u32_f64)
S_OP(uint8_t, uint32_t, s_u32_u8)
S_OP(int64_t, uint32_t, s_u32_i64)
S_OP(uint32_t, uint32_t, s_u32_u32)

S_OP(float, uint8_t, s_u8_f32)
S_OP(double, uint8_t, s_u8_f64)
S_OP(uint8_t, uint8_t, s_u8_u8)
S_OP(uint32_t, uint8_t, s_u8_u32)
S_OP(int64_t, uint8_t, s_u8_i64)
