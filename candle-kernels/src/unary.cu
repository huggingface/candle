#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
#include<math.h>
#include<stdint.h>
#include "cuda_utils.cuh"

#define UNARY_OP(TYPENAME, FN_NAME, FUNC) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const TYPENAME *inp, \
    TYPENAME *out \
) { \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    if (info == nullptr || is_contiguous(num_dims, dims, strides)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            TYPENAME x = inp ? inp[i] : out[i]; \
            out[i] = FUNC; \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            TYPENAME x = inp ? inp[strided_i] : out[i]; \
            out[i] = FUNC; \
        } \
    } \
} \

template<typename T>
__device__ __forceinline__ T gelu_erf_fwd(T x) {
  return x * normcdfg(x);
}

template<typename T>
__device__ __forceinline__ T gelu_fwd(T x) {
    T x_sq = x * x;
    T x_cube = x_sq * x;
    T alpha = x + static_cast<T>(0.044715) * x_cube;
    return static_cast<T>(0.5) * x * (static_cast<T>(1.0) + tanhg(static_cast<T>(M_2_SQRTPI * M_SQRT1_2) * alpha));
}

template<typename T>
__device__ __forceinline__ T elu_fwd(T x, T alpha) {
  if (x > static_cast<T>(0)) {
    return x;
  }
  return alpha * (expg(x) - static_cast<T>(1));
}

template<typename T>
__device__ __forceinline__ T relu_fwd(T x) {
    T zero = 0.;
    return maxg(x, zero);
}

template<typename T>
__device__ __forceinline__ T silu_fwd(T x) {
    return x / (static_cast<T>(1) + expg(-x));
}

template<typename T>
__device__ __forceinline__ T sigmoid_fwd(T x) {
    return recipg(static_cast<T>(1) + expg(-x));
}

#define UNARY_OP1(TYPENAME, FN_NAME, FUNC) \
extern "C" __global__ void FN_NAME( \
    const size_t numel, \
    const size_t num_dims, \
    const size_t *info, \
    const TYPENAME param, \
    const TYPENAME *inp, \
    TYPENAME *out \
) { \
    const size_t *dims = info; \
    const size_t *strides = info + num_dims; \
    if (info == nullptr || is_contiguous(num_dims, dims, strides)) { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            TYPENAME x = inp ? inp[i] : out[i]; \
            out[i] = FUNC; \
        } \
    } \
    else { \
        for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) { \
            unsigned strided_i = get_strided_index(i, num_dims, dims, strides); \
            TYPENAME x = inp ? inp[strided_i] : out[i]; \
            out[i] = FUNC; \
        } \
    } \
} \

template<typename T>
__device__ T sign_(T t) {
  return static_cast<T>(t > static_cast<T>(0)) - static_cast<T>(t < static_cast<T>(0));
}


#if __CUDA_ARCH__ >= 800
UNARY_OP(__hip_bfloat16, ucopy_bf16, x)
UNARY_OP(__hip_bfloat16, uneg_bf16, -x)
UNARY_OP(__hip_bfloat16, urecip_bf16, recipg(x))
UNARY_OP(__hip_bfloat16, uexp_bf16, expg(x))
UNARY_OP(__hip_bfloat16, ulog_bf16, logg(x))
UNARY_OP(__hip_bfloat16, usin_bf16, sing(x))
UNARY_OP(__hip_bfloat16, ucos_bf16, cosg(x))
UNARY_OP(__hip_bfloat16, utanh_bf16, tanhg(x))
UNARY_OP(__hip_bfloat16, uerf_bf16, erfg(x))
UNARY_OP(__hip_bfloat16, uceil_bf16, ceilg(x))
UNARY_OP(__hip_bfloat16, ufloor_bf16, floorg(x))
UNARY_OP(__hip_bfloat16, uround_bf16, roundg(x))
UNARY_OP(__hip_bfloat16, unormcdf_bf16, normcdfg(x))
UNARY_OP(__hip_bfloat16, uabs_bf16, absg(x))
UNARY_OP(__hip_bfloat16, usqr_bf16, x*x)
UNARY_OP(__hip_bfloat16, usqrt_bf16, sqrtg(x))
UNARY_OP(__hip_bfloat16, ugelu_bf16, gelu_fwd(x))
UNARY_OP(__hip_bfloat16, ugelu_erf_bf16, gelu_erf_fwd(x))
UNARY_OP(__hip_bfloat16, urelu_bf16, relu_fwd(x))
UNARY_OP1(__hip_bfloat16, uelu_bf16, elu_fwd(x, param))
UNARY_OP(__hip_bfloat16, usilu_bf16, silu_fwd(x))
UNARY_OP1(__hip_bfloat16, upowf_bf16, powg(x, param))
UNARY_OP(__hip_bfloat16, usign_bf16, sign_(x))
UNARY_OP(__hip_bfloat16, usigmoid_bf16, sigmoid_fwd(x))

#define F8E4M3_TO_FLOAT(x) __half2float(__hip_cvt_fp8_to_halfraw(x.__x, __HIP_E4M3_FNUZ))

UNARY_OP(__hip_fp8_e4m3_fnuz, ucopy_f8_e4m3, x)
UNARY_OP(__hip_fp8_e4m3_fnuz, uneg_fp8_e4m3, __hip_fp8_e4m3_fnuz(-F8E4M3_TO_FLOAT(x)))
UNARY_OP(__hip_fp8_e4m3_fnuz, urecip_fp8_e4m3, recipg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, uexp_fp8_e4m3, expg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, ulog_fp8_e4m3, logg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, usin_fp8_e4m3, sing(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, ucos_fp8_e4m3, cosg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, utanh_fp8_e4m3, tanhg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, uerf_fp8_e4m3, erfg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, uceil_fp8_e4m3, ceilg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, ufloor_fp8_e4m3, floorg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, uround_fp8_e4m3, roundg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, unormcdf_fp8_e4m3, normcdfg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, uabs_fp8_e4m3, absg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, usqr_fp8_e4m3, __hip_fp8_e4m3_fnuz(F8E4M3_TO_FLOAT(x)*F8E4M3_TO_FLOAT(x)))
UNARY_OP(__hip_fp8_e4m3_fnuz, usqrt_fp8_e4m3, sqrtg(x))
UNARY_OP(__hip_fp8_e4m3_fnuz, ugelu_fp8_e4m3, __hip_fp8_e4m3_fnuz(gelu_fwd(F8E4M3_TO_FLOAT(x))))
UNARY_OP(__hip_fp8_e4m3_fnuz, ugelu_erf_fp8_e4m3, __hip_fp8_e4m3_fnuz(gelu_erf_fwd(F8E4M3_TO_FLOAT(x))))
UNARY_OP(__hip_fp8_e4m3_fnuz, urelu_fp8_e4m3, __hip_fp8_e4m3_fnuz(relu_fwd(F8E4M3_TO_FLOAT(x))))
UNARY_OP1(__hip_fp8_e4m3_fnuz, uelu_fp8_e4m3, __hip_fp8_e4m3_fnuz(elu_fwd(F8E4M3_TO_FLOAT(x), F8E4M3_TO_FLOAT(param))))
UNARY_OP(__hip_fp8_e4m3_fnuz, usilu_fp8_e4m3, __hip_fp8_e4m3_fnuz(silu_fwd(F8E4M3_TO_FLOAT(x))))
UNARY_OP1(__hip_fp8_e4m3_fnuz, upowf_fp8_e4m3, powg(x, param))
UNARY_OP(__hip_fp8_e4m3_fnuz, usign_fp8_e4m3, __hip_fp8_e4m3_fnuz(sign_(F8E4M3_TO_FLOAT(x))))
UNARY_OP(__hip_fp8_e4m3_fnuz, usigmoid_fp8_e4m3, __hip_fp8_e4m3_fnuz(sigmoid_fwd(F8E4M3_TO_FLOAT(x))))
#endif

#if __CUDA_ARCH__ >= 530
UNARY_OP(__half, ucopy_f16, x)
UNARY_OP(__half, uneg_f16, -x)
UNARY_OP(__half, urecip_f16, recipg(x))
UNARY_OP(__half, uexp_f16, expg(x))
UNARY_OP(__half, ulog_f16, logg(x))
UNARY_OP(__half, usin_f16, sing(x))
UNARY_OP(__half, ucos_f16, cosg(x))
UNARY_OP(__half, utanh_f16, tanhg(x))
UNARY_OP(__half, uerf_f16, erfg(x))
UNARY_OP(__half, uceil_f16, ceilg(x))
UNARY_OP(__half, ufloor_f16, floorg(x))
UNARY_OP(__half, uround_f16, roundg(x))
UNARY_OP(__half, unormcdf_f16, normcdfg(x))
UNARY_OP(__half, uabs_f16, absg(x))
UNARY_OP(__half, usqr_f16, x*x)
UNARY_OP(__half, usqrt_f16, sqrtg(x))
UNARY_OP(__half, ugelu_f16, gelu_fwd(x))
UNARY_OP(__half, ugelu_erf_f16, gelu_erf_fwd(x))
UNARY_OP(__half, urelu_f16, relu_fwd(x))
UNARY_OP1(__half, uelu_f16, elu_fwd(x, param))
UNARY_OP(__half, usilu_f16, silu_fwd(x))
UNARY_OP1(__half, upowf_f16, powg(x, param))
UNARY_OP(__half, usign_f16, sign_(x))
UNARY_OP(__half, usigmoid_f16, sigmoid_fwd(x))
#endif

UNARY_OP(uint8_t, ucopy_u8, x)
UNARY_OP(uint32_t, ucopy_u32, x)
UNARY_OP(int64_t, ucopy_i64, x)
UNARY_OP(float, ucopy_f32, x)
UNARY_OP(double, ucopy_f64, x)
UNARY_OP(float, uneg_f32, -x)
UNARY_OP(double, uneg_f64, -x)
UNARY_OP(float, urecip_f32, recipg(x))
UNARY_OP(double, urecip_f64, recipg(x))
UNARY_OP(float, uexp_f32, expg(x))
UNARY_OP(double, uexp_f64, expg(x))
UNARY_OP(float, ulog_f32, logg(x))
UNARY_OP(double, ulog_f64, logg(x))
UNARY_OP(float, usin_f32, sing(x))
UNARY_OP(double, usin_f64, sing(x))
UNARY_OP(float, ucos_f32, cosg(x))
UNARY_OP(double, ucos_f64, cosg(x))
UNARY_OP(float, utanh_f32, tanhg(x))
UNARY_OP(double, utanh_f64, tanhg(x))
UNARY_OP(float, uerf_f32, erfg(x))
UNARY_OP(double, uerf_f64, erfg(x))
UNARY_OP(float, uceil_f32, ceilg(x))
UNARY_OP(double, uceil_f64, ceilg(x))
UNARY_OP(float, ufloor_f32, floorg(x))
UNARY_OP(double, ufloor_f64, floorg(x))
UNARY_OP(float, uround_f32, roundg(x))
UNARY_OP(double, uround_f64, roundg(x))
UNARY_OP(float, unormcdf_f32, normcdfg(x))
UNARY_OP(double, unormcdf_f64, normcdfg(x))
UNARY_OP(float, uabs_f32, absg(x))
UNARY_OP(double, uabs_f64, absg(x))
UNARY_OP(float, usqr_f32, x*x)
UNARY_OP(double, usqr_f64, x*x)
UNARY_OP(float, usqrt_f32, sqrtg(x))
UNARY_OP(double, usqrt_f64, sqrtg(x))
UNARY_OP(float, ugelu_f32, gelu_fwd(x))
UNARY_OP(double, ugelu_f64, gelu_fwd(x))
UNARY_OP(float, ugelu_erf_f32, gelu_erf_fwd(x))
UNARY_OP(double, ugelu_erf_f64, gelu_erf_fwd(x))
UNARY_OP(float, urelu_f32, relu_fwd(x))
UNARY_OP(double, urelu_f64, relu_fwd(x))
UNARY_OP1(float, uelu_f32, elu_fwd(x, param))
UNARY_OP1(double, uelu_f64, elu_fwd(x, param))
UNARY_OP(float, usilu_f32, silu_fwd(x))
UNARY_OP(double, usilu_f64, silu_fwd(x))
UNARY_OP1(float, upowf_f32, powg(x, param))
UNARY_OP1(double, upowf_f64, powg(x, param))
UNARY_OP(float, usign_f32, sign_(x))
UNARY_OP(double, usign_f64, sign_(x))
UNARY_OP(float, usigmoid_f32, sigmoid_fwd(x))
UNARY_OP(double, usigmoid_f64, sigmoid_fwd(x))
