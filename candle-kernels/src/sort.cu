#include "hip/hip_runtime.h"
// Adapted from https://github.com/ggerganov/llama.cpp/blob/master/ggml-cuda/argsort.cu
#define SORT_ORDER_ASC 1
#define SORT_ORDER_DESC 0
#include "cuda_utils.cuh"
#include<stdint.h>

template<typename T>
static inline __device__ void ggml_cuda_swap(T & a, T & b) {
    T tmp = a;
    a = b;
    b = tmp;
}

template<int order, typename T>
static __device__ void k_argsort(const T * x, uint32_t * dst, const int ncols, int ncols_pad) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.y;

    if (col >= ncols_pad) {
        return;
    }

    const T * x_row = x + row * ncols;
    extern __shared__ int dst_row[];

    // initialize indices
    dst_row[col] = col;

    __syncthreads();

    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (dst_row[col] >= ncols ||
                        (dst_row[ixj] < ncols && (order == SORT_ORDER_ASC ?
                            x_row[dst_row[col]] > x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] < x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                } else {
                    if (dst_row[ixj] >= ncols ||
                        (dst_row[col] < ncols && (order == SORT_ORDER_ASC ?
                            x_row[dst_row[col]] < x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] > x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }

    // copy the result to dst without the padding
    if (col < ncols) {
        dst[row * ncols + col] = dst_row[col];
    }
}

#define ASORT_OP(TYPENAME, RUST_NAME) \
extern "C" __global__ void asort_asc_##RUST_NAME(  \
    const TYPENAME * x, uint32_t * dst, const int ncols, int ncols_pad \
) { \
    k_argsort<SORT_ORDER_ASC>(x, dst, ncols, ncols_pad); \
} \
extern "C" __global__ void asort_desc_##RUST_NAME(  \
    const TYPENAME * x, uint32_t * dst, const int ncols, int ncols_pad \
) { \
    k_argsort<SORT_ORDER_DESC>(x, dst, ncols, ncols_pad); \
} \
 
#if __CUDA_ARCH__ >= 800
ASORT_OP(__hip_bfloat16, bf16)

// NOTE: No sort ops for f8
// ASORT_OP(__hip_fp8_e4m3_fnuz, fp8_e4m3)
#endif

#if __CUDA_ARCH__ >= 530
ASORT_OP(__half, f16)
#endif

ASORT_OP(float, f32)
ASORT_OP(double, f64)
ASORT_OP(uint8_t, u8)
ASORT_OP(uint32_t, u32)
ASORT_OP(int64_t, i64)
