
#include <hip/hip_runtime.h>
#include<stdint.h>
#include "hip/hip_fp16.h"

template<typename T>
__device__ void fill_with(T *buf, T value, const size_t numel) {
    for (unsigned int i = blockIdx.x * blockDim.x + threadIdx.x; i < numel; i += blockDim.x * gridDim.x) {
        buf[i] = value;
    }
}
extern "C" __global__ void fill_u8(uint8_t *buf, uint8_t value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_u32(uint32_t *buf, uint32_t value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_i64(int64_t *buf, int64_t value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f32(float *buf, float value, const size_t numel) { fill_with(buf, value, numel); }
extern "C" __global__ void fill_f64(double *buf, double value, const size_t numel) { fill_with(buf, value, numel); }

template<typename T>
__device__ void copy2d(const T *src, T *dst, uint32_t d1, uint32_t d2, uint32_t src_s, uint32_t dst_s) {
  uint32_t idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx >= d1 * d2) {
    return;
  }
  uint32_t idx1 = idx / d2;
  uint32_t idx2 = idx - d2 * idx1;
  dst[idx1 * dst_s + idx2] = src[idx1 * src_s + idx2];
}

#define COPY2D_OP(TYPENAME, FNNAME) \
extern "C" __global__ \
void FNNAME(const TYPENAME *src, TYPENAME *dst, uint32_t d1, uint32_t d2, uint32_t src_s, uint32_t dst_s) { \
  copy2d(src, dst, d1, d2, src_s, dst_s); \
} \

COPY2D_OP(float, copy2d_f32)
COPY2D_OP(double, copy2d_f64)
COPY2D_OP(uint8_t, copy2d_u8)
COPY2D_OP(uint32_t, copy2d_u32)
COPY2D_OP(int64_t, copy2d_i64)

#if __CUDA_ARCH__ >= 530
extern "C" __global__ void fill_f16(__half *buf, __half value, const size_t numel) { fill_with(buf, value, numel); }
COPY2D_OP(__half, copy2d_f16)
#endif

#if __CUDA_ARCH__ >= 800
#include <cuda_bf16.h>
#include <cuda_fp8.h>

extern "C" __global__ void fill_bf16(__nv_bfloat16 *buf, __nv_bfloat16 value, const size_t numel) { fill_with(buf, value, numel); }
COPY2D_OP(__nv_bfloat16, copy2d_bf16)

extern "C" __global__ void fill_f8_e4m3(__nv_fp8_e4m3 *buf, __nv_fp8_e4m3 value, const size_t numel) { fill_with(buf, value, numel); }
COPY2D_OP(__nv_fp8_e4m3, copy2d_f8_e4m3)
#endif
