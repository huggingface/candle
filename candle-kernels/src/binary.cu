#include "hip/hip_runtime.h"
#include "binary_op_macros.cuh"
#include<stdint.h>

#if __CUDA_ARCH__ >= 800
BINARY_OP(__hip_bfloat16, badd_bf16, x + y)
BINARY_OP(__hip_bfloat16, bdiv_bf16, x / y)
BINARY_OP(__hip_bfloat16, bmul_bf16, x * y)
BINARY_OP(__hip_bfloat16, bsub_bf16, x - y)
BINARY_OP(__hip_bfloat16, bmaximum_bf16, maxg(x, y))
BINARY_OP(__hip_bfloat16, bminimum_bf16, ming(x, y))
BINARY_OP_OUT(__hip_bfloat16, uint8_t, eq_bf16, x == y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, ne_bf16, x != y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, lt_bf16, x < y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, le_bf16, x <= y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, gt_bf16, x > y)
BINARY_OP_OUT(__hip_bfloat16, uint8_t, ge_bf16, x >= y)

#define F8E4M3_TO_FLOAT(x) __half2float(__hip_cvt_fp8_to_halfraw(x.__x, __HIP_E4M3_FNUZ))

BINARY_OP(__hip_fp8_e4m3_fnuz, badd_f8_e4m3, __hip_fp8_e4m3_fnuz(F8E4M3_TO_FLOAT(x) + F8E4M3_TO_FLOAT(y)))
BINARY_OP(__hip_fp8_e4m3_fnuz, bdiv_f8_e4m3, __hip_fp8_e4m3_fnuz(F8E4M3_TO_FLOAT(x) / F8E4M3_TO_FLOAT(y)))
BINARY_OP(__hip_fp8_e4m3_fnuz, bmul_f8_e4m3, __hip_fp8_e4m3_fnuz(F8E4M3_TO_FLOAT(x) * F8E4M3_TO_FLOAT(y)))
BINARY_OP(__hip_fp8_e4m3_fnuz, bsub_f8_e4m3, __hip_fp8_e4m3_fnuz(F8E4M3_TO_FLOAT(x) - F8E4M3_TO_FLOAT(y)))
BINARY_OP(__hip_fp8_e4m3_fnuz, bmaximum_f8_e4m3, maxg(x, y))
BINARY_OP(__hip_fp8_e4m3_fnuz, bminimum_f8_e4m3, ming(x, y))
BINARY_OP_OUT(__hip_fp8_e4m3_fnuz, uint8_t, eq_f8_e4m3, F8E4M3_TO_FLOAT(x) == F8E4M3_TO_FLOAT(y))
BINARY_OP_OUT(__hip_fp8_e4m3_fnuz, uint8_t, ne_f8_e4m3, F8E4M3_TO_FLOAT(x) != F8E4M3_TO_FLOAT(y))
BINARY_OP_OUT(__hip_fp8_e4m3_fnuz, uint8_t, lt_f8_e4m3, F8E4M3_TO_FLOAT(x) < F8E4M3_TO_FLOAT(y))
BINARY_OP_OUT(__hip_fp8_e4m3_fnuz, uint8_t, le_f8_e4m3, F8E4M3_TO_FLOAT(x) <= F8E4M3_TO_FLOAT(y))
BINARY_OP_OUT(__hip_fp8_e4m3_fnuz, uint8_t, gt_f8_e4m3, F8E4M3_TO_FLOAT(x) > F8E4M3_TO_FLOAT(y))
BINARY_OP_OUT(__hip_fp8_e4m3_fnuz, uint8_t, ge_f8_e4m3, F8E4M3_TO_FLOAT(x) >= F8E4M3_TO_FLOAT(y))
#endif

#if __CUDA_ARCH__ >= 530
BINARY_OP(__half, badd_f16, x + y)
BINARY_OP(__half, bdiv_f16, x / y)
BINARY_OP(__half, bmul_f16, x * y)
BINARY_OP(__half, bsub_f16, x - y)
BINARY_OP(__half, bmaximum_f16, maxg(x, y))
BINARY_OP(__half, bminimum_f16, ming(x, y))
BINARY_OP_OUT(__half, uint8_t, eq_f16, x == y)
BINARY_OP_OUT(__half, uint8_t, ne_f16, x != y)
BINARY_OP_OUT(__half, uint8_t, lt_f16, x < y)
BINARY_OP_OUT(__half, uint8_t, le_f16, x <= y)
BINARY_OP_OUT(__half, uint8_t, gt_f16, x > y)
BINARY_OP_OUT(__half, uint8_t, ge_f16, x >= y)
#endif

BINARY_OP(float, badd_f32, x + y)
BINARY_OP(double, badd_f64, x + y);
BINARY_OP(uint8_t, badd_u8, x + y);
BINARY_OP(uint32_t, badd_u32, x + y);
BINARY_OP(int64_t, badd_i64, x + y);
BINARY_OP(float, bdiv_f32, x / y)
BINARY_OP(double, bdiv_f64, x / y);
BINARY_OP(uint8_t, bdiv_u8, x / y);
BINARY_OP(uint32_t, bdiv_u32, x / y);
BINARY_OP(int64_t, bdiv_i64, x / y);
BINARY_OP(float, bmul_f32, x * y)
BINARY_OP(double, bmul_f64, x * y);
BINARY_OP(uint8_t, bmul_u8, x * y);
BINARY_OP(uint32_t, bmul_u32, x * y);
BINARY_OP(int64_t, bmul_i64, x * y);
BINARY_OP(float, bsub_f32, x - y)
BINARY_OP(double, bsub_f64, x - y);
BINARY_OP(uint8_t, bsub_u8, x - y);
BINARY_OP(uint32_t, bsub_u32, x - y);
BINARY_OP(int64_t, bsub_i64, x - y);
BINARY_OP(float, bminimum_f32, ming(x, y));
BINARY_OP(double, bminimum_f64, ming(x, y));
BINARY_OP(uint8_t, bminimum_u8, ming(x, y));
BINARY_OP(uint32_t, bminimum_u32, ming(x, y));
BINARY_OP(int64_t, bminimum_i64, ming(x, y));
BINARY_OP(float, bmaximum_f32, maxg(x, y));
BINARY_OP(double, bmaximum_f64, maxg(x, y));
BINARY_OP(uint8_t, bmaximum_u8, maxg(x, y));
BINARY_OP(uint32_t, bmaximum_u32, maxg(x, y));
BINARY_OP(int64_t, bmaximum_i64, maxg(x, y));

BINARY_OP_OUT(float, uint8_t, eq_f32, x == y)
BINARY_OP_OUT(double, uint8_t, eq_f64, x == y)
BINARY_OP_OUT(uint8_t, uint8_t, eq_u8, x == y)
BINARY_OP_OUT(uint32_t, uint8_t, eq_u32, x == y)
BINARY_OP_OUT(int64_t, uint8_t, eq_i64, x == y)

BINARY_OP_OUT(float, uint8_t, ne_f32, x != y)
BINARY_OP_OUT(double, uint8_t, ne_f64, x != y)
BINARY_OP_OUT(uint8_t, uint8_t, ne_u8, x != y)
BINARY_OP_OUT(uint32_t, uint8_t, ne_u32, x != y)
BINARY_OP_OUT(int64_t, uint8_t, ne_i64, x != y)

BINARY_OP_OUT(float, uint8_t, lt_f32, x < y)
BINARY_OP_OUT(double, uint8_t, lt_f64, x < y)
BINARY_OP_OUT(uint8_t, uint8_t, lt_u8, x < y)
BINARY_OP_OUT(uint32_t, uint8_t, lt_u32, x < y)
BINARY_OP_OUT(int64_t, uint8_t, lt_i64, x < y)

BINARY_OP_OUT(float, uint8_t, le_f32, x <= y)
BINARY_OP_OUT(double, uint8_t, le_f64, x <= y)
BINARY_OP_OUT(uint8_t, uint8_t, le_u8, x <= y)
BINARY_OP_OUT(uint32_t, uint8_t, le_u32, x <= y)
BINARY_OP_OUT(int64_t, uint8_t, le_i64, x <= y)

BINARY_OP_OUT(float, uint8_t, gt_f32, x > y)
BINARY_OP_OUT(double, uint8_t, gt_f64, x > y)
BINARY_OP_OUT(uint8_t, uint8_t, gt_u8, x > y)
BINARY_OP_OUT(uint32_t, uint8_t, gt_u32, x > y)
BINARY_OP_OUT(int64_t, uint8_t, gt_i64, x > y)

BINARY_OP_OUT(float, uint8_t, ge_f32, x >= y)
BINARY_OP_OUT(double, uint8_t, ge_f64, x >= y)
BINARY_OP_OUT(uint8_t, uint8_t, ge_u8, x >= y)
BINARY_OP_OUT(uint32_t, uint8_t, ge_u32, x >= y)
BINARY_OP_OUT(int64_t, uint8_t, ge_i64, x >= y)
