// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar, Pradeep Ramani, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_gqa_<cutlass::half_t, 256, 32>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim256_gqa<cutlass::half_t, 32>(params, stream);
}
