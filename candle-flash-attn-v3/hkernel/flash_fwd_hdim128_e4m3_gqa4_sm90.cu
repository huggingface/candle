// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar, Pradeep Ramani, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_gqa_<cutlass::float_e4m3_t, 128, 4>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim128_fp8_gqa<cutlass::float_e4m3_t, 4>(params, stream);
}
