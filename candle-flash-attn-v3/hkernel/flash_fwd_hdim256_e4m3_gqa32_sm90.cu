// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar, Pradeep Ramani, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_gqa_<cutlass::float_e4m3_t, 256, 32>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim256_fp8_gqa<cutlass::float_e4m3_t, 32>(params, stream);
}
