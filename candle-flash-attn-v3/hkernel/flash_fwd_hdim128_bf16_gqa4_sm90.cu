// Copyright (c) 2024, Jay Shah, Ganesh Bikshandi, Ying Zhang, Vijay Thakkar, Pradeep Ramani, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_gqa_<cutlass::bfloat16_t, 128, 4>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim128_gqa<cutlass::bfloat16_t, 4>(params, stream);
}
