// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_<cutlass::bfloat16_t, 160, true>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim160<cutlass::bfloat16_t, true>(params, stream);
}
