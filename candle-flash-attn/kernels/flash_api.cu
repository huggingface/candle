#include "hip/hip_runtime.h"
#include "kernels.h"
#include "kernel_helpers.h"
#include "flash_fwd_launch_template.h"

void run_mha_fwd(Flash_fwd_params &params, hipStream_t stream) {
  FP16_SWITCH(!params.is_bf16, [&] {
      HEADDIM_SWITCH(params.d, [&] {
          BOOL_SWITCH(params.is_causal, Is_causal, [&] {
              run_mha_fwd_<elem_type, kHeadDim, Is_causal>(params, stream);
          });
      });
  });
}

extern "C" void run_mha(
    void *q_ptr,
    void *k_ptr,
    void *v_ptr,
    void *o_ptr,
    void *softmax_lse_ptr,
    void *alibi_slopes_ptr,

    int32_t *cu_seqlens_q_ptr,
    int32_t *cu_seqlens_k_ptr,

    uint32_t q_batch_stride,
    uint32_t k_batch_stride,
    uint32_t v_batch_stride,
    uint32_t o_batch_stride,
    uint32_t alibi_slopes_batch_stride,

    uint32_t q_row_stride,
    uint32_t k_row_stride,
    uint32_t v_row_stride,
    uint32_t o_row_stride,

    uint32_t q_head_stride,
    uint32_t k_head_stride,
    uint32_t v_head_stride,
    uint32_t o_head_stride,

    uint32_t b,
    uint32_t h,
    uint32_t h_k,
    uint32_t d,
    uint32_t d_rounded,
    float softmax_scale,

    uint32_t seqlen_q,
    uint32_t seqlen_k,
    uint32_t seqlen_q_rounded,
    uint32_t seqlen_k_rounded,

    int is_bf16,
    int is_causal,

    int window_size_left,
    int window_size_right
) {
    Flash_fwd_params params;
    // Reset the parameters
    memset(&params, 0, sizeof(params));

    // Set the pointers and strides.
    params.q_ptr = q_ptr;
    params.k_ptr = k_ptr;
    params.v_ptr = v_ptr;
    params.o_ptr = o_ptr;

    params.softmax_lse_ptr = softmax_lse_ptr;
    params.alibi_slopes_ptr = alibi_slopes_ptr;

    // All stride are in elements, not bytes.
    params.q_batch_stride = q_batch_stride;
    params.k_batch_stride = k_batch_stride;
    params.v_batch_stride = v_batch_stride;
    params.o_batch_stride = o_batch_stride;
    params.alibi_slopes_batch_stride = alibi_slopes_batch_stride;

    params.q_row_stride = q_row_stride;
    params.k_row_stride = k_row_stride;
    params.v_row_stride = v_row_stride;
    params.o_row_stride = o_row_stride;
    params.q_head_stride = q_head_stride;
    params.k_head_stride = k_head_stride;
    params.v_head_stride = v_head_stride;
    params.o_head_stride = o_head_stride;

    // Set the dimensions.
    params.b = b;
    params.h = h;
    params.h_k = h_k;
    params.h_h_k_ratio = h / h_k;
    params.seqlen_q = seqlen_q;
    params.seqlen_k = seqlen_k;
    params.seqlen_q_rounded = seqlen_q_rounded;
    params.seqlen_k_rounded = seqlen_k_rounded;
    params.d = d;
    params.d_rounded = d_rounded;

    // Set the different scale values.
    params.scale_softmax = softmax_scale;
    params.scale_softmax_log2 = softmax_scale * M_LOG2E;

    params.p_dropout = 1.; // probability to keep
    params.p_dropout_in_uint8_t = uint8_t(std::floor(params.p_dropout * 255.0));
    params.rp_dropout = 1.f / params.p_dropout;
    params.scale_softmax_rp_dropout = params.rp_dropout * params.scale_softmax;
    params.is_bf16 = is_bf16;
    params.cu_seqlens_q = cu_seqlens_q_ptr;
    params.cu_seqlens_k = cu_seqlens_k_ptr;
    params.p_ptr = nullptr; // used for `return_softmax`.
    params.seqused_k = nullptr;

    params.is_causal = is_causal;
    params.window_size_left = window_size_left;
    params.window_size_right = window_size_right;

    params.is_seqlens_k_cumulative = true;
    params.num_splits = 1;

    hipStream_t stream = 0; // Use the default stream.
    run_mha_fwd(params, stream);
}
