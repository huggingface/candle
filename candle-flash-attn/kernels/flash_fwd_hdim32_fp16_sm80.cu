// Copyright (c) 2024, Tri Dao.
// Splitting the different head dimensions to different files to speed up compilation.
// This file is auto-generated. See "generate_kernels.py"

#include "flash_fwd_launch_template.h"

template<>
void run_mha_fwd_<cutlass::half_t, 32, false>(Flash_fwd_params &params, hipStream_t stream) {
    run_mha_fwd_hdim32<cutlass::half_t, false>(params, stream);
}
